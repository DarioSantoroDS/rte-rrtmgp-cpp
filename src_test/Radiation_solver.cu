#include "hip/hip_runtime.h"
/*
 * This file is imported from MicroHH (https://github.com/earth-system-radiation/earth-system-radiation)
 * and is adapted for the testing of the C++ interface to the
 * RTE+RRTMGP radiation code.
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/algorithm/string.hpp>
#include <cmath>
#include <numeric>
#include <hiprand/hiprand_kernel.h>

#include "Radiation_solver.h"
#include "Status.h"
#include "Netcdf_interface.h"

#include "Array.h"
#include "Gas_concs.h"
#include "Gas_optics_rrtmgp.h"
#include "Optical_props.h"
#include "Source_functions.h"
#include "Fluxes.h"
#include "Rte_lw.h"
#include "Rte_sw.h"
#include "rrtmgp_kernel_launcher_cuda.h"
#include "gpoint_kernel_launcher_cuda.h"


namespace
{
    template<typename TF>__global__
    void scaling_to_subset_kernel(
            const int ncol, const int ngpt, TF* __restrict__ toa_src, const TF* __restrict__ tsi_scaling)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        if ( ( icol < ncol)  )
        {
            const int idx = icol;
            toa_src[idx] *= tsi_scaling[icol];
        }
    }

    template<typename TF>
    void scaling_to_subset(
            const int ncol, const int ngpt, Array_gpu<TF,1>& toa_src, const Array_gpu<TF,1>& tsi_scaling)
    {
        const int block_col = 16;
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        
        dim3 grid_gpu(grid_col, 1);
        dim3 block_gpu(block_col, 1);
        scaling_to_subset_kernel<<<grid_gpu, block_gpu>>>(
            ncol, ngpt, toa_src.ptr(), tsi_scaling.ptr());
    }


    std::vector<std::string> get_variable_string(
            const std::string& var_name,
            std::vector<int> i_count,
            Netcdf_handle& input_nc,
            const int string_len,
            bool trim=true)
    {
        // Multiply all elements in i_count.
        int total_count = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Add the string length as the rightmost dimension.
        i_count.push_back(string_len);

        // Read the entire char array;
        std::vector<char> var_char;
        var_char = input_nc.get_variable<char>(var_name, i_count);

        std::vector<std::string> var;

        for (int n=0; n<total_count; ++n)
        {
            std::string s(var_char.begin()+n*string_len, var_char.begin()+(n+1)*string_len);
            if (trim)
                boost::trim(s);
            var.push_back(s);
        }

        return var;
    }

    template<typename TF>
    Gas_optics_rrtmgp_gpu<TF> load_and_init_gas_optics(
            const Gas_concs_gpu<TF>& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        const int n_temps = coef_nc.get_dimension_size("temperature");
        const int n_press = coef_nc.get_dimension_size("pressure");
        const int n_absorbers = coef_nc.get_dimension_size("absorber");
        const int n_char = coef_nc.get_dimension_size("string_len");
        const int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        const int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        const int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        const int n_layers = coef_nc.get_dimension_size("atmos_layer");
        const int n_bnds = coef_nc.get_dimension_size("bnd");
        const int n_gpts = coef_nc.get_dimension_size("gpt");
        const int n_pairs = coef_nc.get_dimension_size("pair");
        const int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        const int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        const int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        const int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<TF,2> band_lims(coef_nc.get_variable<TF>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<TF,1> press_ref(coef_nc.get_variable<TF>("press_ref", {n_press}), {n_press});
        Array<TF,1> temp_ref(coef_nc.get_variable<TF>("temp_ref", {n_temps}), {n_temps});

        TF temp_ref_p = coef_nc.get_variable<TF>("absorption_coefficient_ref_P");
        TF temp_ref_t = coef_nc.get_variable<TF>("absorption_coefficient_ref_T");
        TF press_ref_trop = coef_nc.get_variable<TF>("press_ref_trop");

        Array<TF,3> kminor_lower(
                coef_nc.get_variable<TF>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<TF,3> kminor_upper(
                coef_nc.get_variable<TF>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<BOOL_TYPE,1> minor_scales_with_density_lower(
                coef_nc.get_variable<BOOL_TYPE>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<BOOL_TYPE,1> minor_scales_with_density_upper(
                coef_nc.get_variable<BOOL_TYPE>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<BOOL_TYPE,1> scale_by_complement_lower(
                coef_nc.get_variable<BOOL_TYPE>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<BOOL_TYPE,1> scale_by_complement_upper(
                coef_nc.get_variable<BOOL_TYPE>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<TF,3> vmr_ref(
                coef_nc.get_variable<TF>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<TF,4> kmajor(
                coef_nc.get_variable<TF>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<TF,3> rayl_lower;
        Array<TF,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<TF>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<TF>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<TF,2> totplnk(
                    coef_nc.get_variable<TF>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<TF,4> planck_frac(
                    coef_nc.get_variable<TF>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_gpu<TF>(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<TF,1> solar_src_quiet(
                    coef_nc.get_variable<TF>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<TF,1> solar_src_facular(
                    coef_nc.get_variable<TF>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<TF,1> solar_src_sunspot(
                    coef_nc.get_variable<TF>("solar_source_sunspot", {n_gpts}), {n_gpts});

            TF tsi = coef_nc.get_variable<TF>("tsi_default");
            TF mg_index = coef_nc.get_variable<TF>("mg_default");
            TF sb_index = coef_nc.get_variable<TF>("sb_default");

            return Gas_optics_rrtmgp_gpu<TF>(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }

    template<typename TF>
    Cloud_optics_gpu<TF> load_and_init_cloud_optics(
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<TF,2> band_lims_wvn(coef_nc.get_variable<TF>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        TF radliq_lwr = coef_nc.get_variable<TF>("radliq_lwr");
        TF radliq_upr = coef_nc.get_variable<TF>("radliq_upr");
        TF radliq_fac = coef_nc.get_variable<TF>("radliq_fac");

        TF radice_lwr = coef_nc.get_variable<TF>("radice_lwr");
        TF radice_upr = coef_nc.get_variable<TF>("radice_upr");
        TF radice_fac = coef_nc.get_variable<TF>("radice_fac");

        Array<TF,2> lut_extliq(
                coef_nc.get_variable<TF>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<TF,2> lut_ssaliq(
                coef_nc.get_variable<TF>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<TF,2> lut_asyliq(
                coef_nc.get_variable<TF>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<TF,3> lut_extice(
                coef_nc.get_variable<TF>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<TF,3> lut_ssaice(
                coef_nc.get_variable<TF>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<TF,3> lut_asyice(
                coef_nc.get_variable<TF>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_gpu<TF>(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }
}

template<typename TF>
Radiation_solver_longwave<TF>::Radiation_solver_longwave(
        const Gas_concs_gpu<TF>& gas_concs,
        const std::string& file_name_gas,
        const std::string& file_name_cloud)
{
    // Construct the gas optics classes for the solver.
    this->kdist_gpu = std::make_unique<Gas_optics_rrtmgp_gpu<TF>>(
            load_and_init_gas_optics<TF>(gas_concs, file_name_gas));

    this->cloud_optics_gpu = std::make_unique<Cloud_optics_gpu<TF>>(
            load_and_init_cloud_optics<TF>(file_name_cloud));
}

template<typename TF>
void Radiation_solver_longwave<TF>::solve_gpu(
        const bool switch_fluxes,
        const bool switch_cloud_optics,
        const bool switch_output_optical,
        const bool switch_output_bnd_fluxes,
        const Gas_concs_gpu<TF>& gas_concs,
        const Array_gpu<TF,2>& p_lay, const Array_gpu<TF,2>& p_lev,
        const Array_gpu<TF,2>& t_lay, const Array_gpu<TF,2>& t_lev,
        Array_gpu<TF,2>& col_dry,
        const Array_gpu<TF,1>& t_sfc, const Array_gpu<TF,2>& emis_sfc,
        const Array_gpu<TF,2>& lwp, const Array_gpu<TF,2>& iwp,
        const Array_gpu<TF,2>& rel, const Array_gpu<TF,2>& rei,
        Array_gpu<TF,3>& tau, Array_gpu<TF,3>& lay_source,
        Array_gpu<TF,3>& lev_source_inc, Array_gpu<TF,3>& lev_source_dec, Array_gpu<TF,2>& sfc_source,
        Array_gpu<TF,2>& lw_flux_up, Array_gpu<TF,2>& lw_flux_dn, Array_gpu<TF,2>& lw_flux_net,
        Array_gpu<TF,3>& lw_bnd_flux_up, Array_gpu<TF,3>& lw_bnd_flux_dn, Array_gpu<TF,3>& lw_bnd_flux_net)
{
    const int n_col = p_lay.dim(1);
    const int n_lay = p_lay.dim(2);
    const int n_lev = p_lev.dim(2);
    const int n_gpt = this->kdist_gpu->get_ngpt();
    const int n_bnd = this->kdist_gpu->get_nband();

    const BOOL_TYPE top_at_1 = p_lay({1, 1}) < p_lay({1, n_lay});

    optical_props = std::make_unique<Optical_props_1scl_gpu<TF>>(n_col, n_lay, *kdist_gpu);
    sources = std::make_unique<Source_func_lw_gpu<TF>>(n_col, n_lay, *kdist_gpu);

    if (switch_cloud_optics)
        cloud_optical_props = std::make_unique<Optical_props_1scl_gpu<TF>>(n_col, n_lay, *cloud_optics_gpu);

    if (col_dry.size() == 0)
    {
        col_dry.set_dims({n_col, n_lay});
        Gas_optics_rrtmgp_gpu<TF>::get_col_dry(col_dry, gas_concs.get_vmr("h2o"), p_lev);
    }

    if (switch_fluxes)
    {
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, lw_flux_up);
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, lw_flux_dn);
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, lw_flux_net);
    }
    
    const Array<int, 2>& band_limits_gpt(this->kdist_gpu->get_band_lims_gpoint());
    for (int igpt=1; igpt<=n_gpt; ++igpt)
    {
        int band = 0;
        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
        {
            if (igpt <= band_limits_gpt({2, ibnd}))
            {
                band = ibnd;
                break;
            }
        }
        
        kdist_gpu->gas_optics(
                igpt-1,
                p_lay,
                p_lev,
                t_lay,
                t_sfc,
                gas_concs,
                optical_props,
                *sources,
                col_dry,
                t_lev);

        if (switch_cloud_optics)
        {
            cloud_optics_gpu->cloud_optics(
                    band-1,
                    lwp,
                    iwp,
                    rel,
                    rei,
                    *cloud_optical_props);
            // cloud->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_gpu<TF>&>(*optical_props),
                    dynamic_cast<Optical_props_1scl_gpu<TF>&>(*cloud_optical_props));
        }
        
        // Store the optical properties, if desired.
        if (switch_output_optical)
        {
            gpoint_kernel_launcher_cuda::get_from_gpoint(
                    n_col, n_lay, igpt-1, tau, lay_source, lev_source_inc, lev_source_dec,
                    optical_props->get_tau(), (*sources).get_lay_source(),
                    (*sources).get_lev_source_inc(), (*sources).get_lev_source_dec());

            gpoint_kernel_launcher_cuda::get_from_gpoint(
                    n_col, igpt-1, sfc_source, (*sources).get_sfc_source());
        }


        if (switch_fluxes)
        {
            constexpr int n_ang = 1;

            std::unique_ptr<Fluxes_broadband_gpu<TF>> fluxes =
                    std::make_unique<Fluxes_broadband_gpu<TF>>(n_col, 1, n_lev);

            rte_lw.rte_lw(
                    optical_props,
                    top_at_1,
                    *sources,
                    emis_sfc.subset({{ {band, band}, {1, n_col}}}),
                    Array_gpu<TF,1>(), // Add an empty array, no inc_flux.
                    (*fluxes).get_flux_up(),
                    (*fluxes).get_flux_dn(),
                    n_ang);

            (*fluxes).net_flux();
            
            // Copy the data to the output.
            gpoint_kernel_launcher_cuda::add_from_gpoint(
                    n_col, n_lev, lw_flux_up, lw_flux_dn, lw_flux_net,
                    (*fluxes).get_flux_up(), (*fluxes).get_flux_dn(), (*fluxes).get_flux_net());


            if (switch_output_bnd_fluxes)
            {
                gpoint_kernel_launcher_cuda::get_from_gpoint(
                        n_col, n_lev, igpt-1, lw_bnd_flux_up, lw_bnd_flux_dn, lw_bnd_flux_net,
                        (*fluxes).get_flux_up(), (*fluxes).get_flux_dn(), (*fluxes).get_flux_net());

            }
        }
    }
}

template<typename TF>
Radiation_solver_shortwave<TF>::Radiation_solver_shortwave(
        const Gas_concs_gpu<TF>& gas_concs,
        const std::string& file_name_gas,
        const std::string& file_name_cloud)
{
    // Construct the gas optics classes for the solver.
    this->kdist_gpu = std::make_unique<Gas_optics_rrtmgp_gpu<TF>>(
            load_and_init_gas_optics<TF>(gas_concs, file_name_gas));

    this->cloud_optics_gpu = std::make_unique<Cloud_optics_gpu<TF>>(
            load_and_init_cloud_optics<TF>(file_name_cloud));
}


template<typename TF>
void Radiation_solver_shortwave<TF>::solve_gpu(
        const bool switch_fluxes,
        const bool switch_raytracing,
        const bool switch_cloud_optics,
        const bool switch_output_optical,
        const bool switch_output_bnd_fluxes,
        const Int ray_count,
        const Gas_concs_gpu<TF>& gas_concs,
        const Array_gpu<TF,2>& p_lay, const Array_gpu<TF,2>& p_lev,
        const Array_gpu<TF,2>& t_lay, const Array_gpu<TF,2>& t_lev,
        const Array_gpu<TF,1>& grid_dims,
        Array_gpu<TF,2>& col_dry,
        const Array_gpu<TF,2>& sfc_alb_dir, const Array_gpu<TF,2>& sfc_alb_dif,
        const Array_gpu<TF,1>& tsi_scaling, const Array_gpu<TF,1>& mu0,
        const Array_gpu<TF,2>& lwp, const Array_gpu<TF,2>& iwp,
        const Array_gpu<TF,2>& rel, const Array_gpu<TF,2>& rei,
        Array_gpu<TF,3>& tau, Array_gpu<TF,3>& ssa, Array_gpu<TF,3>& g,
        Array_gpu<TF,2>& toa_source,
        Array_gpu<TF,2>& sw_flux_up, Array_gpu<TF,2>& sw_flux_dn,
        Array_gpu<TF,2>& sw_flux_dn_dir, Array_gpu<TF,2>& sw_flux_net,
        Array_gpu<TF,3>& sw_bnd_flux_up, Array_gpu<TF,3>& sw_bnd_flux_dn,
        Array_gpu<TF,3>& sw_bnd_flux_dn_dir, Array_gpu<TF,3>& sw_bnd_flux_net,
        Array_gpu<TF,2>& rt_flux_toa_up,
        Array_gpu<TF,2>& rt_flux_sfc_dir,
        Array_gpu<TF,2>& rt_flux_sfc_dif,
        Array_gpu<TF,2>& rt_flux_sfc_up,
        Array_gpu<TF,3>& rt_flux_abs_dir,
        Array_gpu<TF,3>& rt_flux_abs_dif)

{
    const int n_col = p_lay.dim(1);
    const int n_lay = p_lay.dim(2);
    const int n_lev = p_lev.dim(2);
    const int n_gpt = this->kdist_gpu->get_ngpt();
    const int n_bnd = this->kdist_gpu->get_nband();
    
    const int n_col_x = (switch_raytracing) ? rt_flux_sfc_dir.dim(1) : n_col;
    const int n_col_y = (switch_raytracing) ? rt_flux_sfc_dir.dim(2) : 1;
    const int dx_grid = (switch_raytracing) ? grid_dims({1}) : 0;
    const int dy_grid = (switch_raytracing) ? grid_dims({2}) : 0;
    const int dz_grid = (switch_raytracing) ? grid_dims({3}) : 0;
    const int n_z     = (switch_raytracing) ? grid_dims({4}) : 0;
    
    const BOOL_TYPE top_at_1 = p_lay({1, 1}) < p_lay({1, n_lay});

    optical_props = std::make_unique<Optical_props_2str_gpu<TF>>(n_col, n_lay, *kdist_gpu);
    cloud_optical_props = std::make_unique<Optical_props_2str_gpu<TF>>(n_col, n_lay, *cloud_optics_gpu);
    
    if (col_dry.size() == 0)
    {
        col_dry.set_dims({n_col, n_lay});
        Gas_optics_rrtmgp_gpu<TF>::get_col_dry(col_dry, gas_concs.get_vmr("h2o"), p_lev);
    }

    Array_gpu<TF,1> toa_src({n_col});
        
    Array<int,2> cld_mask_liq({n_col, n_lay});
    Array<int,2> cld_mask_ice({n_col, n_lay});
    
    if (switch_fluxes)
    {
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, sw_flux_up);
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, sw_flux_dn);
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, sw_flux_dn_dir);
        rrtmgp_kernel_launcher_cuda::zero_array(n_lev, n_col, sw_flux_net);
        if (switch_raytracing)
        {
            rrtmgp_kernel_launcher_cuda::zero_array(n_col_y, n_col_x, rt_flux_toa_up);
            rrtmgp_kernel_launcher_cuda::zero_array(n_col_y, n_col_x, rt_flux_sfc_dir);
            rrtmgp_kernel_launcher_cuda::zero_array(n_col_y, n_col_x, rt_flux_sfc_dif);
            rrtmgp_kernel_launcher_cuda::zero_array(n_col_y, n_col_x, rt_flux_sfc_up);
            rrtmgp_kernel_launcher_cuda::zero_array(n_z, n_col_y, n_col_x, rt_flux_abs_dir);
            rrtmgp_kernel_launcher_cuda::zero_array(n_z, n_col_y, n_col_x, rt_flux_abs_dif);
        }
    }

    const Array<int, 2>& band_limits_gpt(this->kdist_gpu->get_band_lims_gpoint());
    for (int igpt=1; igpt<=n_gpt; ++igpt)
    {
        int band = 0;
        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
        {
            if (igpt <= band_limits_gpt({2, ibnd}))
            {
                band = ibnd;
                break;
            }
        }
        
        kdist_gpu->gas_optics(
                  igpt-1,
                  p_lay,
                  p_lev,
                  t_lay,
                  gas_concs,
                  optical_props,
                  toa_src,
                  col_dry);
        scaling_to_subset(n_col, n_gpt, toa_src, tsi_scaling);
        
        if (switch_cloud_optics)
        {
            cloud_optics_gpu->cloud_optics(
                    band-1,
                    lwp,
                    iwp,
                    rel,
                    rei,
                    *cloud_optical_props);

 
            cloud_optical_props->delta_scale();
        
            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_gpu<TF>&>(*optical_props),
                    dynamic_cast<Optical_props_2str_gpu<TF>&>(*cloud_optical_props));
        }
        
        // Store the optical properties, if desired
        if (switch_output_optical)
        {
            gpoint_kernel_launcher_cuda::get_from_gpoint(
                    n_col, n_lay, igpt-1, tau, ssa, g, optical_props->get_tau(),
                     optical_props->get_ssa(),  optical_props->get_g());

            gpoint_kernel_launcher_cuda::get_from_gpoint(
                    n_col, igpt-1, toa_source, toa_src);
        }
        if (switch_fluxes)
        {  
            std::unique_ptr<Fluxes_broadband_gpu<TF>> fluxes =
                    std::make_unique<Fluxes_broadband_gpu<TF>>(n_col_x, n_col_y, n_lev);
            
            rte_sw.rte_sw(
                    optical_props,
                    top_at_1,
                    mu0,
                    toa_src,
                    sfc_alb_dir.subset({{ {band, band}, {1, n_col}}}),
                    sfc_alb_dif.subset({{ {band, band}, {1, n_col}}}),
                    Array_gpu<TF,1>(), // Add an empty array, no inc_flux.
                    (*fluxes).get_flux_up(),
                    (*fluxes).get_flux_dn(),
                    (*fluxes).get_flux_dn_dir());

            if (switch_raytracing)
            {
                if (!switch_cloud_optics) rrtmgp_kernel_launcher_cuda::zero_array(n_col, n_lay, cloud_optical_props->get_tau());

                TF zenith_angle = std::acos(mu0({1}));
                TF azimuth_angle = 3.14; // sun approximately from south
                
                raytracer.trace_rays(
                        ray_count,
                        n_col_x, n_col_y, n_z,
                        dx_grid, dy_grid, dz_grid,
                        dynamic_cast<Optical_props_2str_gpu<TF>&>(*optical_props),
                        dynamic_cast<Optical_props_2str_gpu<TF>&>(*cloud_optical_props),
                        sfc_alb_dir({band,1}), zenith_angle, 
                        azimuth_angle,
                        (*fluxes).get_flux_dn_dir()({1, n_z}),
                        (*fluxes).get_flux_dn()({1, n_z}) - (*fluxes).get_flux_dn_dir()({1, n_z}),
                        (*fluxes).get_flux_toa_up(),
                        (*fluxes).get_flux_sfc_dir(),
                        (*fluxes).get_flux_sfc_dif(),
                        (*fluxes).get_flux_sfc_up(),
                        (*fluxes).get_flux_abs_dir(),
                        (*fluxes).get_flux_abs_dif());
            }

            (*fluxes).net_flux();

            gpoint_kernel_launcher_cuda::add_from_gpoint(
                    n_col, n_lev, sw_flux_up, sw_flux_dn, sw_flux_dn_dir, sw_flux_net,
                    (*fluxes).get_flux_up(), (*fluxes).get_flux_dn(), (*fluxes).get_flux_dn_dir(), (*fluxes).get_flux_net());
            
            if (switch_raytracing)
            {
                gpoint_kernel_launcher_cuda::add_from_gpoint(
                        n_col_x, n_col_y, rt_flux_toa_up, rt_flux_sfc_dir, rt_flux_sfc_dif, rt_flux_sfc_up,
                        (*fluxes).get_flux_toa_up(), (*fluxes).get_flux_sfc_dir(), (*fluxes).get_flux_sfc_dif(), (*fluxes).get_flux_sfc_up());

                gpoint_kernel_launcher_cuda::add_from_gpoint(
                        n_col, n_z, rt_flux_abs_dir, rt_flux_abs_dif,
                        (*fluxes).get_flux_abs_dir(), (*fluxes).get_flux_abs_dif());
            }

            if (switch_output_bnd_fluxes)
            {
                gpoint_kernel_launcher_cuda::get_from_gpoint(
                        n_col, n_lev, igpt-1, sw_bnd_flux_up, sw_bnd_flux_dn, sw_bnd_flux_dn_dir, sw_bnd_flux_net,
                        (*fluxes).get_flux_up(), (*fluxes).get_flux_dn(), (*fluxes).get_flux_dn_dir(), (*fluxes).get_flux_net());
            }
        }
    }
}

#ifdef RTE_RRTMGP_SINGLE_PRECISION
template class Radiation_solver_longwave<float>;
template class Radiation_solver_shortwave<float>;
#else
template class Radiation_solver_longwave<double>;
template class Radiation_solver_shortwave<double>;
#endif
