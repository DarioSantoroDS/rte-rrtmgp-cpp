#include "hip/hip_runtime.h"
/*
 * This file is a stand-alone executable developed for the
 * testing of the C++ interface to the RTE+RRTMGP radiation code.
 *
 * It is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This software is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this software.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/algorithm/string.hpp>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime_api.h>


#include "Status.h"
#include "Netcdf_interface.h"
#include "Array.h"
#include "raytracer_kernels.h"
#include "Radiation_solver_rt.h"
#include "Gas_concs.h"
#include "Types.h"
#include "Mem_pool_gpu.h"


bool parse_command_line_options(
        std::map<std::string, std::pair<bool, std::string>>& command_line_options,
        Int& ray_count_exponent,
        int argc, char** argv)
{
    for (int i=1; i<argc; ++i)
    {
        std::string argument(argv[i]);
        boost::trim(argument);

        if (argument == "-h" || argument == "--help")
        {
            Status::print_message("Possible usage:");
            for (const auto& clo : command_line_options)
            {
                std::ostringstream ss;
                ss << std::left << std::setw(30) << ("--" + clo.first);
                ss << clo.second.second << std::endl;
                Status::print_message(ss);
            }
            return true;
        }

        //check if option is integer n (2**n rays)
        if (std::isdigit(argument[0]))
        {
            if (argument.size() > 1)
            {
                for (int i=1; i<argument.size(); ++i)
                {
                    if (!std::isdigit(argument[i]))
                    {
                        std::string error = argument + " is an illegal command line option.";
                        throw std::runtime_error(error);
                    }

                }
            }
            ray_count_exponent = Int(std::stoi(argv[i]));
        }
        else
        {
            // Check if option starts with --
            if (argument[0] != '-' || argument[1] != '-')
            {
                std::string error = argument + " is an illegal command line option.";
                throw std::runtime_error(error);
            }
            else
                argument.erase(0, 2);

            // Check if option has prefix no-
            bool enable = true;
            if (argument[0] == 'n' && argument[1] == 'o' && argument[2] == '-')
            {
                enable = false;
                argument.erase(0, 3);
            }

            if (command_line_options.find(argument) == command_line_options.end())
            {
                std::string error = argument + " is an illegal command line option.";
                throw std::runtime_error(error);
            }
            else
                command_line_options.at(argument).first = enable;
        }
    }

    return false;
}


void print_command_line_options(
        const std::map<std::string, std::pair<bool, std::string>>& command_line_options)
{
    Status::print_message("Solver settings:");
    for (const auto& option : command_line_options)
    {
        std::ostringstream ss;
        ss << std::left << std::setw(20) << (option.first);
        ss << " = " << std::boolalpha << option.second.first << std::endl;
        Status::print_message(ss);
    }
}


void solve_radiation(int argc, char** argv)
{
    Status::print_message("###### Starting RTE+RRTMGP solver ######");

    ////// FLOW CONTROL SWITCHES //////
    // Parse the command line options.
    std::map<std::string, std::pair<bool, std::string>> command_line_options {
        {"shortwave"        , { true,  "Enable computation of shortwave radiation."}},
        {"longwave"         , { true,  "Enable computation of longwave radiation." }},
        {"fluxes"           , { true,  "Enable computation of fluxes."             }},
        {"raytracing"       , { false, "Use raytracing for flux computation."      }},
        {"cloud-optics"     , { false, "Enable cloud optics."                      }},
        {"output-optical"   , { false, "Enable output of optical properties."      }},
        {"output-bnd-fluxes", { false, "Enable output of band fluxes."             }} };

    Int ray_count_exponent = 22;

    if (parse_command_line_options(command_line_options, ray_count_exponent, argc, argv))
        return;
    

    const bool switch_shortwave         = command_line_options.at("shortwave"        ).first;
    const bool switch_longwave          = command_line_options.at("longwave"         ).first;
    const bool switch_fluxes            = command_line_options.at("fluxes"           ).first;
    const bool switch_raytracing        = command_line_options.at("raytracing"       ).first;
    const bool switch_cloud_optics      = command_line_options.at("cloud-optics"     ).first;
    const bool switch_output_optical    = command_line_options.at("output-optical"   ).first;
    const bool switch_output_bnd_fluxes = command_line_options.at("output-bnd-fluxes").first;
    
    // Print the options to the screen.
    print_command_line_options(command_line_options);

    Int ray_count;
    if (switch_raytracing)
    {
        ray_count = pow(2,ray_count_exponent);
        if (ray_count < block_size*grid_size)
        {
            std::string error = "Cannot shoot " + std::to_string(ray_count) + " rays with current block/grid sizes.";
            throw std::runtime_error(error);
        }
        else
            Status::print_message("Using "+ std::to_string(Int(pow(2, ray_count_exponent))) + " rays");
    }


    ////// READ THE ATMOSPHERIC DATA //////
    Status::print_message("Reading atmospheric input data from NetCDF.");

    Netcdf_file input_nc("rte_rrtmgp_input.nc", Netcdf_mode::Read);

    const int nx = input_nc.get_dimension_size("x");
    const int ny = input_nc.get_dimension_size("y");
    const int nz = input_nc.get_dimension_size("z");


    ////// CREATE THE OUTPUT FILE //////
    // Create the general dimensions and arrays.
    Status::print_message("Preparing NetCDF output file.");

    Netcdf_file output_nc("rte_rrtmgp_output.nc", Netcdf_mode::Create);
    output_nc.add_dimension("x", nx);
    output_nc.add_dimension("y", ny);
    output_nc.add_dimension("z", nz);


    ////// RUN THE SHORTWAVE SOLVER //////
    if (switch_shortwave)
    {
        // Initialize the solver.
        Status::print_message("Initializing the shortwave solver.");

        // Solve the radiation.
        Status::print_message("Solving the shortwave radiation.");

        auto run_solver = [&]()
        {
            hipDeviceSynchronize();
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);
            // do something.

            /*
            raytracer.trace_rays(
                    ray_count,
                    nx, ny, nz,
                    dx_grid, dy_grid, dz_grid,
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props),
                    sfc_alb_dir, zenith_angle, 
                    azimuth_angle,
                    tod_dir_diff({1}),
                    tod_dir_diff({2}),
                    (*fluxes).get_flux_tod_up(),
                    (*fluxes).get_flux_sfc_dir(),
                    (*fluxes).get_flux_sfc_dif(),
                    (*fluxes).get_flux_sfc_up(),
                    (*fluxes).get_flux_abs_dir(),
                    (*fluxes).get_flux_abs_dif());
                    */

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            Status::print_message("Duration shortwave solver: " + std::to_string(duration) + " (ms)");
        };

        // Tuning step;
        run_solver();

        // Profiling step;
        hipProfilerStart();
        run_solver();
        hipProfilerStop();

        // Store the output.
        Status::print_message("Storing the shortwave output.");
    }

    Status::print_message("###### Finished RTE+RRTMGP solver ######");
}


int main(int argc, char** argv)
{
    try
    {
        solve_radiation(argc, argv);
    }

    // Catch any exceptions and return 1.
    catch (const std::exception& e)
    {
        std::string error = "EXCEPTION: " + std::string(e.what());
        Status::print_message(error);
        return 1;
    }
    catch (...)
    {
        Status::print_message("UNHANDLED EXCEPTION!");
        return 1;
    }

    // Return 0 in case of normal exit.
    return 0;
}
