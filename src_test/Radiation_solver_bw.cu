#include "hip/hip_runtime.h"
/*
 * This file is imported from MicroHH (https://github.com/earth-system-radiation/earth-system-radiation)
 * and is adapted for the testing of the C++ interface to the
 * RTE+RRTMGP radiation code.
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/algorithm/string.hpp>
#include <cmath>
#include <numeric>
#include <hiprand/hiprand_kernel.h>

#include "Radiation_solver_bw.h"
#include "Status.h"
#include "Netcdf_interface.h"

#include "Array.h"
#include "Gas_concs_rt.h"
#include "Gas_optics_rrtmgp_rt.h"
#include "Optical_props_rt.h"
#include "Source_functions_rt.h"
#include "Fluxes_rt.h"
#include "Rte_lw_rt.h"
#include "Rte_sw_rt.h"
#include "rrtmgp_kernel_launcher_cuda_rt.h"
#include "gpt_combine_kernel_launcher_cuda_rt.h"


namespace
{
    __global__
    void move_optprop_kernel(
        const int ncol, const int nlay, const Float* __restrict__ tau_in, const Float* __restrict__ ssa_in, 
        Float* __restrict__ tau_out, Float* __restrict__ ssa_out)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        if ((icol<ncol) && (ilay<nlay))
        {
            const int idx = icol + ilay*ncol;
            tau_out[idx] = tau_in[idx];
            ssa_out[idx] = ssa_in[idx];
        }

    }


    __global__
    void scaling_to_subset_kernel(
            const int ncol, const int ngpt, Float* __restrict__ toa_src, const Float tsi_scaling)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        if ( ( icol < ncol)  )
        {
            const int idx = icol;
            toa_src[idx] *= tsi_scaling;
        }
    }

    
    void scaling_to_subset(
            const int ncol, const int ngpt, Array_gpu<Float,1>& toa_src, const Float tsi_scaling)
    {
        const int block_col = 16;
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        
        dim3 grid_gpu(grid_col, 1);
        dim3 block_gpu(block_col, 1);
        scaling_to_subset_kernel<<<grid_gpu, block_gpu>>>(
            ncol, ngpt, toa_src.ptr(), tsi_scaling);
    }
    
    __global__
    void scaling_to_subset_kernel(
            const int ncol, const int ngpt, Float* __restrict__ toa_src, const Float* __restrict__ tsi_scaling)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        if ( ( icol < ncol)  )
        {
            const int idx = icol;
            toa_src[idx] *= tsi_scaling[icol];
        }
    }

    
    void scaling_to_subset(
            const int ncol, const int ngpt, Array_gpu<Float,1>& toa_src, const Array_gpu<Float,1>& tsi_scaling)
    {
        const int block_col = 16;
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        
        dim3 grid_gpu(grid_col, 1);
        dim3 block_gpu(block_col, 1);
        scaling_to_subset_kernel<<<grid_gpu, block_gpu>>>(
            ncol, ngpt, toa_src.ptr(), tsi_scaling.ptr());
    }


    std::vector<std::string> get_variable_string(
            const std::string& var_name,
            std::vector<int> i_count,
            Netcdf_handle& input_nc,
            const int string_len,
            bool trim=true)
    {
        // Multiply all elements in i_count.
        int total_count = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Add the string length as the rightmost dimension.
        i_count.push_back(string_len);

        // Read the entire char array;
        std::vector<char> var_char;
        var_char = input_nc.get_variable<char>(var_name, i_count);

        std::vector<std::string> var;

        for (int n=0; n<total_count; ++n)
        {
            std::string s(var_char.begin()+n*string_len, var_char.begin()+(n+1)*string_len);
            if (trim)
                boost::trim(s);
            var.push_back(s);
        }

        return var;
    }

    
    Gas_optics_rrtmgp_rt load_and_init_gas_optics(
            const Gas_concs_rt& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        const int n_temps = coef_nc.get_dimension_size("temperature");
        const int n_press = coef_nc.get_dimension_size("pressure");
        const int n_absorbers = coef_nc.get_dimension_size("absorber");
        const int n_char = coef_nc.get_dimension_size("string_len");
        const int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        const int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        const int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        const int n_layers = coef_nc.get_dimension_size("atmos_layer");
        const int n_bnds = coef_nc.get_dimension_size("bnd");
        const int n_gpts = coef_nc.get_dimension_size("gpt");
        const int n_pairs = coef_nc.get_dimension_size("pair");
        const int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        const int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        const int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        const int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<Float,2> band_lims(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<Float,1> press_ref(coef_nc.get_variable<Float>("press_ref", {n_press}), {n_press});
        Array<Float,1> temp_ref(coef_nc.get_variable<Float>("temp_ref", {n_temps}), {n_temps});

        Float temp_ref_p = coef_nc.get_variable<Float>("absorption_coefficient_ref_P");
        Float temp_ref_t = coef_nc.get_variable<Float>("absorption_coefficient_ref_T");
        Float press_ref_trop = coef_nc.get_variable<Float>("press_ref_trop");

        Array<Float,3> kminor_lower(
                coef_nc.get_variable<Float>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<Float,3> kminor_upper(
                coef_nc.get_variable<Float>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<Bool,1> minor_scales_with_density_lower(
                coef_nc.get_variable<Bool>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> minor_scales_with_density_upper(
                coef_nc.get_variable<Bool>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Bool,1> scale_by_complement_lower(
                coef_nc.get_variable<Bool>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> scale_by_complement_upper(
                coef_nc.get_variable<Bool>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Float,3> vmr_ref(
                coef_nc.get_variable<Float>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<Float,4> kmajor(
                coef_nc.get_variable<Float>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<Float,3> rayl_lower;
        Array<Float,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<Float>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<Float>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<Float,2> totplnk(
                    coef_nc.get_variable<Float>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<Float,4> planck_frac(
                    coef_nc.get_variable<Float>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_rt(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<Float,1> solar_src_quiet(
                    coef_nc.get_variable<Float>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_facular(
                    coef_nc.get_variable<Float>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_sunspot(
                    coef_nc.get_variable<Float>("solar_source_sunspot", {n_gpts}), {n_gpts});

            Float tsi = coef_nc.get_variable<Float>("tsi_default");
            Float mg_index = coef_nc.get_variable<Float>("mg_default");
            Float sb_index = coef_nc.get_variable<Float>("sb_default");

            return Gas_optics_rrtmgp_rt(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }

    
    Cloud_optics_rt load_and_init_cloud_optics(
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<Float,2> band_lims_wvn(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        Float radliq_lwr = coef_nc.get_variable<Float>("radliq_lwr");
        Float radliq_upr = coef_nc.get_variable<Float>("radliq_upr");
        Float radliq_fac = coef_nc.get_variable<Float>("radliq_fac");

        Float radice_lwr = coef_nc.get_variable<Float>("radice_lwr");
        Float radice_upr = coef_nc.get_variable<Float>("radice_upr");
        Float radice_fac = coef_nc.get_variable<Float>("radice_fac");

        Array<Float,2> lut_extliq(
                coef_nc.get_variable<Float>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_ssaliq(
                coef_nc.get_variable<Float>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_asyliq(
                coef_nc.get_variable<Float>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<Float,3> lut_extice(
                coef_nc.get_variable<Float>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_ssaice(
                coef_nc.get_variable<Float>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_asyice(
                coef_nc.get_variable<Float>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_rt(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }
}


Radiation_solver_longwave::Radiation_solver_longwave(
        const Gas_concs_rt& gas_concs,
        const std::string& file_name_gas,
        const std::string& file_name_cloud)
{
    // Construct the gas optics classes for the solver.
    this->kdist_gpu = std::make_unique<Gas_optics_rrtmgp_rt>(
            load_and_init_gas_optics(gas_concs, file_name_gas));

    this->cloud_optics_gpu = std::make_unique<Cloud_optics_rt>(
            load_and_init_cloud_optics(file_name_cloud));
}


void Radiation_solver_longwave::solve_gpu(
        const bool switch_fluxes,
        const bool switch_cloud_optics,
        const bool switch_output_optical,
        const bool switch_output_bnd_fluxes,
        const Gas_concs_rt& gas_concs,
        const Array_gpu<Float,2>& p_lay, const Array_gpu<Float,2>& p_lev,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev,
        Array_gpu<Float,2>& col_dry,
        const Array_gpu<Float,1>& t_sfc, const Array_gpu<Float,2>& emis_sfc,
        const Array_gpu<Float,2>& lwp, const Array_gpu<Float,2>& iwp,
        const Array_gpu<Float,2>& rel, const Array_gpu<Float,2>& rei,
        Array_gpu<Float,3>& tau, Array_gpu<Float,3>& lay_source,
        Array_gpu<Float,3>& lev_source_inc, Array_gpu<Float,3>& lev_source_dec, Array_gpu<Float,2>& sfc_source,
        Array_gpu<Float,2>& lw_flux_up, Array_gpu<Float,2>& lw_flux_dn, Array_gpu<Float,2>& lw_flux_net,
        Array_gpu<Float,3>& lw_bnd_flux_up, Array_gpu<Float,3>& lw_bnd_flux_dn, Array_gpu<Float,3>& lw_bnd_flux_net)
{
    const int n_col = p_lay.dim(1);
    const int n_lay = p_lay.dim(2);
    const int n_lev = p_lev.dim(2);
    const int n_gpt = this->kdist_gpu->get_ngpt();
    const int n_bnd = this->kdist_gpu->get_nband();

    const Bool top_at_1 = p_lay({1, 1}) < p_lay({1, n_lay});

    optical_props = std::make_unique<Optical_props_1scl_rt>(n_col, n_lay, *kdist_gpu);
    sources = std::make_unique<Source_func_lw_rt>(n_col, n_lay, *kdist_gpu);

    if (switch_cloud_optics)
        cloud_optical_props = std::make_unique<Optical_props_1scl_rt>(n_col, n_lay, *cloud_optics_gpu);

    if (col_dry.size() == 0)
    {
        col_dry.set_dims({n_col, n_lay});
        Gas_optics_rrtmgp_rt::get_col_dry(col_dry, gas_concs.get_vmr("h2o"), p_lev);
    }

    if (switch_fluxes)
    {
        rrtmgp_kernel_launcher_cuda_rt::zero_array(n_lev, n_col, lw_flux_up.ptr());
        rrtmgp_kernel_launcher_cuda_rt::zero_array(n_lev, n_col, lw_flux_dn.ptr());
        rrtmgp_kernel_launcher_cuda_rt::zero_array(n_lev, n_col, lw_flux_net.ptr());
    }
    
    const Array<int, 2>& band_limits_gpt(this->kdist_gpu->get_band_lims_gpoint());
    for (int igpt=1; igpt<=n_gpt; ++igpt)
    {
        int band = 0;
        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
        {
            if (igpt <= band_limits_gpt({2, ibnd}))
            {
                band = ibnd;
                break;
            }
        }
        
        kdist_gpu->gas_optics(
                igpt-1,
                p_lay,
                p_lev,
                t_lay,
                t_sfc,
                gas_concs,
                optical_props,
                *sources,
                col_dry,
                t_lev);

        if (switch_cloud_optics)
        {
            cloud_optics_gpu->cloud_optics(
                    band-1,
                    lwp,
                    iwp,
                    rel,
                    rei,
                    *cloud_optical_props);
            // cloud->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_rt&>(*optical_props),
                    dynamic_cast<Optical_props_1scl_rt&>(*cloud_optical_props));
        }
        
        // Store the optical properties, if desired.
        if (switch_output_optical)
        {
            gpt_combine_kernel_launcher_cuda_rt::get_from_gpoint(
                    n_col, n_lay, igpt-1, tau.ptr(), lay_source.ptr(), lev_source_inc.ptr(), lev_source_dec.ptr(),
                    optical_props->get_tau().ptr(), (*sources).get_lay_source().ptr(),
                    (*sources).get_lev_source_inc().ptr(), (*sources).get_lev_source_dec().ptr());

            gpt_combine_kernel_launcher_cuda_rt::get_from_gpoint(
                    n_col, igpt-1, sfc_source.ptr(), (*sources).get_sfc_source().ptr());
        }


        if (switch_fluxes)
        {
            constexpr int n_ang = 1;

            std::unique_ptr<Fluxes_broadband_rt> fluxes =
                    std::make_unique<Fluxes_broadband_rt>(n_col, 1, n_lev);

            rte_lw.rte_lw(
                    optical_props,
                    top_at_1,
                    *sources,
                    emis_sfc.subset({{ {band, band}, {1, n_col}}}),
                    Array_gpu<Float,1>(), // Add an empty array, no inc_flux.
                    (*fluxes).get_flux_up(),
                    (*fluxes).get_flux_dn(),
                    n_ang);

            (*fluxes).net_flux();
            
            // Copy the data to the output.
            gpt_combine_kernel_launcher_cuda_rt::add_from_gpoint(
                    n_col, n_lev, lw_flux_up.ptr(), lw_flux_dn.ptr(), lw_flux_net.ptr(),
                    (*fluxes).get_flux_up().ptr(), (*fluxes).get_flux_dn().ptr(), (*fluxes).get_flux_net().ptr());


            if (switch_output_bnd_fluxes)
            {
                gpt_combine_kernel_launcher_cuda_rt::get_from_gpoint(
                        n_col, n_lev, igpt-1, lw_bnd_flux_up.ptr(), lw_bnd_flux_dn.ptr(), lw_bnd_flux_net.ptr(),
                        (*fluxes).get_flux_up().ptr(), (*fluxes).get_flux_dn().ptr(), (*fluxes).get_flux_net().ptr());

            }
        }
    }
}



Float get_x(const Float wv)
{
    const Float a = (wv - Float(442.0)) * ((wv < Float(442.0)) ? Float(0.0624) : Float(0.0374));  
    const Float b = (wv - Float(599.8)) * ((wv < Float(599.8)) ? Float(0.0264) : Float(0.0323));  
    const Float c = (wv - Float(501.1)) * ((wv < Float(501.1)) ? Float(0.0490) : Float(0.0382));  
    return Float(0.362) * std::exp(Float(-0.5)*a*a) + Float(1.056) * std::exp(Float(-0.5)*b*b) - Float(0.065) * std::exp(Float(-0.5)*c*c);
}


Float get_y(const Float wv)
{
    const Float a = (wv - Float(568.8)) * ((wv < Float(568.8)) ? Float(0.0213) : Float(0.0247));  
    const Float b = (wv - Float(530.9)) * ((wv < Float(530.9)) ? Float(0.0613) : Float(0.0322));  
    return Float(0.821) * std::exp(Float(-0.5)*a*a) + Float(.286) * std::exp(Float(-0.5)*b*b);
}

Float get_z(const Float wv)
{
    const Float a = (wv - Float(437.0)) * ((wv < Float(437.0)) ? Float(0.0845) : Float(0.0278));  
    const Float b = (wv - Float(459.0)) * ((wv < Float(459.0)) ? Float(0.0385) : Float(0.0725));  
    return Float(1.217) * std::exp(Float(-0.5)*a*a) + Float(0.681) * std::exp(Float(-0.5)*b*b);
}
    


Float Planck(Float wv)
{
    const Float h = Float(6.62607015e-34);
    const Float c = Float(299792458.);
    const Float k = Float(1.380649e-23);
    const Float nom = 2*h*c*c / (wv*wv*wv*wv*wv);
    const Float denom = exp(h*c/(wv*k*Float(5778)))-Float(1.);
    return (nom/denom);
}


Float Planck_integrator(
        const Float wv1, const Float wv2)
{
    const int n = 100;
    const Float sa = 6.771e-5;
    const Float dwv = (wv2-wv1)/Float(n);
    Float sum = 0;
    for (int i=0; i<n; ++i)
    {
        const Float wv = (wv1 + i*dwv)*1e-9;
        sum += Planck(wv) * dwv;
    }
    return sum * Float(1e-9) * sa;
}


Float rayleigh_mean(
    const Float wv1, const Float wv2)
{
    const Float n = 1.000287;
    const Float Ns = 2.546899e19;
    const Float dwv = (wv2-wv1)/100.;
    Float sigma_mean = 0;
    for (int i=0; i<100; ++i)
    {
        const Float wv = (wv1 + i*dwv);
        const Float n = 1+1e-8*(8060.77 + 2481070/(132.274-pow((wv/1e3),-2)) + 17456.3/(39.32957-pow((wv/1e3),-2)));
        const Float nom = 24*M_PI*M_PI*M_PI*pow((n*n-1),2);
        const Float denom = pow((wv/1e7),4) * Ns*Ns * pow((n*n +2), 2);
        sigma_mean += nom/denom * 1.055;
    }
    return sigma_mean / 100.;
}


Float xyz_irradiance(
        const Float wv1, const Float wv2,
        Float (*get_xyz)(Float))
{
    Float wv = wv1; //int n = 1000; 
    const Float dwv = Float(0.1);//(wv2-wv1)/Float(n);
    Float sum = 0;
    //for (int i=0; i<n; ++i)
    while (wv < wv2)
    {
        const Float wv_tmp = wv + dwv/Float(2.);// = (wv1 + i*dwv) + dwv/Float(2.);
        //const Float wv = (wv1 + i*dwv) + dwv/Float(2.);
        sum += get_xyz(wv_tmp) * Planck(wv_tmp*Float(1e-9)) * dwv;
        wv += dwv;
    }
    return sum * Float(1e-9);
}


Radiation_solver_shortwave::Radiation_solver_shortwave(
        const Gas_concs_rt& gas_concs,
        const std::string& file_name_gas,
        const std::string& file_name_cloud)
{
    // Construct the gas optics classes for the solver.
    this->kdist_gpu = std::make_unique<Gas_optics_rrtmgp_rt>(
            load_and_init_gas_optics(gas_concs, file_name_gas));

    this->cloud_optics_gpu = std::make_unique<Cloud_optics_rt>(
            load_and_init_cloud_optics(file_name_cloud));
}



void Radiation_solver_shortwave::solve_gpu(
        const bool tune_step,
        const bool switch_cloud_optics,
        const bool switch_output_bnd_fluxes,
        const Int ray_count,
        const Gas_concs_rt& gas_concs,
        const Array_gpu<Float,2>& p_lay, const Array_gpu<Float,2>& p_lev,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev,
        const Array_gpu<Float,1>& z_lev,
        const Array_gpu<Float,1>& grid_dims,
        Array_gpu<Float,2>& col_dry,
        const Array_gpu<Float,2>& sfc_alb_dir, const Array_gpu<Float,2>& sfc_alb_dif,
        const Array_gpu<Float,1>& tsi_scaling, const Array_gpu<Float,1>& mu0,
        const Array_gpu<Float,2>& lwp, const Array_gpu<Float,2>& iwp,
        const Array_gpu<Float,2>& rel, const Array_gpu<Float,2>& rei,
        const Array_gpu<Float,1>& cam_data,
        Array_gpu<Float,3>& XYZ)

{
    const int n_col = p_lay.dim(1);
    const int n_lay = p_lay.dim(2);
    const int n_lev = p_lev.dim(2);
    const int n_gpt = this->kdist_gpu->get_ngpt();
    const int n_bnd = this->kdist_gpu->get_nband();
    
    const int dx_grid = grid_dims({1});
    const int dy_grid = grid_dims({2});
    const int dz_grid = grid_dims({3});
    const int n_z     = grid_dims({4});
    const int n_col_y = grid_dims({5});
    const int n_col_x = grid_dims({6});
    
    const int cam_nx = XYZ.dim(1);
    const int cam_ny = XYZ.dim(2);
    const int cam_ns = XYZ.dim(3);
    const Bool top_at_1 = p_lay({1, 1}) < p_lay({1, n_lay});

    optical_props = std::make_unique<Optical_props_2str_rt>(n_col, n_lay, *kdist_gpu);
    cloud_optical_props = std::make_unique<Optical_props_2str_rt>(n_col, n_lay, *cloud_optics_gpu);
    
    if (col_dry.size() == 0)
    {
        col_dry.set_dims({n_col, n_lay});
        Gas_optics_rrtmgp_rt::get_col_dry(col_dry, gas_concs.get_vmr("h2o"), p_lev);
    }

    Array_gpu<Float,1> toa_src({n_col});
    Array_gpu<Float,2> flux_camera({cam_nx, cam_ny});

    Array<int,2> cld_mask_liq({n_col, n_lay});
    Array<int,2> cld_mask_ice({n_col, n_lay});
    
    rrtmgp_kernel_launcher_cuda_rt::zero_array(cam_ns, cam_nx, cam_ny, XYZ.ptr());

    const Array<int, 2>& band_limits_gpt(this->kdist_gpu->get_band_lims_gpoint());
    Float total_source = 0.;
    
    for (int igpt=1; igpt<=n_gpt; ++igpt)
    {
        int band = 0;
        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
        {
            if (igpt <= band_limits_gpt({2, ibnd}))
            {
                band = ibnd;
                break;
            }
        }
        if (!tune_step && (! (band == 10 || band == 11 || band ==12))) continue; 
        //if (band !=11) continue; 

        const Float solar_source_band = kdist_gpu->band_source(band_limits_gpt({1,band}), band_limits_gpt({2,band}));
        
        printf("-> %d %f \n", band, solar_source_band);
        kdist_gpu->gas_optics(
                  igpt-1,
                  p_lay,
                  p_lev,
                  t_lay,
                  gas_concs,
                  optical_props,
                  toa_src,
                  col_dry);
        
        // scaling_to_subset(n_col, n_gpt, toa_src, tsi_scaling);
        // scaling_to_subset(n_col, n_gpt, toa_src, local_planck/total_planck);
        
        if (switch_cloud_optics)
        {
            cloud_optics_gpu->cloud_optics(
                    band-1,
                    lwp,
                    iwp,
                    rel,
                    rei,
                    *cloud_optical_props);

 
            cloud_optical_props->delta_scale();
        
            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props));
        }
        if (tune_step) return;
        const Array<Float, 2>& band_limits_wn(this->kdist_gpu->get_band_lims_wavenumber());
        
        /* rrtmgp's bands are quite broad, we divide each spectral band in three equally broad spectral intervals 
           and run each g-point for each spectral interval, using the mean rayleigh scattering coefficient of each spectral interval
           in stead of RRTMGP's rayleigh scattering coefficients. 
           The contribution of each spectral interval to the spectral band is based on the integrated (<>) Planck source function:
           <Planck(spectral interval)> / <Planck(spectral band)>, with a sun temperature of 5778 K. This is not entirely accurate because 
           the sun is not a black body radiatior, but the approximations comes close enough.
          
           */

        // number of intervals
        const int nwv = 3;
        const Float wv1 = 1. / band_limits_wn({2,band}) * Float(1.e7);
        const Float wv2 = 1. / band_limits_wn({1,band}) * Float(1.e7);
        const Float dwv = (wv2-wv1)/Float(nwv);

        // 
        const Float total_planck = Planck_integrator(wv1,wv2);
        
        for (int iwv=0; iwv<nwv; ++iwv)
        {
            const Float wv1_sub = wv1 + iwv*dwv;
            const Float wv2_sub = wv1 + (iwv+1)*dwv;
            const Float local_planck = Planck_integrator(wv1_sub,wv2_sub);   
            const Float rayleigh = rayleigh_mean(wv1_sub, wv2_sub);   
            const Float toa_factor = local_planck / total_planck * Float(1.)/solar_source_band; 
            
            std::unique_ptr<Fluxes_broadband_rt> fluxes =
                    std::make_unique<Fluxes_broadband_rt>(cam_nx, cam_ny, 1);

            // XYZ factors
            Array<Float,1> xyz_factor({3});
            xyz_factor({1}) = xyz_irradiance(wv1_sub,wv2_sub,&get_x);
            xyz_factor({2}) = xyz_irradiance(wv1_sub,wv2_sub,&get_y);
            xyz_factor({3}) = xyz_irradiance(wv1_sub,wv2_sub,&get_z);
            Array_gpu<Float,1> xyz_factor_gpu(xyz_factor);
            if (!switch_cloud_optics) 
            {
                rrtmgp_kernel_launcher_cuda_rt::zero_array(n_col, n_lay, cloud_optical_props->get_tau().ptr());
                rrtmgp_kernel_launcher_cuda_rt::zero_array(n_col, n_lay, cloud_optical_props->get_ssa().ptr());
            }

            Float zenith_angle = Float(0.)/Float(180.) * M_PI;//std::acos(mu0({1}));
            Float azimuth_angle = Float(0.);//M_PI;//Float(3.4906585); //3.14; // sun approximately from south
            
            
            raytracer.trace_rays(
                    ray_count,
                    n_col_x, n_col_y, n_z, n_lay,
                    dx_grid, dy_grid, dz_grid,
                    z_lev,
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props),
                    sfc_alb_dir.subset({{ {band, band}, {1, n_col}}}),
                    zenith_angle, 
                    azimuth_angle,
                    toa_src,
                    toa_factor,
                    rayleigh,
                    col_dry,
                    gas_concs.get_vmr("h2o"),
                    cam_data,
                    flux_camera); 
            
            //return; 
            raytracer.add_xyz_camera(
                    cam_nx, cam_ny,
                    xyz_factor_gpu,
                    flux_camera,
                    XYZ);
            
            //    (*fluxes).net_flux();

            //    gpt_combine_kernel_launcher_cuda_rt::add_from_gpoint(
            //            n_col, n_lev, sw_flux_up, sw_flux_dn, sw_flux_dn_dir, sw_flux_net,
            //            (*fluxes).get_flux_up(), (*fluxes).get_flux_dn(), (*fluxes).get_flux_dn_dir(), (*fluxes).get_flux_net());
                
            // gpt_combine_kernel_launcher_cuda_rt::add_from_gpoint(
            //         n_col_x, n_col_y, rt_flux_toa_up, rt_flux_sfc_dir, rt_flux_sfc_dif, rt_flux_sfc_up,
            //         (*fluxes).get_flux_toa_up(), (*fluxes).get_flux_sfc_dir(), (*fluxes).get_flux_sfc_dif(), (*fluxes).get_flux_sfc_up());

            // gpt_combine_kernel_launcher_cuda_rt::add_from_gpoint(
            //         n_col, n_z, rt_flux_abs_dir, rt_flux_abs_dif,
            //         (*fluxes).get_flux_abs_dir(), (*fluxes).get_flux_abs_dif());

            //    if (switch_output_bnd_fluxes)
            //    {
            //        gpt_combine_kernel_launcher_cuda_rt::get_from_gpoint(
            //                n_col, n_lev, igpt-1, sw_bnd_flux_up, sw_bnd_flux_dn, sw_bnd_flux_dn_dir, sw_bnd_flux_net,
            //                (*fluxes).get_flux_up(), (*fluxes).get_flux_dn(), (*fluxes).get_flux_dn_dir(), (*fluxes).get_flux_net());
            //    }
            
        }
    }
}

