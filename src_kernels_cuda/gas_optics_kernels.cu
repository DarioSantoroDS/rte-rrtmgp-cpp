#include "hip/hip_runtime.h"
#include "Types.h"

template<typename TF> __device__
TF interpolate1D(const TF val,
                 const TF offset,
                 const TF delta,
                 const int len,
                 const TF* __restrict__ table)
{
    TF val0 = (val - offset)/delta;
    TF frac = val0 - int(val0);
    int idx = min(len-1, max(1, int(val0)+1));
    return table[idx-1] + frac * (table[idx] - table[idx-1]);
}


template<typename TF> __device__
void interpolate2D_byflav_kernel(const TF* __restrict__ fminor,
                                 const TF* __restrict__ kin,
                                 const int gpt_start, const int gpt_end,
                                 TF* __restrict__ k,
                                 const int* __restrict__ jeta,
                                 const int jtemp,
                                 const int ngpt,
                                 const int neta)
{
    const int band_gpt = gpt_end-gpt_start;
    const int j0 = jeta[0];
    const int j1 = jeta[1];

    #pragma unroll
    for (int igpt=0; igpt<band_gpt; ++igpt)
    {
        k[igpt] = fminor[0] * kin[igpt + (j0-1)*ngpt + (jtemp-1)*neta*ngpt] +
                  fminor[1] * kin[igpt +  j0   *ngpt + (jtemp-1)*neta*ngpt] +
                  fminor[2] * kin[igpt + (j1-1)*ngpt + jtemp    *neta*ngpt] +
                  fminor[3] * kin[igpt +  j1   *ngpt + jtemp    *neta*ngpt];
    }
}


template<typename TF> __device__
void interpolate3D_byflav_kernel(
        const TF* __restrict__ scaling,
        const TF* __restrict__ fmajor,
        const TF* __restrict__ k,
        const int gpt_start, const int gpt_end,
        const int* __restrict__ jeta,
        const int jtemp,
        const int jpress,
        const int ngpt,
        const int neta,
        const int npress,
        TF* __restrict__ tau_major)
{
    const int band_gpt = gpt_end-gpt_start;
    const int j0 = jeta[0];
    const int j1 = jeta[1];

    #pragma unroll
    for (int igpt=0; igpt<band_gpt; ++igpt)
    {
        tau_major[igpt] = scaling[0]*
                          (fmajor[0] * k[igpt + (j0-1)*ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                           fmajor[1] * k[igpt +  j0   *ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                           fmajor[2] * k[igpt + (j0-1)*ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress] +
                           fmajor[3] * k[igpt +  j0   *ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress])
                        + scaling[1]*
                          (fmajor[4] * k[igpt + (j1-1)*ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                           fmajor[5] * k[igpt +  j1   *ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                           fmajor[6] * k[igpt + (j1-1)*ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress] +
                           fmajor[7] * k[igpt +  j1   *ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress]);
    }
}


template<typename TF> __global__
void reorder12x21_kernel(
        const int ni, const int nj,
        const TF* __restrict__ arr_in, TF* __restrict__ arr_out)
{
    const int ii = blockIdx.x*blockDim.x + threadIdx.x;
    const int ij = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (ii < ni) && (ij < nj) )
    {
        const int idx_out = ii + ij*ni;
        const int idx_in = ij + ii*nj;

        arr_out[idx_out] = arr_in[idx_in];
    }
}


template<typename TF> __global__
void reorder123x321_kernel(
        const int ni, const int nj, const int nk,
        const TF* __restrict__ arr_in, TF* __restrict__ arr_out)
{
    const int ii = blockIdx.x*blockDim.x + threadIdx.x;
    const int ij = blockIdx.y*blockDim.y + threadIdx.y;
    const int ik = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (ii < ni) && (ij < nj) && (ik < nk))
    {
        const int idx_out = ii + ij*ni + ik*nj*ni;
        const int idx_in  = ik + ij*nk + ii*nj*nk;

        arr_out[idx_out] = arr_in[idx_in];
    }
}


template<typename TF> __global__
void zero_array_kernel(
        const int ni, const int nj, const int nk,
        TF* __restrict__ arr)
{
    const int ii = blockIdx.x*blockDim.x + threadIdx.x;
    const int ij = blockIdx.y*blockDim.y + threadIdx.y;
    const int ik = blockIdx.z*blockDim.z + threadIdx.z;
    if ( (ii < ni) && (ij < nj) && (ik < nk))
    {
        const int idx = ii + ij*ni + ik*nj*ni;
        arr[idx] = TF(0.);
    }
}


template<typename TF> __global__
void Planck_source_kernel(
        const int ncol, const int nlay, const int nband, const int ngpt,
        const int nflav, const int neta, const int npres, const int ntemp,
        const int nPlanckTemp,
        const TF* __restrict__ tlay, const TF* __restrict__ tlev,
        const TF* __restrict__ tsfc,
        const int sfc_lay,
        const TF* __restrict__ fmajor, const int* __restrict__ jeta,
        const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
        const int* __restrict__ jpress, const int* __restrict__ gpoint_bands,
        const int* __restrict__ band_lims_gpt, const TF* __restrict__ pfracin,
        const TF temp_ref_min, const TF totplnk_delta,
        const TF* __restrict__ totplnk, const int* __restrict__ gpoint_flavor,
        const TF* __restrict__ ones, const TF delta_Tsurf,
        TF* __restrict__ sfc_src, TF* __restrict__ lay_src,
        TF* __restrict__ lev_src_inc, TF* __restrict__ lev_src_dec,
        TF* __restrict__ sfc_src_jac, TF* __restrict__ pfrac)
{
    const int ilay = blockIdx.x*blockDim.x + threadIdx.x;
    const int ibnd = blockIdx.y*blockDim.y + threadIdx.y;
    const int icol = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (ibnd < nband))
    {
        const int idx_collay = icol + ilay * ncol;
        const int itropo = !tropo[idx_collay];
        const int gpt_start = band_lims_gpt[2*ibnd] - 1;
        const int gpt_end = band_lims_gpt[2*ibnd + 1];
        const int iflav = gpoint_flavor[itropo + 2 * gpt_start] - 1;
        const int idx_fcl3 = 2 * 2 * 2 * (iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_fcl1 = 2 * (iflav + icol*nflav + ilay*ncol*nflav);

        const int j0 = jeta[idx_fcl1+0];
        const int j1 = jeta[idx_fcl1+1];
        const int npress1 = npres+1;

        const int jtemp_idx = jtemp[idx_collay];
        const int jpress_idx = jpress[idx_collay]+itropo;

        // compute layer source irradiances.
        const int idx_tmp = icol + ilay*ncol;
        const TF planck_function_lay = interpolate1D(tlay[idx_tmp], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);

        // compute level source irradiances.
        const int idx_tmp1 = icol + (ilay+1)*ncol;
        const int idx_tmp2 = icol + ilay*ncol;
        const TF planck_function_lev1 = interpolate1D(tlev[idx_tmp1], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);
        const TF planck_function_lev2 = interpolate1D(tlev[idx_tmp2], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);

        for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
        {
            const int iigpt = igpt-gpt_start;
            const int idx = igpt + ilay*ngpt + icol*nlay*ngpt;
            const int idx_sfc = igpt + icol*ngpt;

            const TF pfrac_loc =
                  (fmajor[idx_fcl3+0] * pfracin[gpt_start + iigpt + (j0-1)*ngpt + (jpress_idx-1)*neta*ngpt + (jtemp_idx-1)*neta*ngpt*npress1] +
                   fmajor[idx_fcl3+1] * pfracin[gpt_start + iigpt +  j0   *ngpt + (jpress_idx-1)*neta*ngpt + (jtemp_idx-1)*neta*ngpt*npress1] +
                   fmajor[idx_fcl3+2] * pfracin[gpt_start + iigpt + (j0-1)*ngpt + jpress_idx*neta*ngpt     + (jtemp_idx-1)*neta*ngpt*npress1] +
                   fmajor[idx_fcl3+3] * pfracin[gpt_start + iigpt +  j0   *ngpt + jpress_idx*neta*ngpt     + (jtemp_idx-1)*neta*ngpt*npress1])
                + 
                  (fmajor[idx_fcl3+4] * pfracin[gpt_start + iigpt + (j1-1)*ngpt + (jpress_idx-1)*neta*ngpt + jtemp_idx*neta*ngpt*npress1] +
                   fmajor[idx_fcl3+5] * pfracin[gpt_start + iigpt +  j1   *ngpt + (jpress_idx-1)*neta*ngpt + jtemp_idx*neta*ngpt*npress1] +
                   fmajor[idx_fcl3+6] * pfracin[gpt_start + iigpt + (j1-1)*ngpt + jpress_idx*neta*ngpt     + jtemp_idx*neta*ngpt*npress1] +
                   fmajor[idx_fcl3+7] * pfracin[gpt_start + iigpt +  j1   *ngpt + jpress_idx*neta*ngpt     + jtemp_idx*neta*ngpt*npress1]);

            // Layer source
            lay_src[idx] = pfrac_loc * planck_function_lay;

            // Level source
            lev_src_inc[idx] = pfrac_loc * planck_function_lev1;
            lev_src_dec[idx] = pfrac_loc * planck_function_lev2;

            // Surface
            if (ilay == sfc_lay - 1) // Subtract one to correct for fortran indexing.
            {
                const TF planck_function_sfc1 = interpolate1D(
                        tsfc[icol], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);
                const TF planck_function_sfc2 = interpolate1D(
                        tsfc[icol] + delta_Tsurf, temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);

                sfc_src[idx_sfc] = pfrac_loc * planck_function_sfc1;
                sfc_src_jac[idx_sfc] = pfrac_loc * (planck_function_sfc2 - planck_function_sfc1);
            }
        }
    }
}


template<typename TF> __global__
void interpolation_kernel(
        const int ncol, const int nlay, const int ngas, const int nflav,
        const int neta, const int npres, const int ntemp, const TF tmin,
        const int* __restrict__ flavor,
        const TF* __restrict__ press_ref_log,
        const TF* __restrict__ temp_ref,
        TF press_ref_log_delta,
        TF temp_ref_min,
        TF temp_ref_delta,
        TF press_ref_trop_log,
        const TF* __restrict__ vmr_ref,
        const TF* __restrict__ play,
        const TF* __restrict__ tlay,
        TF* __restrict__ col_gas,
        int* __restrict__ jtemp,
        TF* __restrict__ fmajor, TF* __restrict__ fminor,
        TF* __restrict__ col_mix,
        BOOL_TYPE* __restrict__ tropo,
        int* __restrict__ jeta,
        int* __restrict__ jpress)
{
    const int iflav = blockIdx.x*blockDim.x + threadIdx.x;
    const int icol  = blockIdx.y*blockDim.y + threadIdx.y;
    const int ilay  = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (iflav < nflav) )
    {
        const int idx = icol + ilay*ncol;

        jtemp[idx] = int((tlay[idx] - (temp_ref_min-temp_ref_delta)) / temp_ref_delta);
        jtemp[idx] = min(ntemp-1, max(1, jtemp[idx]));
        const TF ftemp = (tlay[idx] - temp_ref[jtemp[idx]-1]) / temp_ref_delta;

        const TF locpress = TF(1.) + (log(play[idx]) - press_ref_log[0]) / press_ref_log_delta;
        jpress[idx] = min(npres-1, max(1, int(locpress)));
        const TF fpress = locpress - TF(jpress[idx]);

        tropo[idx] = log(play[idx]) > press_ref_trop_log;
        const int itropo = !tropo[idx];

        const int gas1 = flavor[2*iflav];
        const int gas2 = flavor[2*iflav+1];

        for (int itemp=0; itemp<2; ++itemp)
        {
            const int vmr_base_idx = itropo + (jtemp[idx]+itemp-1) * (ngas+1) * 2;
            const int colmix_idx = itemp + 2*(iflav + nflav*icol + nflav*ncol*ilay);
            const int colgas1_idx = icol + ilay*ncol + gas1*nlay*ncol;
            const int colgas2_idx = icol + ilay*ncol + gas2*nlay*ncol;
            const TF ratio_eta_half = vmr_ref[vmr_base_idx + 2*gas1] /
                                      vmr_ref[vmr_base_idx + 2*gas2];
            col_mix[colmix_idx] = col_gas[colgas1_idx] + ratio_eta_half * col_gas[colgas2_idx];

            TF eta;
            if (col_mix[colmix_idx] > TF(2.)*tmin)
                eta = col_gas[colgas1_idx] / col_mix[colmix_idx];
            else
                eta = TF(0.5);

            const TF loceta = eta * TF(neta-1);
            jeta[colmix_idx] = min(int(loceta)+1, neta-1);
            const TF feta = fmod(loceta, TF(1.));
            const TF ftemp_term  = TF(1-itemp) + TF(2*itemp-1)*ftemp;

            // Compute interpolation fractions needed for minot species.
            const int fminor_idx = 2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
            fminor[fminor_idx] = (TF(1.0)-feta) * ftemp_term;
            fminor[fminor_idx+1] = feta * ftemp_term;

            // Compute interpolation fractions needed for major species.
            const int fmajor_idx = 2*2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
            fmajor[fmajor_idx] = (TF(1.0)-fpress) * fminor[fminor_idx];
            fmajor[fmajor_idx+1] = (TF(1.0)-fpress) * fminor[fminor_idx+1];
            fmajor[fmajor_idx+2] = fpress * fminor[fminor_idx];
            fmajor[fmajor_idx+3] = fpress * fminor[fminor_idx+1];
        }
    }
}


template<typename TF> __global__
void compute_tau_major_absorption_kernel(
        const int ncol, const int nlay, const int nband, const int ngpt,
        const int nflav, const int neta, const int npres, const int ntemp,
        const int* __restrict__ gpoint_flavor,
        const int* __restrict__ band_lims_gpt,
        const TF* __restrict__ kmajor,
        const TF* __restrict__ col_mix, const TF* __restrict__ fmajor,
        const int* __restrict__ jeta, const BOOL_TYPE* __restrict__ tropo,
        const int* __restrict__ jtemp, const int* __restrict__ jpress,
        TF* __restrict__ tau, TF* __restrict__ tau_major)
{
    // Fetch the three coordinates.
    const int ibnd = (blockIdx.z * blockDim.z) + threadIdx.z;
    const int ilay = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int icol = (blockIdx.x * blockDim.x) + threadIdx.x;

    if ( (icol < ncol) && (ilay < nlay) && (ibnd < nband) )
    {
        const int idx_collay = icol + ilay * ncol;
        const int itropo = !tropo[idx_collay];
        const int gpt_start = band_lims_gpt[2*ibnd] - 1;
        const int gpt_end = band_lims_gpt[2*ibnd + 1];
        const int iflav = gpoint_flavor[itropo + 2*gpt_start] - 1;
        const int idx_fcl3 = 2 * 2 * 2 * (iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_fcl1 = 2 * (iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

        //major gases//
        interpolate3D_byflav_kernel(
                &col_mix[idx_fcl1], &fmajor[idx_fcl3],
                &kmajor[gpt_start], gpt_start, gpt_end,
                &jeta[idx_fcl1], jtemp[idx_collay],
                jpress[idx_collay]+itropo, ngpt, neta, npres+1,
                &tau_major[idx_tau]);

        #pragma unroll
        for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
        {
            const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
            tau[idx_out] += tau_major[idx_out];
        }
    }
}


template<typename TF> __global__
void compute_tau_minor_absorption_kernel(
        const int ncol, const int nlay, const int ngpt,
        const int ngas, const int nflav, const int ntemp, const int neta,
        const int nscale,
        const int nminor,
        const int nminork,
        const int idx_h2o, const int idx_tropo,
        const int* __restrict__ gpoint_flavor,
        const TF* __restrict__ kminor,
        const int* __restrict__ minor_limits_gpt,
        const BOOL_TYPE* __restrict__ minor_scales_with_density,
        const BOOL_TYPE* __restrict__ scale_by_complement,
        const int* __restrict__ idx_minor,
        const int* __restrict__ idx_minor_scaling,
        const int* __restrict__ kminor_start,
        const TF* __restrict__ play,
        const TF* __restrict__ tlay,
        const TF* __restrict__ col_gas,
        const TF* __restrict__ fminor,
        const int* __restrict__ jeta,
        const int* __restrict__ jtemp,
        const BOOL_TYPE* __restrict__ tropo,
        TF* __restrict__ tau,
        TF* __restrict__ tau_minor)
{
    // Fetch the three coordinates.
    const int icol = blockIdx.x * blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y * blockDim.y + threadIdx.y;

    const TF PaTohPa = 0.01;
    const int ncl = ncol * nlay;

    if ((icol < ncol) && (ilay < nlay))
    {
        const int idx_collay = icol + ilay*ncol;
        const int idx_collaywv = icol + ilay*ncol + idx_h2o*ncl;

        if (tropo[idx_collay] == idx_tropo)
        {
            for (int imnr = 0; imnr < nscale; ++imnr)
            {
                TF scaling = col_gas[idx_collay + idx_minor[imnr] * ncl];

                if (minor_scales_with_density[imnr])
                {
                    scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];

                    if (idx_minor_scaling[imnr] > 0)
                    {
                        TF vmr_fact = TF(1.) / col_gas[idx_collay];
                        TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);

                        if (scale_by_complement[imnr])
                            scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling[imnr] * ncl] * vmr_fact * dry_fact);
                        else
                            scaling *= col_gas[idx_collay + idx_minor_scaling[imnr] * ncl] * vmr_fact * dry_fact;
                    }
                }

                const int gpt_start = minor_limits_gpt[2*imnr]-1;
                const int gpt_end = minor_limits_gpt[2*imnr+1];
                const int gpt_offs = 1-idx_tropo;
                const int iflav = gpoint_flavor[2*gpt_start + gpt_offs]-1;
                const int idx_fcl2 = 2 * 2 * (iflav + icol*nflav + ilay*ncol*nflav);
                const int idx_fcl1 = 2 * (iflav + icol*nflav + ilay*ncol*nflav);
                const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                interpolate2D_byflav_kernel(
                        &fminor[idx_fcl2],
                        &kminor[kminor_start[imnr]-1],
                        kminor_start[imnr]-1,
                        kminor_start[imnr]-1 + (gpt_end - gpt_start),
                        &tau_minor[idx_tau],
                        &jeta[idx_fcl1],
                        jtemp[idx_collay],
                        nminork, neta);

                #pragma unroll
                for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                {
                    const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
                    tau[idx_out] += tau_minor[idx_out] * scaling;
                }
            }
        }
    }
}


template<typename TF> __global__
void compute_tau_rayleigh_kernel(
        const int ncol, const int nlay, const int nbnd, const int ngpt,
        const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
        const int* __restrict__ gpoint_flavor,
        const int* __restrict__ band_lims_gpt,
        const TF* __restrict__ krayl,
        int idx_h2o, const TF* __restrict__ col_dry, const TF* __restrict__ col_gas,
        const TF* __restrict__ fminor, const int* __restrict__ jeta,
        const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
        TF* __restrict__ tau_rayleigh, TF* __restrict__ k)
{
    // Fetch the three coordinates.
    const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int icol = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (ibnd < nbnd) )
    {
        //kernel implementation
        const int idx_collay = icol + ilay*ncol;
        const int idx_collaywv = icol + ilay*ncol + idx_h2o*nlay*ncol;
        const int itropo = !tropo[idx_collay];
        const int gpt_start = band_lims_gpt[2*ibnd]-1;
        const int gpt_end = band_lims_gpt[2*ibnd+1];
        const int iflav = gpoint_flavor[itropo+2*gpt_start]-1;
        const int idx_fcl2 = 2*2*(iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_fcl1   = 2*(iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_krayl  = gpt_start + ngpt*neta*ntemp*itropo;
        const int idx_k = gpt_start + ilay*ngpt + icol*nlay*ngpt;
        interpolate2D_byflav_kernel(&fminor[idx_fcl2],
                                    &krayl[idx_krayl],
                                    gpt_start, gpt_end, &k[idx_k],
                                    &jeta[idx_fcl1],
                                    jtemp[idx_collay],
                                    ngpt, neta);

        for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
        {
            const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
            tau_rayleigh[idx_out] = k[idx_k+igpt-gpt_start]*(col_gas[idx_collaywv]+col_dry[idx_collay]);
        }
    }
}


template<typename TF> __global__
void combine_and_reorder_2str_kernel(
        const int ncol, const int nlay, const int ngpt, const TF tmin,
        const TF* __restrict__ tau_abs, const TF* __restrict__ tau_rayleigh,
        TF* __restrict__ tau, TF* __restrict__ ssa, TF* __restrict__ g)
{
    // Fetch the three coordinates.
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
    const int ilay = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx_in  = igpt + ilay*ngpt + icol*(ngpt*nlay);
        const int idx_out = icol + ilay*ncol + igpt*(ncol*nlay);

        const TF tau_tot = tau_abs[idx_in] + tau_rayleigh[idx_in];
        tau[idx_out] = tau_tot;
        g  [idx_out] = TF(0.);
        if (tau_tot>(TF(2.)*tmin))
            ssa[idx_out] = tau_rayleigh[idx_in]/tau_tot;
        else
            ssa[idx_out] = TF(0.);
    }
}

