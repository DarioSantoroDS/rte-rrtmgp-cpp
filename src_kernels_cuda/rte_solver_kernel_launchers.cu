#include <chrono>

#include "rte_kernel_launcher_cuda.h"
#include "tools_gpu.h"
#include "Array.h"
#include "tuner.h"

#include <iomanip>


namespace
{
    #include "rte_solver_kernels.cu"
}


namespace rte_kernel_launcher_cuda
{
    template<typename TF>
    void apply_BC(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
                  const Array_gpu<TF,2>& inc_flux_dir, const Array_gpu<TF,1>& mu0, Array_gpu<TF,3>& gpt_flux_dir)
    {
        const int block_col = 32;
        const int block_gpt = 32;

        const int grid_col = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt = ngpt/block_gpt + (ngpt%block_gpt > 0);

        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        apply_BC_kernel<<<grid_gpu, block_gpu>>>(ncol, nlay, ngpt, top_at_1, inc_flux_dir.ptr(), mu0.ptr(), gpt_flux_dir.ptr());

    }


    template<typename TF>
    void apply_BC(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, Array_gpu<TF,3>& gpt_flux_dn)
    {
        const int block_col = 32;
        const int block_gpt = 32;

        const int grid_col = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt = ngpt/block_gpt + (ngpt%block_gpt > 0);

        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        apply_BC_kernel<<<grid_gpu, block_gpu>>>(ncol, nlay, ngpt, top_at_1, gpt_flux_dn.ptr());
    }


    template<typename TF>
    void apply_BC(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const Array_gpu<TF,2>& inc_flux_dif, Array_gpu<TF,3>& gpt_flux_dn)
    {
        const int block_col = 32;
        const int block_gpt = 32;

        const int grid_col = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt = ngpt/block_gpt + (ngpt%block_gpt > 0);

        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        apply_BC_kernel<<<grid_gpu, block_gpu>>>(ncol, nlay, ngpt, top_at_1, inc_flux_dif.ptr(), gpt_flux_dn.ptr());
    }


    template<typename TF>
    void lw_solver_noscat_gaussquad(
            const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
            const Array_gpu<TF,2>& ds, const Array_gpu<TF,2>& weights, const Array_gpu<TF,3>& tau, const Array_gpu<TF,3> lay_source,
            const Array_gpu<TF,3>& lev_source_inc, const Array_gpu<TF,3>& lev_source_dec, const Array_gpu<TF,2>& sfc_emis,
            const Array_gpu<TF,2>& sfc_src, Array_gpu<TF,3>& flux_up, Array_gpu<TF,3>& flux_dn,
            const Array_gpu<TF,2>& sfc_src_jac, Array_gpu<TF,3>& flux_up_jac,
            Tuner_map& tunings)
    {
        TF eps = std::numeric_limits<TF>::epsilon();

        const int flx_size = flux_dn.size();
        const int opt_size = tau.size();
        const int sfc_size = sfc_src.size();

        Array_gpu<TF,2> source_sfc(sfc_src.get_dims());
        Array_gpu<TF,2> source_sfc_jac(sfc_src.get_dims());
        Array_gpu<TF,2> sfc_albedo(sfc_src.get_dims());
        Array_gpu<TF,3> tau_loc(tau.get_dims());
        Array_gpu<TF,3> trans(tau.get_dims());
        Array_gpu<TF,3> source_dn(tau.get_dims());
        Array_gpu<TF,3> source_up(tau.get_dims());
        Array_gpu<TF,3> radn_dn(flux_dn.get_dims());
        Array_gpu<TF,3> radn_up(flux_dn.get_dims());
        Array_gpu<TF,3> radn_up_jac(flux_dn.get_dims());

        // Running some permutations of block sizes.
        /*`
        {
            std::cout << "TUNING lw_solver_noscat_gaussquad_kernel" << std::endl;
            std::vector<std::pair<int, int>> col_gpt_combis;
            std::vector<int> cols{ 1, 1, 2, 4, 8, 16, 32, 64, 128, 256, 512};
            std::vector<int> gpts{ 1, 2, 4, 8, 16, 32, 64, 128};
            for (const int igpt : gpts)
                for (const int icol : cols)
                    col_gpt_combis.emplace_back(icol, igpt);

            // Create tmp arrays to write output to.
            Array_gpu<TF,3> flux_up_tmp{flux_up}, flux_dn_tmp{flux_dn}, flux_up_jac_tmp{flux_up_jac};

            for (const auto& p : col_gpt_combis)
            {
                std::cout << "(" << p.first << ", " << p.second << "): ";

                const int block_col2d = p.first;
                const int block_gpt2d = p.second;

                const int grid_col2d = ncol/block_col2d + (ncol%block_col2d > 0);
                const int grid_gpt2d = ngpt/block_gpt2d + (ngpt%block_gpt2d > 0);

                dim3 grid_gpu2d(grid_col2d, grid_gpt2d);
                dim3 block_gpu2d(block_col2d, block_gpt2d);

                // Warm it up.
                lw_solver_noscat_gaussquad_kernel<<<grid_gpu2d, block_gpu2d>>>(
                        ncol, nlay, ngpt, eps, top_at_1, nmus, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                        lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), radn_up,
                        radn_dn, sfc_src_jac.ptr(), radn_up_jac, tau_loc, trans, source_dn, source_up,
                        source_sfc, sfc_albedo, source_sfc_jac, flux_up_tmp.ptr(), flux_dn_tmp.ptr(), flux_up_jac_tmp.ptr());

                hipEvent_t start;
                hipEvent_t stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                const int n_samples = 10;

                hipEventRecord(start, 0);
                for (int i=0; i<n_samples; ++i)
                    lw_solver_noscat_gaussquad_kernel<<<grid_gpu2d, block_gpu2d>>>(
                            ncol, nlay, ngpt, eps, top_at_1, nmus, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                            lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), radn_up,
                            radn_dn, sfc_src_jac.ptr(), radn_up_jac, tau_loc, trans, source_dn, source_up,
                            source_sfc, sfc_albedo, source_sfc_jac, flux_up_tmp.ptr(), flux_dn_tmp.ptr(), flux_up_jac_tmp.ptr());
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                float duration = 0.f;
                hipEventElapsedTime(&duration, start, stop);

                std::cout << std::setprecision(10) << duration << " (ns), check: " << flux_up_tmp({ncol, nlay+1, ngpt}) << ", ";

                // Check whether kernel has succeeded;
                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                    std::cout << hipGetErrorString(err) << std::endl;
                else
                    std::cout << std::endl;
            }

            std::cout << "STOP TUNING lw_solver_noscat_gaussquad_kernel" << std::endl;
        }
        */
        // End of performance tuning.

        const int block_col2d = 64;
        const int block_gpt2d = 2;

        const int grid_col2d = ncol/block_col2d + (ncol%block_col2d > 0);
        const int grid_gpt2d = ngpt/block_gpt2d + (ngpt%block_gpt2d > 0);

        dim3 grid_gpu2d(grid_col2d, grid_gpt2d);
        dim3 block_gpu2d(block_col2d, block_gpt2d);

        const int block_col3d = 96;
        const int block_lay3d = 1;
        const int block_gpt3d = 1;

        const int grid_col3d = ncol/block_col3d + (ncol%block_col3d > 0);
        const int grid_lay3d = (nlay+1)/block_lay3d + ((nlay+1)%block_lay3d > 0);
        const int grid_gpt3d = ngpt/block_gpt3d + (ngpt%block_gpt3d > 0);

        dim3 grid_gpu3d(grid_col3d, grid_lay3d, grid_gpt3d);
        dim3 block_gpu3d(block_col3d, block_lay3d, block_gpt3d);

        const int top_level = top_at_1 ? 0 : nlay;


        // Step 1.
        dim3 grid_1, block_1;

        if (tunings.count("lw_step_1") == 0)
        {
            std::tie(grid_1, block_1) = tune_kernel(
                    "lw_step_1",
                    {ncol, nlay, ngpt}, {8, 16, 24, 32, 48, 64, 96, 128, 256, 512, 1024}, {1, 2, 4, 8}, {1},
                    lw_solver_noscat_step_1_kernel<TF>,
                    ncol, nlay, ngpt, eps, top_at_1, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                    lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), flux_up.ptr(), flux_dn.ptr(), sfc_src_jac.ptr(),
                    flux_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

            tunings["lw_step_1"].first = grid_1;
            tunings["lw_step_1"].second = block_1;
        }
        else
        {
            grid_1 = tunings["lw_step_1"].first;
            block_1 = tunings["lw_step_1"].second;
        }

        lw_solver_noscat_step_1_kernel<<<grid_1, block_1>>>(
                ncol, nlay, ngpt, eps, top_at_1, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), flux_up.ptr(), flux_dn.ptr(), sfc_src_jac.ptr(),
                flux_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());


        // Step 2.
        dim3 grid_2, block_2;

        if (tunings.count("lw_step_2") == 0)
        {
            std::tie(grid_2, block_2) = tune_kernel(
                    "lw_step_2",
                    {ncol, ngpt}, {64, 128, 256, 384, 512, 768, 1024}, {1, 2, 4}, {1},
                    lw_solver_noscat_step_2_kernel<TF>,
                    ncol, nlay, ngpt, eps, top_at_1, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                    lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), flux_up.ptr(), flux_dn.ptr(), sfc_src_jac.ptr(),
                    flux_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

            tunings["lw_step_2"].first = grid_2;
            tunings["lw_step_2"].second = block_2;
        }
        else
        {
            grid_2 = tunings["lw_step_2"].first;
            block_2 = tunings["lw_step_2"].second;
        }

        lw_solver_noscat_step_2_kernel<<<grid_2, block_2>>>(
                ncol, nlay, ngpt, eps, top_at_1, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), flux_up.ptr(), flux_dn.ptr(), sfc_src_jac.ptr(),
                flux_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());


        // Step 3.
        dim3 grid_3, block_3;

        if (tunings.count("lw_step_3") == 0)
        {
            std::tie(grid_3, block_3) = tune_kernel(
                    "lw_step_3",
                    {ncol, nlay+1, ngpt}, {8, 16, 24, 32, 48, 64, 96, 128, 256}, {1, 2, 4, 8}, {1},
                    lw_solver_noscat_step_3_kernel<TF>,
                    ncol, nlay, ngpt, eps, top_at_1, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                    lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), flux_up.ptr(), flux_dn.ptr(), sfc_src_jac.ptr(),
                    flux_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

            tunings["lw_step_3"].first = grid_3;
            tunings["lw_step_3"].second = block_3;
        }
        else
        {
            grid_3 = tunings["lw_step_3"].first;
            block_3 = tunings["lw_step_3"].second;
        }

        lw_solver_noscat_step_3_kernel<<<grid_3, block_3>>>(
                ncol, nlay, ngpt, eps, top_at_1, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), flux_up.ptr(), flux_dn.ptr(), sfc_src_jac.ptr(),
                flux_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

        apply_BC_kernel_lw<<<grid_gpu2d, block_gpu2d>>>(top_level, ncol, nlay, ngpt, top_at_1, flux_dn.ptr(), radn_dn.ptr());

        if (nmus > 1)
        {
            for (int imu=1; imu<nmus; ++imu)
            {
                lw_solver_noscat_step_1_kernel<<<grid_1, block_1>>>(
                        ncol, nlay, ngpt, eps, top_at_1, ds.ptr()+imu, weights.ptr()+imu, tau.ptr(), lay_source.ptr(),
                        lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), radn_up.ptr(), radn_dn.ptr(), sfc_src_jac.ptr(),
                        radn_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

                lw_solver_noscat_step_2_kernel<<<grid_2, block_2>>>(
                        ncol, nlay, ngpt, eps, top_at_1, ds.ptr()+imu, weights.ptr()+imu, tau.ptr(), lay_source.ptr(),
                        lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), radn_up.ptr(), radn_dn.ptr(), sfc_src_jac.ptr(),
                        radn_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

                lw_solver_noscat_step_3_kernel<<<grid_3, block_3>>>(
                        ncol, nlay, ngpt, eps, top_at_1, ds.ptr()+imu, weights.ptr()+imu, tau.ptr(), lay_source.ptr(),
                        lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), radn_up.ptr(), radn_dn.ptr(), sfc_src_jac.ptr(),
                        radn_up_jac.ptr(), tau_loc.ptr(), trans.ptr(), source_dn.ptr(), source_up.ptr(), source_sfc.ptr(), sfc_albedo.ptr(), source_sfc_jac.ptr());

                add_fluxes_kernel<<<grid_gpu3d, block_gpu3d>>>(
                        ncol, nlay+1, ngpt,
                        radn_up.ptr(), radn_dn.ptr(), radn_up_jac.ptr(),
                        flux_up.ptr(), flux_dn.ptr(), flux_up_jac.ptr());
            }
        }
    }


    template<typename TF>
    void sw_solver_2stream(
            const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
            const Array_gpu<TF,3>& tau, const Array_gpu<TF,3>& ssa, const Array_gpu<TF,3>& g,
            const Array_gpu<TF,1>& mu0,
            const Array_gpu<TF,2>& sfc_alb_dir, const Array_gpu<TF,2>& sfc_alb_dif,
            const Array_gpu<TF,2>& inc_flux_dir,
            Array_gpu<TF,3>& flux_up, Array_gpu<TF,3>& flux_dn, Array_gpu<TF,3>& flux_dir,
            const BOOL_TYPE has_dif_bc, const Array_gpu<TF,2>& inc_flux_dif,
            const BOOL_TYPE do_broadband, Array_gpu<TF,3>& flux_up_loc, Array_gpu<TF,3>& flux_dn_loc, Array_gpu<TF,3>& flux_dir_loc,
            Tuner_map& tunings)
    {
        const int opt_size = tau.size();
        const int alb_size = sfc_alb_dir.size();
        const int flx_size = flux_up.size();

        TF* r_dif = Tools_gpu::allocate_gpu<TF>(opt_size);
        TF* t_dif = Tools_gpu::allocate_gpu<TF>(opt_size);
        TF* r_dir = nullptr;
        TF* t_dir = nullptr;
        TF* t_noscat = nullptr;
        TF* source_up = Tools_gpu::allocate_gpu<TF>(opt_size);
        TF* source_dn = Tools_gpu::allocate_gpu<TF>(opt_size);
        TF* source_sfc = Tools_gpu::allocate_gpu<TF>(alb_size);
        TF* albedo = Tools_gpu::allocate_gpu<TF>(flx_size);
        TF* src = Tools_gpu::allocate_gpu<TF>(flx_size);
        TF* denom = Tools_gpu::allocate_gpu<TF>(opt_size);

        dim3 grid_source{ncol, ngpt}, block_source;


        // Step 1.
        if (tunings.count("sw_source_2stream_kernel") == 0)
        {
            if (top_at_1)
            {
                std::tie(grid_source, block_source) = tune_kernel(
                        "sw_source_2stream_kernel",
                        {ncol, ngpt}, {8, 16, 32, 64, 96, 128, 256, 384, 512, 768, 1024}, {1, 2, 4, 8, 16}, {1},
                        sw_source_2stream_kernel<TF, 1>,
                        ncol, nlay, ngpt, tau.ptr(), ssa.ptr(), g.ptr(), mu0.ptr(), r_dif, t_dif,
                        sfc_alb_dir.ptr(), source_up, source_dn, source_sfc, flux_dir.ptr());
            }
            else
            {
                std::tie(grid_source, block_source) = tune_kernel(
                        "sw_source_2stream_kernel",
                        {ncol, ngpt}, {8, 16, 32, 64, 96, 128, 256, 384, 512, 768, 1024}, {1, 2, 4, 8, 16}, {1},
                        sw_source_2stream_kernel<TF, 0>,
                        ncol, nlay, ngpt, tau.ptr(), ssa.ptr(), g.ptr(), mu0.ptr(), r_dif, t_dif,
                        sfc_alb_dir.ptr(), source_up, source_dn, source_sfc, flux_dir.ptr());
            }

            tunings["sw_source_2stream_kernel"].first = grid_source;
            tunings["sw_source_2stream_kernel"].second = block_source;
        }
        else
        {
            grid_source = tunings["sw_source_2stream_kernel"].first;
            block_source = tunings["sw_source_2stream_kernel"].second;
        }

        if (top_at_1)
        {
            sw_source_2stream_kernel<TF, 1><<<grid_source, block_source>>>(
                    ncol, nlay, ngpt, tau.ptr(), ssa.ptr(), g.ptr(), mu0.ptr(), r_dif, t_dif,
                    sfc_alb_dir.ptr(), source_up, source_dn, source_sfc, flux_dir.ptr());
        }
        else
        {
            sw_source_2stream_kernel<TF, 0><<<grid_source, block_source>>>(
                    ncol, nlay, ngpt, tau.ptr(), ssa.ptr(), g.ptr(), mu0.ptr(), r_dif, t_dif,
                    sfc_alb_dir.ptr(), source_up, source_dn, source_sfc, flux_dir.ptr());
        }


        // Step 2.
        dim3 grid_adding, block_adding;

        if (tunings.count("sw_adding") == 0)
        {
            if (top_at_1)
            {
                std::tie(grid_adding, block_adding) = tune_kernel(
                        "sw_adding",
                        {ncol, ngpt}, {8, 16, 32, 64, 96, 128, 256, 384, 512, 768, 1024}, {1, 2, 4, 8, 16}, {1},
                        sw_adding_kernel<TF, 1>,
                        ncol, nlay, ngpt, top_at_1,
                        sfc_alb_dif.ptr(), r_dif, t_dif,
                        source_dn, source_up, source_sfc,
                        flux_up.ptr(), flux_dn.ptr(), flux_dir.ptr(), albedo, src, denom);
            }
            else
            {
                std::tie(grid_adding, block_adding) = tune_kernel(
                        "sw_adding",
                        {ncol, ngpt}, {8, 16, 32, 64, 96, 128, 256, 384, 512, 768, 1024}, {1, 2, 4, 8, 16}, {1},
                        sw_adding_kernel<TF, 0>,
                        ncol, nlay, ngpt, top_at_1,
                        sfc_alb_dif.ptr(), r_dif, t_dif,
                        source_dn, source_up, source_sfc,
                        flux_up.ptr(), flux_dn.ptr(), flux_dir.ptr(), albedo, src, denom);
            }

            tunings["sw_adding"].first = grid_adding;
            tunings["sw_adding"].second = block_adding;
        }
        else
        {
            grid_adding = tunings["sw_adding"].first;
            block_adding = tunings["sw_adding"].second;
        }

        if (top_at_1)
        {
            sw_adding_kernel<TF, 1><<<grid_adding, block_adding>>>(
                ncol, nlay, ngpt, top_at_1,
                sfc_alb_dif.ptr(), r_dif, t_dif,
                source_dn, source_up, source_sfc,
                flux_up.ptr(), flux_dn.ptr(), flux_dir.ptr(), albedo, src, denom);
        }
        else
        {
            sw_adding_kernel<TF, 0><<<grid_adding, block_adding>>>(
                        ncol, nlay, ngpt, top_at_1,
                        sfc_alb_dif.ptr(), r_dif, t_dif,
                        source_dn, source_up, source_sfc,
                        flux_up.ptr(), flux_dn.ptr(), flux_dir.ptr(), albedo, src, denom);
        }

        Tools_gpu::free_gpu(r_dif);
        Tools_gpu::free_gpu(t_dif);
        Tools_gpu::free_gpu(source_up);
        Tools_gpu::free_gpu(source_dn);
        Tools_gpu::free_gpu(source_sfc);
        Tools_gpu::free_gpu(albedo);
        Tools_gpu::free_gpu(src);
        Tools_gpu::free_gpu(denom);
    }
}


#ifdef RTE_RRTMGP_SINGLE_PRECISION
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE,
                  const Array_gpu<float,2>&, const Array_gpu<float,1>&, Array_gpu<float,3>&);
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE, Array_gpu<float,3>&);
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE,
                  const Array_gpu<float,2>&, Array_gpu<float,3>&);

template void rte_kernel_launcher_cuda::sw_solver_2stream<float>(
            const int, const int, const int, const BOOL_TYPE,
            const Array_gpu<float,3>&, const Array_gpu<float,3>&, const Array_gpu<float,3>&,
            const Array_gpu<float,1>&,
            const Array_gpu<float,2>&, const Array_gpu<float,2>&,
            const Array_gpu<float,2>&,
            Array_gpu<float,3>&, Array_gpu<float,3>&, Array_gpu<float,3>&,
            const BOOL_TYPE, const Array_gpu<float,2>&,
            const BOOL_TYPE, Array_gpu<float,3>&, Array_gpu<float,3>&, Array_gpu<float,3>&,
            Tuner_map&);

template void rte_kernel_launcher_cuda::lw_solver_noscat_gaussquad<float>(
            const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
            const Array_gpu<float,2>& ds, const Array_gpu<float,2>& weights, const Array_gpu<float,3>& tau, const Array_gpu<float,3> lay_source,
            const Array_gpu<float,3>& lev_source_inc, const Array_gpu<float,3>& lev_source_dec, const Array_gpu<float,2>& sfc_emis,
            const Array_gpu<float,2>& sfc_src, Array_gpu<float,3>& flux_dn, Array_gpu<float,3>& flux_up,
            const Array_gpu<float,2>& sfc_src_jac, Array_gpu<float,3>& flux_up_jac, Tuner_map& tunings);
#else
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE,
                  const Array_gpu<double,2>&, const Array_gpu<double,1>&, Array_gpu<double,3>&);
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE, Array_gpu<double,3>&);
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE,
                  const Array_gpu<double,2>&, Array_gpu<double,3>&);

template void rte_kernel_launcher_cuda::sw_solver_2stream<double>(
            const int, const int, const int, const BOOL_TYPE,
            const Array_gpu<double,3>&, const Array_gpu<double,3>&, const Array_gpu<double,3>&,
            const Array_gpu<double,1>&,
            const Array_gpu<double,2>&, const Array_gpu<double,2>&,
            const Array_gpu<double,2>&,
            Array_gpu<double,3>&, Array_gpu<double,3>&, Array_gpu<double,3>&,
            const BOOL_TYPE, const Array_gpu<double,2>&,
            const BOOL_TYPE, Array_gpu<double,3>&, Array_gpu<double,3>&, Array_gpu<double,3>&,
            Tuner_map&);

template void rte_kernel_launcher_cuda::lw_solver_noscat_gaussquad<double>(
            const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
            const Array_gpu<double,2>& ds, const Array_gpu<double,2>& weights, const Array_gpu<double,3>& tau, const Array_gpu<double,3> lay_source,
            const Array_gpu<double,3>& lev_source_inc, const Array_gpu<double,3>& lev_source_dec, const Array_gpu<double,2>& sfc_emis,
            const Array_gpu<double,2>& sfc_src, Array_gpu<double,3>& flux_up, Array_gpu<double,3>& flux_dn,
            const Array_gpu<double,2>& sfc_src_jac,Array_gpu<double,3>& flux_up_jac, Tuner_map& tunings);
#endif
