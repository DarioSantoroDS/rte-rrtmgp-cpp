#include "hip/hip_runtime.h"
#include <chrono>

#include "rrtmgp_kernel_launcher_cuda.h"
#include "tools_gpu.h"
#include "Array.h"

namespace
{
    // Add the kernels here.
    template<typename TF>__device__
    void interpolate2D_byflav_kernel(const TF* __restrict__ fminor,
                                     const TF* __restrict__ kin,
                                     const int gpt_start, const int gpt_end,
                                     TF* __restrict__ k,
                                     const int* __restrict__ jeta,
                                     const int jtemp,
                                     const int ngpt,
                                     const int neta)
    {
        const int band_gpt = gpt_end-gpt_start;
        const int j0 = jeta[0];
        const int j1 = jeta[1];
        for (int igpt=0; igpt<band_gpt; ++igpt)
        {
            k[igpt] = fminor[0] * kin[igpt + (j0-1)*ngpt + (jtemp-1)*neta*ngpt] +
                      fminor[1] * kin[igpt +  j0   *ngpt + (jtemp-1)*neta*ngpt] +
                      fminor[2] * kin[igpt + (j1-1)*ngpt + jtemp    *neta*ngpt] +
                      fminor[3] * kin[igpt +  j1   *ngpt + jtemp    *neta*ngpt];
        }
    }

    template<typename TF>__device__
    void interpolate3D_byflav_kernel(const TF* __restrict__ scaling,
                                     const TF* __restrict__ fmajor,
                                     const TF* __restrict__ k,
                                     const int gpt_start, const int gpt_end,
                                     const int* __restrict__ jeta,
                                     const int jtemp,
                                     const int jpress,
                                     const int ngpt,
                                     const int neta,
                                     const int npress,
                                     TF* __restrict__ tau_major)
    {
        const int band_gpt = gpt_end-gpt_start;
        const int j0 = jeta[0];
        const int j1 = jeta[1];
        for (int igpt=0; igpt<band_gpt; ++igpt)
        {
            tau_major[igpt] = scaling[0]*
                              (fmajor[0] * k[igpt + (j0-1)*ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                               fmajor[1] * k[igpt +  j0   *ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                               fmajor[2] * k[igpt + (j0-1)*ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress] +
                               fmajor[3] * k[igpt +  j0   *ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress])
                            + scaling[1]*
                              (fmajor[4] * k[igpt + (j1-1)*ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                               fmajor[5] * k[igpt +  j1   *ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                               fmajor[6] * k[igpt + (j1-1)*ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress] +
                               fmajor[7] * k[igpt +  j1   *ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress]);
        }
    }

    template<typename TF>__device__
    int locate_val(const TF* __restrict__ arr,
                   const int ncol,
                   const int nlay,
                   const BOOL_TYPE maxmin, //False: find minimum
                   const BOOL_TYPE* __restrict__ mask,
                   const BOOL_TYPE maskval)
    {
        TF temp = arr[0];
        for (int i=0; i<nlay; ++i)
        {
            const int ii = i*ncol;
            if (mask[ii]==maskval)
            {
                temp = arr[ii];
                break;
            }
        }
        int idx = 0;
        for (int i=0; i<nlay; ++i)
        {
            const int ii = i*ncol;
            if ((arr[ii]>temp) == maxmin and mask[ii]==maskval)
            {
                idx = i;
                temp = arr[ii];
            }
        }
        return idx;
    }

    template<typename TF>__global__
    void interpolation_kernel(
            const int ncol, const int nlay, const int ngas, const int nflav,
            const int neta, const int npres, const int ntemp, const TF tmin,
            const int* __restrict__ flavor,
            const TF* __restrict__ press_ref_log,
            const TF* __restrict__ temp_ref,
            TF press_ref_log_delta,
            TF temp_ref_min,
            TF temp_ref_delta,
            TF press_ref_trop_log,
            const TF* __restrict__ vmr_ref,
            const TF* __restrict__ play,
            const TF* __restrict__ tlay,
            TF* __restrict__ col_gas,
            int* __restrict__ jtemp,
            TF* __restrict__ fmajor, TF* __restrict__ fminor,
            TF* __restrict__ col_mix,
            BOOL_TYPE* __restrict__ tropo,
            int* __restrict__ jeta,
            int* __restrict__ jpress)
    {
        const int ilay = blockIdx.x*blockDim.x + threadIdx.x;
        const int icol = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (ilay < nlay) )
        {
            const int idx = icol + ilay*ncol;

            jtemp[idx] = int((tlay[idx] - (temp_ref_min-temp_ref_delta)) / temp_ref_delta);
            jtemp[idx] = min(ntemp-1, max(1, jtemp[idx]));
            const TF ftemp = (tlay[idx] - temp_ref[jtemp[idx]-1]) / temp_ref_delta;

            const TF locpress = TF(1.) + (log(play[idx]) - press_ref_log[0]) / press_ref_log_delta;
            jpress[idx] = min(npres-1, max(1, int(locpress)));
            const TF fpress = locpress - TF(jpress[idx]);

            tropo[idx] = log(play[idx]) > press_ref_trop_log;
            const int itropo = !tropo[idx];

            for (int iflav=0; iflav<nflav; ++iflav)
            {
                const int gas1 = flavor[2*iflav];
                const int gas2 = flavor[2*iflav+1];
                for (int itemp=0; itemp<2; ++itemp)
                {
                    const int vmr_base_idx = itropo + (jtemp[idx]+itemp-1) * (ngas+1) * 2;
                    const int colmix_idx = itemp + 2*(iflav + nflav*icol + nflav*ncol*ilay);
                    const int colgas1_idx = icol + ilay*ncol + gas1*nlay*ncol;
                    const int colgas2_idx = icol + ilay*ncol + gas2*nlay*ncol;
                    TF eta;
                    const TF ratio_eta_half = vmr_ref[vmr_base_idx + 2 * gas1] /
                                              vmr_ref[vmr_base_idx + 2 * gas2];
                    col_mix[colmix_idx] = col_gas[colgas1_idx] + ratio_eta_half * col_gas[colgas2_idx];
                    if (col_mix[colmix_idx] > TF(2.)*tmin)
                    {
                        eta = col_gas[colgas1_idx] / col_mix[colmix_idx];
                    } else
                    {
                        eta = TF(0.5);
                    }
                    const TF loceta = eta * TF(neta-1);
                    jeta[colmix_idx] = min(int(loceta)+1, neta-1);
                    const TF feta = fmod(loceta, TF(1.));
                    const TF ftemp_term  = TF(1-itemp) + TF(2*itemp-1)*ftemp;
                    // compute interpolation fractions needed for minot species
                    const int fminor_idx = 2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
                    fminor[fminor_idx] = (TF(1.0)-feta) * ftemp_term;
                    fminor[fminor_idx+1] = feta * ftemp_term;
                    // compute interpolation fractions needed for major species
                    const int fmajor_idx = 2*2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
                    fmajor[fmajor_idx] = (TF(1.0)-fpress) * fminor[fminor_idx];
                    fmajor[fmajor_idx+1] = (TF(1.0)-fpress) * fminor[fminor_idx+1];
                    fmajor[fmajor_idx+2] = fpress * fminor[fminor_idx];
                    fmajor[fmajor_idx+3] = fpress * fminor[fminor_idx+1];

                }
            }
        }
    }

    template<typename TF>__global__
    void compute_tau_major_absorption_kernel(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int* __restrict__ gpoint_flavor,
            const int* __restrict__ band_lims_gpt,
            const TF* __restrict__ kmajor,
            const TF* __restrict__ col_mix, const TF* __restrict__ fmajor,
            const int* __restrict__ jeta, const BOOL_TYPE* __restrict__ tropo,
            const int* __restrict__ jtemp, const int* __restrict__ jpress,
            TF* __restrict__ tau, TF* __restrict__ tau_major)
    {
        // Fetch the three coordinates.
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nband) ) {
            const int idx_collay = icol + ilay * ncol;
            const int itropo = !tropo[idx_collay];
            const int gpt_start = band_lims_gpt[2 * ibnd] - 1;
            const int gpt_end = band_lims_gpt[2 * ibnd + 1];
            const int iflav = gpoint_flavor[itropo + 2 * gpt_start] - 1;
            const int idx_fcl3 = 2 * 2 * 2* (iflav + icol * nflav + ilay * ncol * nflav);
            const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
            const int idx_tau = gpt_start + ilay * ngpt + icol * nlay * ngpt;

            //major gases//
            interpolate3D_byflav_kernel(&col_mix[idx_fcl1], &fmajor[idx_fcl3],
                                        &kmajor[gpt_start], gpt_start, gpt_end,
                                        &jeta[idx_fcl1], jtemp[idx_collay], 
                                        jpress[idx_collay]+itropo, ngpt, neta, npres+1,
                                        &tau_major[idx_tau]);
            
            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
                tau[idx_out] = tau_major[idx_out];
                //should be '+=' later on, but we first need the zero_arrays for that
            }
        }
    }

    template<typename TF>__global__
    void compute_tau_minor_absorption_kernel(
            const int ncol, const int nlay, const int ngpt,
            const int ngas, const int nflav, const int ntemp, const int neta,
            const int nscale_lower,
            const int nscale_upper,
            const int nminor_lower,
            const int nminor_upper,
            const int nminork_lower,
            const int nminork_upper,
            const int idx_h2o,
            const int* __restrict__ gpoint_flavor,
            const TF* __restrict__ kminor_lower,
            const TF* __restrict__ kminor_upper,
            const int* __restrict__ minor_limits_gpt_lower,
            const int* __restrict__ minor_limits_gpt_upper,
            const BOOL_TYPE* __restrict__ minor_scales_with_density_lower,
            const BOOL_TYPE* __restrict__ minor_scales_with_density_upper,
            const BOOL_TYPE* __restrict__ scale_by_complement_lower,
            const BOOL_TYPE* __restrict__ scale_by_complement_upper,
            const int* __restrict__ idx_minor_lower,
            const int* __restrict__ idx_minor_upper,
            const int* __restrict__ idx_minor_scaling_lower,
            const int* __restrict__ idx_minor_scaling_upper,
            const int* __restrict__ kminor_start_lower,
            const int* __restrict__ kminor_start_upper,
            const TF* __restrict__ play,
            const TF* __restrict__ tlay,
            const TF* __restrict__ col_gas,
            const TF* __restrict__ fminor,
            const int* __restrict__ jeta,
            const int* __restrict__ jtemp,
            const BOOL_TYPE* __restrict__ tropo,
            TF* __restrict__ tau,
            TF* __restrict__ tau_minor) 
    {
        // Fetch the three coordinates.
        const int ilay = blockIdx.x * blockDim.x + threadIdx.x;
        const int icol = blockIdx.y * blockDim.y + threadIdx.y;
        const TF PaTohPa = 0.01;
        const int ncl = ncol * nlay;
        if ((icol < ncol) && (ilay < nlay)) 
        {
            //kernel implementation
            const int idx_collay = icol + ilay * ncol;
            const int idx_collaywv = icol + ilay * ncol + idx_h2o * ncl;

            if (tropo[idx_collay] == 1) 
            {
                for (int imnr = 0; imnr < nscale_lower; ++imnr)
                {
                    TF scaling = col_gas[idx_collay + idx_minor[imnr] * ncl];
                    if (minor_scales_with_density_lower[imnr])
                    {
                        scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];
                        if (idx_minor_scaling_lower[imnr] > 0)
                        {
                            TF vmr_fact = TF(1.) / col_gas[idx_collay];
                            TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);
                            if (scale_by_complement_lower[imnr])
                            {
                                scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling_lower[imnr] * ncl] * vmr_fact * dry_fact);
                            } 
                            else
                            {
                                scaling *= col_gas[idx_collay + idx_minor_scaling_lower[imnr] * ncl] * vmr_fact * dry_fact;
                            }
                        }
                    }
                    const int gpt_start = minor_limits_gpt_lower[2*imnr]-1;
                    const int gpt_end = minor_limits_gpt_lower[2*imnr+1];
                    const int iflav = gpoint_flavor[2*gpt_start]-1;
                    const int idx_fcl2 = 2 * 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                    interpolate2D_byflav_kernel(&fminor[idx_fcl2], &kminor[kminor_start_lower[imnr]-1],
                                                kminor_start_lower[imnr]-1, kminor_start_lower[imnr]-1 + (gpt_end - gpt_start),
                                                &tau_minor[idx_tau], &jeta[idx_fcl1],
                                                jtemp[idx_collay], nminork_lower, neta);

                    for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                    {
                        const int idx_out = igpt + ilay * ngpt + icol * nlay * ngpt;
                        tau[idx_out] += tau_minor[idx_out] * scaling;
                    }
                }
            }
            else
            {
                for (int imnr = 0; imnr < nscale_upper; ++imnr)
                {
                    TF scaling = col_gas[idx_collay + idx_minor[imnr] * ncl];
                    if (minor_scales_with_density_upper[imnr])
                    {
                        scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];
                        if (idx_minor_scaling_upper[imnr] > 0)
                        {
                            TF vmr_fact = TF(1.) / col_gas[idx_collay];
                            TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);
                            if (scale_by_complement_upper[imnr])
                            {
                                scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling_upper[imnr] * ncl] * vmr_fact * dry_fact);
                            }
                            else
                            {
                                scaling *= col_gas[idx_collay + idx_minor_scaling_upper[imnr] * ncl] * vmr_fact * dry_fact;
                            }
                        }
                    }
                    const int gpt_start = minor_limits_gpt_upper[2*imnr]-1;
                    const int gpt_end = minor_limits_gpt_upper[2*imnr+1];
                    const int iflav = gpoint_flavor[2*gpt_start+1]-1;
                    const int idx_fcl2 = 2 * 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                    interpolate2D_byflav_kernel(&fminor[idx_fcl2], &kminor[kminor_start_upper[imnr]-1],
                                                kminor_start_upper[imnr]-1, kminor_start_upper[imnr]-1 + (gpt_end - gpt_start),
                                                &tau_minor[idx_tau], &jeta[idx_fcl1],
                                                jtemp[idx_collay], nminork_upper, neta);

                    for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                    {
                        const int idx_out = igpt + ilay * ngpt + icol * nlay * ngpt;
                        tau[idx_out] += tau_minor[idx_out] * scaling;
                    }
                }
            }
        }
    }

    template<typename TF>__global__
    void compute_tau_rayleigh_kernel(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* __restrict__ gpoint_flavor,
            const int* __restrict__ band_lims_gpt,
            const TF* __restrict__ krayl,
            int idx_h2o, const TF* __restrict__ col_dry, const TF* __restrict__ col_gas,
            const TF* __restrict__ fminor, const int* __restrict__ jeta,
            const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
            TF* __restrict__ tau_rayleigh, TF* __restrict__ k)
    {
        // Fetch the three coordinates.
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nbnd) )
        {
            //kernel implementation
            const int idx_collay = icol + ilay*ncol;
            const int idx_collaywv = icol + ilay*ncol + idx_h2o*nlay*ncol;
            const int itropo = !tropo[idx_collay];
            const int gpt_start = band_lims_gpt[2*ibnd]-1;
            const int gpt_end = band_lims_gpt[2*ibnd+1];
            const int iflav = gpoint_flavor[itropo+2*gpt_start]-1;
            const int idx_fcl2 = 2*2*(iflav + icol*nflav + ilay*ncol*nflav);
            const int idx_fcl1   = 2*(iflav + icol*nflav + ilay*ncol*nflav);
            const int idx_krayl  = gpt_start + ngpt*neta*ntemp*itropo;
            const int idx_k = gpt_start + ilay*ngpt + icol*nlay*ngpt;
            interpolate2D_byflav_kernel(&fminor[idx_fcl2],
                                        &krayl[idx_krayl],
                                        gpt_start, gpt_end, &k[idx_k],
                                        &jeta[idx_fcl1],
                                        jtemp[idx_collay],
                                        ngpt, neta);

            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
                tau_rayleigh[idx_out] = k[idx_k+igpt-gpt_start]*(col_gas[idx_collaywv]+col_dry[idx_collay]);
            }
        }
    }

    
    template<typename TF>__global__
    void combine_and_reorder_2str_kernel(
            const int ncol, const int nlay, const int ngpt, const TF tmin,
            const TF* __restrict__ tau_abs, const TF* __restrict__ tau_rayleigh,
            TF* __restrict__ tau, TF* __restrict__ ssa, TF* __restrict__ g)
    {
        // Fetch the three coordinates.
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
        const int ilay = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
        {
            const int idx_in  = igpt + ilay*ngpt + icol*(ngpt*nlay);
            const int idx_out = icol + ilay*ncol + igpt*(ncol*nlay);
	   
            const TF tau_tot = tau_abs[idx_in] + tau_rayleigh[idx_in];
            tau[idx_out] = tau_tot;
            g  [idx_out] = TF(0.);
            if (tau_tot>(TF(2.)*tmin))
                ssa[idx_out] = tau_rayleigh[idx_in]/tau_tot;
            else
                ssa[idx_out] = 0.;
        }
    }
    
    
    
}

namespace rrtmgp_kernel_launcher_cuda
{
    template<typename TF>
    void interpolation(
            const int ncol, const int nlay,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const Array<int,2>& flavor,
            const Array<TF,1>& press_ref_log,
            const Array<TF,1>& temp_ref,
            TF press_ref_log_delta,
            TF temp_ref_min,
            TF temp_ref_delta,
            TF press_ref_trop_log,
            const Array<TF,3>& vmr_ref,
            const Array<TF,2>& play,
            const Array<TF,2>& tlay,
            Array<TF,3>& col_gas,
            Array<int,2>& jtemp,
            Array<TF,6>& fmajor, Array<TF,5>& fminor,
            Array<TF,4>& col_mix,
            Array<BOOL_TYPE,2>& tropo,
            Array<int,4>& jeta,
            Array<int,2>& jpress)
    {
        const int flavor_size = flavor.size() * sizeof(int);
        const int press_ref_log_size = press_ref_log.size() * sizeof(TF);
        const int temp_ref_size = temp_ref.size() * sizeof(TF);
        const int vmr_ref_size = vmr_ref.size() * sizeof(TF);
        const int collay_tf_size = ncol * nlay * sizeof(TF);
        const int collay_int_size = ncol * nlay * sizeof(int);
        const int col_gas_size = col_gas.size() * sizeof(TF);
        const int jtemp_size = jtemp.size() * sizeof(int);
        const int fmajor_size = fmajor.size() * sizeof(TF);
        const int fminor_size = fminor.size() * sizeof(TF);
        const int col_mix_size = col_mix.size() * sizeof(TF);
        const int tropo_size = tropo.size() * sizeof(BOOL_TYPE);
        const int jeta_size = jeta.size() * sizeof(int);
        const int jpress_size = jpress.size() * sizeof(int);

        int* flavor_gpu;
        TF* press_ref_log_gpu;
        TF* temp_ref_gpu;
        TF* vmr_ref_gpu;
        TF* play_gpu;
        TF* tlay_gpu;
        TF* col_gas_gpu;
        int* jtemp_gpu;
        TF* fmajor_gpu;
        TF* fminor_gpu;
        TF* col_mix_gpu;
        BOOL_TYPE* tropo_gpu;
        int* jeta_gpu;
        int* jpress_gpu;

        cuda_safe_call(hipMalloc((void **) &flavor_gpu, flavor_size));
        cuda_safe_call(hipMalloc((void **) &press_ref_log_gpu, press_ref_log_size));
        cuda_safe_call(hipMalloc((void **) &temp_ref_gpu, temp_ref_size));
        cuda_safe_call(hipMalloc((void **) &vmr_ref_gpu, vmr_ref_size));
        cuda_safe_call(hipMalloc((void **) &play_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void **) &tlay_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void **) &col_gas_gpu, col_gas_size));
        cuda_safe_call(hipMalloc((void **) &jtemp_gpu, collay_int_size));
        cuda_safe_call(hipMalloc((void **) &fmajor_gpu, fmajor_size));
        cuda_safe_call(hipMalloc((void **) &fminor_gpu, fminor_size));
        cuda_safe_call(hipMalloc((void **) &col_mix_gpu, col_mix_size));
        cuda_safe_call(hipMalloc((void **) &tropo_gpu, tropo_size));
        cuda_safe_call(hipMalloc((void **) &jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void **) &jpress_gpu, collay_int_size));

        cuda_safe_call(hipMemcpy(flavor_gpu, flavor.ptr(), flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(press_ref_log_gpu, press_ref_log.ptr(), press_ref_log_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(temp_ref_gpu, temp_ref.ptr(), temp_ref_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(vmr_ref_gpu, vmr_ref.ptr(), vmr_ref_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(play_gpu, play.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tlay_gpu, tlay.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_gas_gpu, col_gas.ptr(), col_gas_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        const int block_lay = 16;
        const int block_col = 32;

        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);

        dim3 grid_gpu(grid_lay, grid_col);
        dim3 block_gpu(block_lay, block_col);

        TF tmin = std::numeric_limits<TF>::min();
        interpolation_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngas, nflav, neta, npres, ntemp, tmin,
                flavor_gpu, press_ref_log_gpu, temp_ref_gpu,
                press_ref_log_delta, temp_ref_min,
                temp_ref_delta, press_ref_trop_log,
                vmr_ref_gpu, play_gpu, tlay_gpu,
                col_gas_gpu, jtemp_gpu, fmajor_gpu,
                fminor_gpu, col_mix_gpu, tropo_gpu,
                jeta_gpu, jpress_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU interpolation: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(jtemp.ptr(), jtemp_gpu, jtemp_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(fmajor.ptr(), fmajor_gpu, fmajor_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(fminor.ptr(), fminor_gpu, fminor_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(col_mix.ptr(), col_mix_gpu, col_mix_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(tropo.ptr(), tropo_gpu, tropo_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(jeta.ptr(), jeta_gpu, jeta_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(jpress.ptr(), jpress_gpu, jpress_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(flavor_gpu));
        cuda_safe_call(hipFree(press_ref_log_gpu));
        cuda_safe_call(hipFree(temp_ref_gpu));
        cuda_safe_call(hipFree(vmr_ref_gpu));
        cuda_safe_call(hipFree(play_gpu));
        cuda_safe_call(hipFree(tlay_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(fmajor_gpu));
        cuda_safe_call(hipFree(fminor_gpu));
        cuda_safe_call(hipFree(col_mix_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(jpress_gpu));
    }

    template<typename TF>
    void combine_and_reorder_2str(
            const int ncol, const int nlay, const int ngpt,
            const Array<TF,3>& tau_abs, const Array<TF,3>& tau_rayleigh,
            Array<TF,3>& tau, Array<TF,3>& ssa, Array<TF,3>& g)
    {
        // Store the sizes, all the same
        const int array_size = tau_abs.size()*sizeof(TF);

        TF* tau_abs_gpu;
        TF* tau_rayleigh_gpu;
        TF* tau_gpu;
        TF* ssa_gpu;
        TF* g_gpu;

        TF tmin = std::numeric_limits<TF>::min();
        // Allocate a CUDA array.
        cuda_safe_call(hipMalloc((void**)&tau_abs_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&tau_rayleigh_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&tau_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&ssa_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&g_gpu, array_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(tau_abs_gpu, tau_abs.ptr(), array_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tau_rayleigh_gpu, tau_rayleigh.ptr(), array_size, hipMemcpyHostToDevice));
        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);

        hipEventRecord(startEvent, 0);
        // Call the kernel.
        const int block_col = 32;
        const int block_gpt = 32;
        const int block_lay = 1;

        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt  = ngpt/block_gpt + (ngpt%block_gpt > 0);
        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);

        dim3 grid_gpu(grid_col, grid_gpt, grid_lay);
        dim3 block_gpu(block_col, block_gpt, block_lay);

        combine_and_reorder_2str_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngpt, tmin,
                tau_abs_gpu, tau_rayleigh_gpu,
                tau_gpu, ssa_gpu, g_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU combine_and_reorder_2str: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(tau.ptr(), tau_gpu, array_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(ssa.ptr(), ssa_gpu, array_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(g.ptr(), g_gpu, array_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(tau_abs_gpu));
        cuda_safe_call(hipFree(tau_rayleigh_gpu));
        cuda_safe_call(hipFree(tau_gpu));
        cuda_safe_call(hipFree(ssa_gpu));
        cuda_safe_call(hipFree(g_gpu));
    }
    
    template<typename TF>
    void compute_tau_rayleigh(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const Array<int,2>& gpoint_flavor,
            const Array<int,2>& band_lims_gpt,
            const Array<TF,4>& krayl,
            int idx_h2o, const Array<TF,2>& col_dry, const Array<TF,3>& col_gas,
            const Array<TF,5>& fminor, const Array<int,4>& jeta,
            const Array<BOOL_TYPE,2>& tropo, const Array<int,2>& jtemp,
            Array<TF,3>& tau_rayleigh)
    {
        float elapsedtime;
        const int gpoint_flavor_size = gpoint_flavor.size()*sizeof(int);
        const int band_lims_gpt_size = band_lims_gpt.size()*sizeof(int);
        const int krayl_size = krayl.size()*sizeof(TF);
        const int col_dry_size = col_dry.size()*sizeof(TF);
        const int col_gas_size = col_gas.size()*sizeof(TF);
        const int fminor_size = fminor.size()*sizeof(TF);
        const int jeta_size = jeta.size()*sizeof(int);
        const int tropo_size = tropo.size()*sizeof(BOOL_TYPE);
        const int jtemp_size = jtemp.size()*sizeof(int);
        const int tau_rayleigh_size = tau_rayleigh.size()*sizeof(TF);

        int* gpoint_flavor_gpu;
        int* band_lims_gpt_gpu;
        int* jeta_gpu;
        int* jtemp_gpu;
        BOOL_TYPE* tropo_gpu;
        TF* krayl_gpu;
        TF* col_dry_gpu;
        TF* col_gas_gpu;
        TF* fminor_gpu;
        TF* tau_rayleigh_gpu;
        TF* k_gpu;

        // Allocate a CUDA array.
        cuda_safe_call(hipMalloc((void**)&gpoint_flavor_gpu, gpoint_flavor_size));
        cuda_safe_call(hipMalloc((void**)&band_lims_gpt_gpu, band_lims_gpt_size));
        cuda_safe_call(hipMalloc((void**)&krayl_gpu, krayl_size));
        cuda_safe_call(hipMalloc((void**)&col_dry_gpu, col_dry_size));
        cuda_safe_call(hipMalloc((void**)&col_gas_gpu, col_gas_size));
        cuda_safe_call(hipMalloc((void**)&fminor_gpu, fminor_size));
        cuda_safe_call(hipMalloc((void**)&jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void**)&tropo_gpu, tropo_size));
        cuda_safe_call(hipMalloc((void**)&jtemp_gpu, jtemp_size));
        cuda_safe_call(hipMalloc((void**)&tau_rayleigh_gpu, tau_rayleigh_size));
        cuda_safe_call(hipMalloc((void**)&k_gpu, tau_rayleigh_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(gpoint_flavor_gpu, gpoint_flavor.ptr(), gpoint_flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(band_lims_gpt_gpu, band_lims_gpt.ptr(), band_lims_gpt_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(krayl_gpu, krayl.ptr(), krayl_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_dry_gpu, col_dry.ptr(), col_dry_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_gas_gpu, col_gas.ptr(), col_gas_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fminor_gpu, fminor.ptr(), fminor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jeta_gpu, jeta.ptr(), jeta_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tropo_gpu, tropo.ptr(), tropo_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jtemp_gpu, jtemp.ptr(), jtemp_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        // Call the kernel.
        const int block_bnd = 14;
        const int block_lay = 1;
        const int block_col = 32;

        const int grid_bnd  = nbnd/block_bnd + (nbnd%block_bnd > 0);
        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);

        dim3 grid_gpu(grid_bnd, grid_lay, grid_col);
        dim3 block_gpu(block_bnd, block_lay, block_col);

        compute_tau_rayleigh_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                gpoint_flavor_gpu,
                band_lims_gpt_gpu,
                krayl_gpu,
                idx_h2o, col_dry_gpu, col_gas_gpu,
                fminor_gpu, jeta_gpu,
                tropo_gpu, jtemp_gpu,
                tau_rayleigh_gpu, k_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU compute_tau_rayleigh: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(tau_rayleigh.ptr(), tau_rayleigh_gpu, tau_rayleigh_size, hipMemcpyDeviceToHost));
        
        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(gpoint_flavor_gpu));
        cuda_safe_call(hipFree(band_lims_gpt_gpu));
        cuda_safe_call(hipFree(krayl_gpu));
        cuda_safe_call(hipFree(col_dry_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(fminor_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(tau_rayleigh_gpu));
        cuda_safe_call(hipFree(k_gpu));
    }

    template<typename TF>
    void compute_tau_absorption(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int nminorlower, const int nminorklower,
            const int nminorupper, const int nminorkupper,
            const int idx_h2o,
            const Array<int,2>& gpoint_flavor,
            const Array<int,2>& band_lims_gpt,
            const Array<TF,4>& kmajor,
            const Array<TF,3>& kminor_lower,
            const Array<TF,3>& kminor_upper,
            const Array<int,2>& minor_limits_gpt_lower,
            const Array<int,2>& minor_limits_gpt_upper,
            const Array<BOOL_TYPE,1>& minor_scales_with_density_lower,
            const Array<BOOL_TYPE,1>& minor_scales_with_density_upper,
            const Array<BOOL_TYPE,1>& scale_by_complement_lower,
            const Array<BOOL_TYPE,1>& scale_by_complement_upper,
            const Array<int,1>& idx_minor_lower,
            const Array<int,1>& idx_minor_upper,
            const Array<int,1>& idx_minor_scaling_lower,
            const Array<int,1>& idx_minor_scaling_upper,
            const Array<int,1>& kminor_start_lower,
            const Array<int,1>& kminor_start_upper,
            const Array<BOOL_TYPE,2>& tropo,
            const Array<TF,4>& col_mix, const Array<TF,6>& fmajor,
            const Array<TF,5>& fminor, const Array<TF,2>& play,
            const Array<TF,2>& tlay, const Array<TF,3>& col_gas,
            const Array<int,4>& jeta, const Array<int,2>& jtemp,
            const Array<int,2>& jpress, Array<TF,3>& tau)
    {
        float elapsedtime;
        const int gpoint_flavor_size = gpoint_flavor.size()*sizeof(int);
        const int band_lims_gpt_size = band_lims_gpt.size()*sizeof(int);
        const int kmajor_size = kmajor.size()*sizeof(TF);
        const int kminor_lower_size = kminor_lower.size()*sizeof(TF);
        const int kminor_upper_size = kminor_upper.size()*sizeof(TF);
        const int minor_limits_gpt_lower_size = minor_limits_gpt_lower.size()*sizeof(int);
        const int minor_limits_gpt_upper_size = minor_limits_gpt_upper.size()*sizeof(int);
        const int nminorlower_bool_size = nminorlower*sizeof(BOOL_TYPE); //minor scales with/scale by complement
        const int nminorupper_bool_size = nminorlower*sizeof(BOOL_TYPE); //minor scales with/scale by complement
        const int nminorlower_int_size = nminorlower*sizeof(int); //idx_minor(scaling) kminor
        const int nminorupper_int_size = nminorlower*sizeof(int);
        const int tropo_size = tropo.size()*sizeof(BOOL_TYPE);
        const int col_mix_size = col_mix.size()*sizeof(TF);
        const int fmajor_size = fmajor.size()*sizeof(TF);
        const int fminor_size = fminor.size()*sizeof(TF);
        const int collay_tf_size = ncol*nlay*sizeof(TF); //tlay,play
        const int col_gas_size = col_gas.size()*sizeof(TF);
        const int jeta_size =  jeta.size()*sizeof(int);
        const int collay_int_size = ncol*nlay*sizeof(int);
        const int itropo_size = 2*ncol*sizeof(int);
        const int tau_size = tau.size()*sizeof(TF);
        
        int* gpoint_flavor_gpu;
        int* band_lims_gpt_gpu;
        TF* kmajor_gpu;
        TF* kminor_lower_gpu;
        TF* kminor_upper_gpu;
        int* minor_limits_gpt_lower_gpu;
        int* minor_limits_gpt_upper_gpu;
        BOOL_TYPE* minor_scales_with_density_lower_gpu;
        BOOL_TYPE* minor_scales_with_density_upper_gpu;
        BOOL_TYPE* scale_by_complement_lower_gpu;
        BOOL_TYPE* scale_by_complement_upper_gpu;
        int* idx_minor_lower_gpu;
        int* idx_minor_upper_gpu;
        int* idx_minor_scaling_lower_gpu;
        int* idx_minor_scaling_upper_gpu;
        int* kminor_start_lower_gpu;
        int* kminor_start_upper_gpu;
        BOOL_TYPE* tropo_gpu;
        TF* col_mix_gpu;
        TF* fmajor_gpu;
        TF* fminor_gpu;
        TF* play_gpu;
        TF* tlay_gpu;
        TF* col_gas_gpu;
        int* jeta_gpu;
        int* jtemp_gpu;
        int* jpress_gpu;
        int* itropo_lower_gpu;
        int* itropo_upper_gpu;
        TF* tau_gpu;
        TF* tau_major_gpu;
        TF* tau_minor_gpu;

        // Allocate a CUDA array.
        cuda_safe_call(hipMalloc((void**)& gpoint_flavor_gpu, gpoint_flavor_size));
        cuda_safe_call(hipMalloc((void**)& band_lims_gpt_gpu, band_lims_gpt_size));
        cuda_safe_call(hipMalloc((void**)& kmajor_gpu, kmajor_size));
        cuda_safe_call(hipMalloc((void**)& kminor_lower_gpu, kminor_lower_size));
        cuda_safe_call(hipMalloc((void**)& kminor_upper_gpu, kminor_upper_size));
        cuda_safe_call(hipMalloc((void**)& minor_limits_gpt_lower_gpu, minor_limits_gpt_lower_size));
        cuda_safe_call(hipMalloc((void**)& minor_limits_gpt_upper_gpu, minor_limits_gpt_upper_size));
        cuda_safe_call(hipMalloc((void**)& minor_scales_with_density_lower_gpu, nminorlower_bool_size));
        cuda_safe_call(hipMalloc((void**)& minor_scales_with_density_upper_gpu, nminorupper_bool_size));
        cuda_safe_call(hipMalloc((void**)& scale_by_complement_lower_gpu, nminorlower_bool_size));
        cuda_safe_call(hipMalloc((void**)& scale_by_complement_upper_gpu, nminorupper_bool_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_lower_gpu, nminorlower_int_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_upper_gpu, nminorupper_int_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_scaling_lower_gpu, nminorlower_int_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_scaling_upper_gpu, nminorupper_int_size));
        cuda_safe_call(hipMalloc((void**)& kminor_start_lower_gpu, nminorlower_int_size));
        cuda_safe_call(hipMalloc((void**)& kminor_start_upper_gpu, nminorupper_int_size));
        cuda_safe_call(hipMalloc((void**)& tropo_gpu, tropo_size));
        cuda_safe_call(hipMalloc((void**)& col_mix_gpu, col_mix_size));
        cuda_safe_call(hipMalloc((void**)& fmajor_gpu, fmajor_size));
        cuda_safe_call(hipMalloc((void**)& fminor_gpu, fminor_size));
        cuda_safe_call(hipMalloc((void**)& play_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void**)& tlay_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void**)& col_gas_gpu, col_gas_size));
        cuda_safe_call(hipMalloc((void**)& jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void**)& jtemp_gpu, collay_int_size));
        cuda_safe_call(hipMalloc((void**)& jpress_gpu, collay_int_size));
        cuda_safe_call(hipMalloc((void**)& itropo_lower_gpu, itropo_size));
        cuda_safe_call(hipMalloc((void**)& itropo_upper_gpu, itropo_size));
        cuda_safe_call(hipMalloc((void**)& tau_gpu, tau_size));
        cuda_safe_call(hipMalloc((void**)& tau_major_gpu, tau_size));
        cuda_safe_call(hipMalloc((void**)& tau_minor_gpu, tau_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(gpoint_flavor_gpu, gpoint_flavor.ptr(), gpoint_flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(band_lims_gpt_gpu, band_lims_gpt.ptr(), band_lims_gpt_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kmajor_gpu, kmajor.ptr(), kmajor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_lower_gpu, kminor_lower.ptr(), kminor_lower_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_upper_gpu, kminor_upper.ptr(), kminor_upper_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_limits_gpt_lower_gpu, minor_limits_gpt_lower.ptr(), minor_limits_gpt_lower_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_limits_gpt_upper_gpu, minor_limits_gpt_upper.ptr(), minor_limits_gpt_upper_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_scales_with_density_lower_gpu, minor_scales_with_density_lower.ptr(), nminorlower_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_scales_with_density_upper_gpu, minor_scales_with_density_upper.ptr(), nminorupper_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(scale_by_complement_lower_gpu, scale_by_complement_lower.ptr(), nminorlower_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(scale_by_complement_upper_gpu, scale_by_complement_upper.ptr(), nminorupper_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_lower_gpu, idx_minor_lower.ptr(), nminorlower_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_upper_gpu, idx_minor_upper.ptr(), nminorupper_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_scaling_lower_gpu, idx_minor_scaling_lower.ptr(), nminorlower_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_scaling_upper_gpu, idx_minor_scaling_upper.ptr(), nminorupper_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_start_lower_gpu, kminor_start_lower.ptr(), nminorlower_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_start_upper_gpu, kminor_start_upper.ptr(), nminorupper_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tropo_gpu, tropo.ptr(), tropo_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_mix_gpu, col_mix.ptr(), col_mix_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fmajor_gpu, fmajor.ptr(), fmajor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fminor_gpu, fminor.ptr(), fminor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(play_gpu, play.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tlay_gpu, tlay.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_gas_gpu, col_gas.ptr(), col_gas_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jeta_gpu, jeta.ptr(), jeta_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jtemp_gpu, jtemp.ptr(), collay_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jpress_gpu, jpress.ptr(), collay_int_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        // Call the kernel.
        const int block_bnd_maj = 14;
        const int block_lay_maj = 1;
        const int block_col_maj = 32;

        const int grid_bnd_maj  = nband/block_bnd_maj + (nband%block_bnd_maj > 0);
        const int grid_lay_maj  = nlay/block_lay_maj + (nlay%block_lay_maj > 0);
        const int grid_col_maj  = ncol/block_col_maj + (ncol%block_col_maj > 0);

        dim3 grid_gpu_maj(grid_bnd_maj, grid_lay_maj, grid_col_maj);
        dim3 block_gpu_maj(block_bnd_maj, block_lay_maj, block_col_maj);

        compute_tau_major_absorption_kernel<<<grid_gpu_maj, block_gpu_maj>>>(
                ncol, nlay, nband, ngpt,
                nflav, neta, npres, ntemp,
                gpoint_flavor_gpu, band_lims_gpt_gpu,
                kmajor_gpu, col_mix_gpu, fmajor_gpu, jeta_gpu,
                tropo_gpu, jtemp_gpu, jpress_gpu,
                tau_gpu, tau_major_gpu);

        const int nscale_lower = scale_by_complement_lower.dim(1);
        const int nscale_upper = scale_by_complement_upper.dim(1);
        const int block_lay_min = 14;
        const int block_col_min = 32;

        const int grid_lay_min  = nlay/block_lay_min + (nlay%block_lay_min > 0);
        const int grid_col_min  = ncol/block_col_min + (ncol%block_col_min > 0);

        dim3 grid_gpu_min(grid_lay_min, grid_col_min);
        dim3 block_gpu_min(block_lay_min, block_col_min);

        compute_tau_minor_absorption_kernel<<<grid_gpu_min, block_gpu_min>>>(
                ncol, nlay, ngpt,
                ngas, nflav, ntemp, neta,
                nscale_lower, nscale_upper
                nminorlower, nminorupper,
                nminorklower,nminorkupper,
                idx_h2o,
                gpoint_flavor_gpu,
                kminor_lower_gpu, kminor_upper_gpu,
                minor_limits_gpt_lower_gpu, minor_limits_gpt_upper_gpu,
                minor_scales_with_density_lower_gpu, inor_scales_with_density_upper_gpu,
                scale_by_complement_lower_gpu, scale_by_complement_upper_gpu,
                idx_minor_lower_gpu, idx_minor_upper_gpu,
                idx_minor_scaling_lower_gpu, idx_minor_scaling_upper_gpu,
                kminor_start_lower_gpu, kminor_start_upper_gpu,
                play_gpu, tlay_gpu, col_gas_gpu,
                fminor_gpu, jeta_gpu, jtemp_gpu,
                tropo_gpu, tau_gpu, tau_minor_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU compute_tau_abs: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(tau.ptr(), tau_gpu, tau_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(gpoint_flavor_gpu));
        cuda_safe_call(hipFree(band_lims_gpt_gpu));
        cuda_safe_call(hipFree(kmajor_gpu));
        cuda_safe_call(hipFree(kminor_lower_gpu));
        cuda_safe_call(hipFree(kminor_upper_gpu));
        cuda_safe_call(hipFree(minor_limits_gpt_lower_gpu));
        cuda_safe_call(hipFree(minor_limits_gpt_upper_gpu));
        cuda_safe_call(hipFree(minor_scales_with_density_lower_gpu));
        cuda_safe_call(hipFree(minor_scales_with_density_upper_gpu));
        cuda_safe_call(hipFree(scale_by_complement_lower_gpu));
        cuda_safe_call(hipFree(scale_by_complement_upper_gpu));
        cuda_safe_call(hipFree(idx_minor_lower_gpu));
        cuda_safe_call(hipFree(idx_minor_upper_gpu));
        cuda_safe_call(hipFree(idx_minor_scaling_lower_gpu));
        cuda_safe_call(hipFree(idx_minor_scaling_upper_gpu));
        cuda_safe_call(hipFree(kminor_start_lower_gpu));
        cuda_safe_call(hipFree(kminor_start_upper_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
        cuda_safe_call(hipFree(col_mix_gpu));
        cuda_safe_call(hipFree(fmajor_gpu));
        cuda_safe_call(hipFree(fminor_gpu));
        cuda_safe_call(hipFree(play_gpu));
        cuda_safe_call(hipFree(tlay_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(jpress_gpu));
        cuda_safe_call(hipFree(itropo_lower_gpu));
        cuda_safe_call(hipFree(itropo_upper_gpu));
        cuda_safe_call(hipFree(tau_major_gpu));
        cuda_safe_call(hipFree(tau_minor_gpu));
        cuda_safe_call(hipFree(tau_gpu));
    }
}


#ifdef FLOAT_SINGLE_RRTMGP
template void rrtmgp_kernel_launcher_cuda::interpolation(
        const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<single,1>&, const Array<single,1>&,
        single, single, single, single, const Array<single,3>&, const Array<single,2>&,
        const Array<single,2>&, Array<single,3>&, Array<int,2>&, Array<single,6>&, Array<single,5>&,
        Array<single,4>&, Array<BOOL_TYPE,2>&, Array<int,4>&, Array<int,2>&);

template void rrtmgp_kernel_launcher_cuda::combine_and_reorder_2str<float>(
        const int, const int, const int, const Array<float,3>&, const Array<float,3>&, Array<float,3>&, Array<float,3>&, Array<float,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_rayleigh<float>(
        const int, const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<float,4>&, int, const Array<float,2>&, 
        const Array<float,3>&, const Array<float,5>&, const Array<int,4>&, const Array<BOOL_TYPE,2>&, 
        const Array<int,2>&, Array<float,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_absorption<float>(const int, const int, const int, const int, const int, const int, 
	const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<float,4>&, const Array<float,3>&, const Array<float,3>&,
        const Array<int,2>&, const Array<int,2>&, const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&,
        const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&, const Array<int,1>&, const Array<int,1>&,
        const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<BOOL_TYPE,2>& tropo,
        const Array<float,4>&, const Array<float,6>&, const Array<float,5>&, const Array<float,2>&, const Array<float,2>&, const Array<float,3>&,
        const Array<int,4>&, const Array<int,2>&, const Array<int,2>&, Array<float,3>&);

#else
template void rrtmgp_kernel_launcher_cuda::interpolation(
        const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<double,1>&, const Array<double,1>&,
        double, double, double, double, const Array<double,3>&, const Array<double,2>&,
        const Array<double,2>&, Array<double,3>&, Array<int,2>&, Array<double,6>&, Array<double,5>&,
        Array<double,4>&, Array<BOOL_TYPE,2>&, Array<int,4>&, Array<int,2>&);

template void rrtmgp_kernel_launcher_cuda::combine_and_reorder_2str<double>(
        const int, const int, const int, const Array<double,3>&, const Array<double,3>&, Array<double,3>&, Array<double,3>&, Array<double,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_rayleigh<double>(
        const int, const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<double,4>&, int, const Array<double,2>&, 
        const Array<double,3>&, const Array<double,5>&, const Array<int,4>&, const Array<BOOL_TYPE,2>&, 
        const Array<int,2>&, Array<double,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_absorption<double>(const int, const int, const int, const int, const int, const int, 
	const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<double,4>&, const Array<double,3>&, const Array<double,3>&,
        const Array<int,2>&, const Array<int,2>&, const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&,
        const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&, const Array<int,1>&, const Array<int,1>&,
        const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<BOOL_TYPE,2>& tropo,
        const Array<double,4>&, const Array<double,6>&, const Array<double,5>&, const Array<double,2>&, const Array<double,2>&, const Array<double,3>&,
        const Array<int,4>&, const Array<int,2>&, const Array<int,2>&, Array<double,3>&);
#endif


