#include "hip/hip_runtime.h"
#include <chrono>

#include "rrtmgp_kernel_launcher_cuda.h"
#include "tools_gpu.h"
#include "Array.h"

namespace
{

    template<typename TF>__device__
    TF interpolate1D(const TF val,
                     const TF offset,
                     const TF delta,
                     const int len,
                     const TF* __restrict__ table)   
    {
        TF val0 = (val - offset)/delta;
        TF frac = val0 - int(val0);
        int idx = min(len-1, max(1, int(val0)+1));
        return table[idx-1] + frac * (table[idx] - table[idx-1]);
    }
    
    template<typename TF>__device__
    void interpolate2D_byflav_kernel(const TF* __restrict__ fminor,
                                     const TF* __restrict__ kin,
                                     const int gpt_start, const int gpt_end,
                                     TF* __restrict__ k,
                                     const int* __restrict__ jeta,
                                     const int jtemp,
                                     const int ngpt,
                                     const int neta)
    {
        const int band_gpt = gpt_end-gpt_start;
        const int j0 = jeta[0];
        const int j1 = jeta[1];
        for (int igpt=0; igpt<band_gpt; ++igpt)
        {
            k[igpt] = fminor[0] * kin[igpt + (j0-1)*ngpt + (jtemp-1)*neta*ngpt] +
                      fminor[1] * kin[igpt +  j0   *ngpt + (jtemp-1)*neta*ngpt] +
                      fminor[2] * kin[igpt + (j1-1)*ngpt + jtemp    *neta*ngpt] +
                      fminor[3] * kin[igpt +  j1   *ngpt + jtemp    *neta*ngpt];
        }
    }

    template<typename TF>__device__
    void interpolate3D_byflav_kernel(const TF* __restrict__ scaling,
                                     const TF* __restrict__ fmajor,
                                     const TF* __restrict__ k,
                                     const int gpt_start, const int gpt_end,
                                     const int* __restrict__ jeta,
                                     const int jtemp,
                                     const int jpress,
                                     const int ngpt,
                                     const int neta,
                                     const int npress,
                                     TF* __restrict__ tau_major)
    {
        const int band_gpt = gpt_end-gpt_start;
        const int j0 = jeta[0];
        const int j1 = jeta[1];
        for (int igpt=0; igpt<band_gpt; ++igpt)
        {
            tau_major[igpt] = scaling[0]*
                              (fmajor[0] * k[igpt + (j0-1)*ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                               fmajor[1] * k[igpt +  j0   *ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                               fmajor[2] * k[igpt + (j0-1)*ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress] +
                               fmajor[3] * k[igpt +  j0   *ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress])
                            + scaling[1]*
                              (fmajor[4] * k[igpt + (j1-1)*ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                               fmajor[5] * k[igpt +  j1   *ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                               fmajor[6] * k[igpt + (j1-1)*ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress] +
                               fmajor[7] * k[igpt +  j1   *ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress]);
        }
    }


    template<typename TF>__global__
    void fill_gases_kernel(
            const int ncol, const int nlay, const int ngas, const int igas, const int dim1, const int dim2, 
            TF* __restrict__ vmr_out, TF* __restrict__ vmr_in,
            TF* __restrict__ col_gas, const TF* __restrict__ col_dry)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        if ( ( icol < ncol) && (ilay < nlay) )
        {
            const int idx_in = icol + ilay*ncol;
            const int idx_out = icol + ilay*ncol;
            if (igas > 0)
            {
                if (dim1 == 1 && dim2 == 1)
                { 
                     vmr_out[idx_out] = vmr_in[0];
                }
                else if (dim1 == 1)
                {
                     vmr_out[idx_out] = vmr_in[ilay];
                }
                else
                {
                    vmr_out[idx_out] = vmr_in[idx_in];
                }
                col_gas[idx_out] = vmr_out[idx_out] * col_dry[idx_in];
            }
            else if (igas == 0)
            {
                col_gas[idx_out] = col_dry[idx_in];
            }
        }
    }

    template<typename TF>__global__
    void reorder12x21_kernel(
            const int ni, const int nj,
            const TF* __restrict__ arr_in, TF* __restrict__ arr_out)
    {
        const int ii = blockIdx.x*blockDim.x + threadIdx.x;
        const int ij = blockIdx.y*blockDim.y + threadIdx.y;
        if ( (ii < ni) && (ij < nj) )
        {
            const int idx_out = ii + ij*ni;
            const int idx_in = ij + ii*nj;
            arr_out[idx_out] = arr_in[idx_in];
        }
    }

    template<typename TF>__global__
    void reorder123x321_kernel(
            const int ni, const int nj, const int nk,
            const TF* __restrict__ arr_in, TF* __restrict__ arr_out)
    {
        const int ii = blockIdx.x*blockDim.x + threadIdx.x;
        const int ij = blockIdx.y*blockDim.y + threadIdx.y;
        const int ik = blockIdx.z*blockDim.z + threadIdx.z;
        if ( (ii < ni) && (ij < nj) && (ik < nk))
        {
            const int idx_out = ii + ij*ni + ik*nj*ni;
            const int idx_in = ik + ij*nk + ii*nj*nk;
            arr_out[idx_out] = arr_in[idx_in];
        }
    }

    template<typename TF>__global__
    void zero_array_kernel(
            const int ni, const int nj, const int nk,
            TF* __restrict__ arr)
    {
        const int ii = blockIdx.x*blockDim.x + threadIdx.x;
        const int ij = blockIdx.y*blockDim.y + threadIdx.y;
        const int ik = blockIdx.z*blockDim.z + threadIdx.z;
        if ( (ii < ni) && (ij < nj) && (ik < nk))
        {
            const int idx = ii + ij*ni + ik*nj*ni;
            arr[idx] = TF(0.);
        }
    }

    template<typename TF>__global__
    void Planck_source_kernel(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int nPlanckTemp,
            const TF* __restrict__ tlay, const TF* __restrict__ tlev,
            const TF* __restrict__ tsfc,
            const int sfc_lay,
            const TF* __restrict__ fmajor, const int* __restrict__ jeta,
            const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
            const int* __restrict__ jpress, const int* __restrict__ gpoint_bands,
            const int* __restrict__ band_lims_gpt, const TF* __restrict__ pfracin,
            const TF temp_ref_min, const TF totplnk_delta,
            const TF* __restrict__ totplnk, const int* __restrict__ gpoint_flavor,
            const TF* __restrict__ ones, const TF delta_Tsurf,
            TF* __restrict__ sfc_src, TF* __restrict__ lay_src,
            TF* __restrict__ lev_src_inc, TF* __restrict__ lev_src_dec,
            TF* __restrict__ sfc_src_jac, TF* __restrict__ pfrac)
    {
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nband))
        {
            const int idx_collay = icol + ilay * ncol;
            const int itropo = !tropo[idx_collay];
            const int gpt_start = band_lims_gpt[2 * ibnd] - 1;
            const int gpt_end = band_lims_gpt[2 * ibnd + 1];
            const int iflav = gpoint_flavor[itropo + 2 * gpt_start] - 1;
            const int idx_fcl3 = 2 * 2 * 2* (iflav + icol * nflav + ilay * ncol * nflav);
            const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
            const int idx_tau = gpt_start + ilay * ngpt + icol * nlay * ngpt;
            
            //major gases//
            interpolate3D_byflav_kernel(ones, &fmajor[idx_fcl3],
                                        &pfracin[gpt_start], gpt_start, gpt_end,
                                        &jeta[idx_fcl1], jtemp[idx_collay],
                                        jpress[idx_collay]+itropo, ngpt, neta, npres+1,
                                        &pfrac[idx_tau]);

            // compute surface source irradiances
            if (ilay == 0)
            {
                const TF planck_function_sfc1 = interpolate1D(tsfc[icol],               temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);
                const TF planck_function_sfc2 = interpolate1D(tsfc[icol] + delta_Tsurf, temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);                 for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
                {
                    const int idx_in  = igpt + ilay*ngpt + icol*nlay*ngpt;
                    const int idx_out = igpt + icol*ngpt;
                    sfc_src[idx_out] = pfrac[idx_in] * planck_function_sfc1;
                    sfc_src_jac[idx_out] = pfrac[idx_in] * (planck_function_sfc2 - planck_function_sfc1);
                }   
            }    
            
            // compute layer source irradiances.
            const int idx_tmp = icol + ilay*ncol;
            const TF planck_function_lay = interpolate1D(tlay[idx_tmp], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]); 
            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_inout  = igpt + ilay*ngpt + icol*nlay*ngpt;
                lay_src[idx_inout] = pfrac[idx_inout] * planck_function_lay;
            }   

            // compute level source irradiances.
            const int idx_tmp1 = icol + (ilay+1)*ncol;
            const int idx_tmp2 = icol + ilay*ncol;
            const TF planck_function_lev1 = interpolate1D(tlev[idx_tmp1], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]); 
            const TF planck_function_lev2 = interpolate1D(tlev[idx_tmp2], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]); 
            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_inout  = igpt + ilay*ngpt + icol*nlay*ngpt;
                lev_src_inc[idx_inout] = pfrac[idx_inout] * planck_function_lev1;
                lev_src_dec[idx_inout] = pfrac[idx_inout] * planck_function_lev2;
            }   
        }
    }

    template<typename TF>__global__
    void interpolation_kernel(
            const int ncol, const int nlay, const int ngas, const int nflav,
            const int neta, const int npres, const int ntemp, const TF tmin,
            const int* __restrict__ flavor,
            const TF* __restrict__ press_ref_log,
            const TF* __restrict__ temp_ref,
            TF press_ref_log_delta,
            TF temp_ref_min,
            TF temp_ref_delta,
            TF press_ref_trop_log,
            const TF* __restrict__ vmr_ref,
            const TF* __restrict__ play,
            const TF* __restrict__ tlay,
            TF* __restrict__ col_gas,
            int* __restrict__ jtemp,
            TF* __restrict__ fmajor, TF* __restrict__ fminor,
            TF* __restrict__ col_mix,
            BOOL_TYPE* __restrict__ tropo,
            int* __restrict__ jeta,
            int* __restrict__ jpress)
    {
        const int ilay = blockIdx.x*blockDim.x + threadIdx.x;
        const int icol = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (ilay < nlay) )
        {
            const int idx = icol + ilay*ncol;

            jtemp[idx] = int((tlay[idx] - (temp_ref_min-temp_ref_delta)) / temp_ref_delta);
            jtemp[idx] = min(ntemp-1, max(1, jtemp[idx]));
            const TF ftemp = (tlay[idx] - temp_ref[jtemp[idx]-1]) / temp_ref_delta;

            const TF locpress = TF(1.) + (log(play[idx]) - press_ref_log[0]) / press_ref_log_delta;
            jpress[idx] = min(npres-1, max(1, int(locpress)));
            const TF fpress = locpress - TF(jpress[idx]);

            tropo[idx] = log(play[idx]) > press_ref_trop_log;
            const int itropo = !tropo[idx];

            for (int iflav=0; iflav<nflav; ++iflav)
            {
                const int gas1 = flavor[2*iflav];
                const int gas2 = flavor[2*iflav+1];
                for (int itemp=0; itemp<2; ++itemp)
                {
                    const int vmr_base_idx = itropo + (jtemp[idx]+itemp-1) * (ngas+1) * 2;
                    const int colmix_idx = itemp + 2*(iflav + nflav*icol + nflav*ncol*ilay);
                    const int colgas1_idx = icol + ilay*ncol + gas1*nlay*ncol;
                    const int colgas2_idx = icol + ilay*ncol + gas2*nlay*ncol;
                    TF eta;
                    const TF ratio_eta_half = vmr_ref[vmr_base_idx + 2 * gas1] /
                                              vmr_ref[vmr_base_idx + 2 * gas2];
                    col_mix[colmix_idx] = col_gas[colgas1_idx] + ratio_eta_half * col_gas[colgas2_idx];
                    if (col_mix[colmix_idx] > TF(2.)*tmin)
                    {
                        eta = col_gas[colgas1_idx] / col_mix[colmix_idx];
                    } else
                    {
                        eta = TF(0.5);
                    }
                    const TF loceta = eta * TF(neta-1);
                    jeta[colmix_idx] = min(int(loceta)+1, neta-1);
                    const TF feta = fmod(loceta, TF(1.));
                    const TF ftemp_term  = TF(1-itemp) + TF(2*itemp-1)*ftemp;
                    // compute interpolation fractions needed for minot species
                    const int fminor_idx = 2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
                    fminor[fminor_idx] = (TF(1.0)-feta) * ftemp_term;
                    fminor[fminor_idx+1] = feta * ftemp_term;
                    // compute interpolation fractions needed for major species
                    const int fmajor_idx = 2*2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
                    fmajor[fmajor_idx] = (TF(1.0)-fpress) * fminor[fminor_idx];
                    fmajor[fmajor_idx+1] = (TF(1.0)-fpress) * fminor[fminor_idx+1];
                    fmajor[fmajor_idx+2] = fpress * fminor[fminor_idx];
                    fmajor[fmajor_idx+3] = fpress * fminor[fminor_idx+1];

                }
            }
        }
    }

    template<typename TF>__global__
    void compute_tau_major_absorption_kernel(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int* __restrict__ gpoint_flavor,
            const int* __restrict__ band_lims_gpt,
            const TF* __restrict__ kmajor,
            const TF* __restrict__ col_mix, const TF* __restrict__ fmajor,
            const int* __restrict__ jeta, const BOOL_TYPE* __restrict__ tropo,
            const int* __restrict__ jtemp, const int* __restrict__ jpress,
            TF* __restrict__ tau, TF* __restrict__ tau_major)
    {
        // Fetch the three coordinates.
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nband) ) {
            const int idx_collay = icol + ilay * ncol;
            const int itropo = !tropo[idx_collay];
            const int gpt_start = band_lims_gpt[2 * ibnd] - 1;
            const int gpt_end = band_lims_gpt[2 * ibnd + 1];
            const int iflav = gpoint_flavor[itropo + 2 * gpt_start] - 1;
            const int idx_fcl3 = 2 * 2 * 2* (iflav + icol * nflav + ilay * ncol * nflav);
            const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
            const int idx_tau = gpt_start + ilay * ngpt + icol * nlay * ngpt;

            //major gases//
            interpolate3D_byflav_kernel(&col_mix[idx_fcl1], &fmajor[idx_fcl3],
                                        &kmajor[gpt_start], gpt_start, gpt_end,
                                        &jeta[idx_fcl1], jtemp[idx_collay], 
                                        jpress[idx_collay]+itropo, ngpt, neta, npres+1,
                                        &tau_major[idx_tau]);
            
            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
                tau[idx_out] = tau_major[idx_out];
                //should be '+=' later on, but we first need the zero_arrays for that
            }
        }
    }

    template<typename TF>__global__
    void compute_tau_minor_absorption_kernel(
            const int ncol, const int nlay, const int ngpt,
            const int ngas, const int nflav, const int ntemp, const int neta,
            const int nscale_lower,
            const int nscale_upper,
            const int nminor_lower,
            const int nminor_upper,
            const int nminork_lower,
            const int nminork_upper,
            const int idx_h2o,
            const int* __restrict__ gpoint_flavor,
            const TF* __restrict__ kminor_lower,
            const TF* __restrict__ kminor_upper,
            const int* __restrict__ minor_limits_gpt_lower,
            const int* __restrict__ minor_limits_gpt_upper,
            const BOOL_TYPE* __restrict__ minor_scales_with_density_lower,
            const BOOL_TYPE* __restrict__ minor_scales_with_density_upper,
            const BOOL_TYPE* __restrict__ scale_by_complement_lower,
            const BOOL_TYPE* __restrict__ scale_by_complement_upper,
            const int* __restrict__ idx_minor_lower,
            const int* __restrict__ idx_minor_upper,
            const int* __restrict__ idx_minor_scaling_lower,
            const int* __restrict__ idx_minor_scaling_upper,
            const int* __restrict__ kminor_start_lower,
            const int* __restrict__ kminor_start_upper,
            const TF* __restrict__ play,
            const TF* __restrict__ tlay,
            const TF* __restrict__ col_gas,
            const TF* __restrict__ fminor,
            const int* __restrict__ jeta,
            const int* __restrict__ jtemp,
            const BOOL_TYPE* __restrict__ tropo,
            TF* __restrict__ tau,
            TF* __restrict__ tau_minor) 
    {
        // Fetch the three coordinates.
        const int ilay = blockIdx.x * blockDim.x + threadIdx.x;
        const int icol = blockIdx.y * blockDim.y + threadIdx.y;
        const TF PaTohPa = 0.01;
        const int ncl = ncol * nlay;
        if ((icol < ncol) && (ilay < nlay)) 
        {
            //kernel implementation
            const int idx_collay = icol + ilay * ncol;
            const int idx_collaywv = icol + ilay * ncol + idx_h2o * ncl;

            if (tropo[idx_collay] == 1) 
            {
                for (int imnr = 0; imnr < nscale_lower; ++imnr)
                {
                    TF scaling = col_gas[idx_collay + idx_minor_lower[imnr] * ncl];
                    if (minor_scales_with_density_lower[imnr])
                    {
                        scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];
                        if (idx_minor_scaling_lower[imnr] > 0)
                        {
                            TF vmr_fact = TF(1.) / col_gas[idx_collay];
                            TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);
                            if (scale_by_complement_lower[imnr])
                            {
                                scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling_lower[imnr] * ncl] * vmr_fact * dry_fact);
                            } 
                            else
                            {
                                scaling *= col_gas[idx_collay + idx_minor_scaling_lower[imnr] * ncl] * vmr_fact * dry_fact;
                            }
                        }
                    }
                    const int gpt_start = minor_limits_gpt_lower[2*imnr]-1;
                    const int gpt_end = minor_limits_gpt_lower[2*imnr+1];
                    const int iflav = gpoint_flavor[2*gpt_start]-1;
                    const int idx_fcl2 = 2 * 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                    interpolate2D_byflav_kernel(&fminor[idx_fcl2], &kminor_lower[kminor_start_lower[imnr]-1],
                                                kminor_start_lower[imnr]-1, kminor_start_lower[imnr]-1 + (gpt_end - gpt_start),
                                                &tau_minor[idx_tau], &jeta[idx_fcl1],
                                                jtemp[idx_collay], nminork_lower, neta);

                    for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                    {
                        const int idx_out = igpt + ilay * ngpt + icol * nlay * ngpt;
                        tau[idx_out] += tau_minor[idx_out] * scaling;
                    }
                }
            }
            else
            {
                for (int imnr = 0; imnr < nscale_upper; ++imnr)
                {
                    TF scaling = col_gas[idx_collay + idx_minor_upper[imnr] * ncl];
                    if (minor_scales_with_density_upper[imnr])
                    {
                        scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];
                        if (idx_minor_scaling_upper[imnr] > 0)
                        {
                            TF vmr_fact = TF(1.) / col_gas[idx_collay];
                            TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);
                            if (scale_by_complement_upper[imnr])
                            {
                                scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling_upper[imnr] * ncl] * vmr_fact * dry_fact);
                            }
                            else
                            {
                                scaling *= col_gas[idx_collay + idx_minor_scaling_upper[imnr] * ncl] * vmr_fact * dry_fact;
                            }
                        }
                    }
                    const int gpt_start = minor_limits_gpt_upper[2*imnr]-1;
                    const int gpt_end = minor_limits_gpt_upper[2*imnr+1];
                    const int iflav = gpoint_flavor[2*gpt_start+1]-1;
                    const int idx_fcl2 = 2 * 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                    const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                    interpolate2D_byflav_kernel(&fminor[idx_fcl2], &kminor_upper[kminor_start_upper[imnr]-1],
                                                kminor_start_upper[imnr]-1, kminor_start_upper[imnr]-1 + (gpt_end - gpt_start),
                                                &tau_minor[idx_tau], &jeta[idx_fcl1],
                                                jtemp[idx_collay], nminork_upper, neta);

                    for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                    {
                        const int idx_out = igpt + ilay * ngpt + icol * nlay * ngpt;
                        tau[idx_out] += tau_minor[idx_out] * scaling;
                    }
                }
            }
        }
    }

    template<typename TF>__global__
    void compute_tau_rayleigh_kernel(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* __restrict__ gpoint_flavor,
            const int* __restrict__ band_lims_gpt,
            const TF* __restrict__ krayl,
            int idx_h2o, const TF* __restrict__ col_dry, const TF* __restrict__ col_gas,
            const TF* __restrict__ fminor, const int* __restrict__ jeta,
            const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
            TF* __restrict__ tau_rayleigh, TF* __restrict__ k)
    {
        // Fetch the three coordinates.
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nbnd) )
        {
            //kernel implementation
            const int idx_collay = icol + ilay*ncol;
            const int idx_collaywv = icol + ilay*ncol + idx_h2o*nlay*ncol;
            const int itropo = !tropo[idx_collay];
            const int gpt_start = band_lims_gpt[2*ibnd]-1;
            const int gpt_end = band_lims_gpt[2*ibnd+1];
            const int iflav = gpoint_flavor[itropo+2*gpt_start]-1;
            const int idx_fcl2 = 2*2*(iflav + icol*nflav + ilay*ncol*nflav);
            const int idx_fcl1   = 2*(iflav + icol*nflav + ilay*ncol*nflav);
            const int idx_krayl  = gpt_start + ngpt*neta*ntemp*itropo;
            const int idx_k = gpt_start + ilay*ngpt + icol*nlay*ngpt;
            interpolate2D_byflav_kernel(&fminor[idx_fcl2],
                                        &krayl[idx_krayl],
                                        gpt_start, gpt_end, &k[idx_k],
                                        &jeta[idx_fcl1],
                                        jtemp[idx_collay],
                                        ngpt, neta);

            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
                tau_rayleigh[idx_out] = k[idx_k+igpt-gpt_start]*(col_gas[idx_collaywv]+col_dry[idx_collay]);
            }
        }
    }

    
    template<typename TF>__global__
    void combine_and_reorder_2str_kernel(
            const int ncol, const int nlay, const int ngpt, const TF tmin,
            const TF* __restrict__ tau_abs, const TF* __restrict__ tau_rayleigh,
            TF* __restrict__ tau, TF* __restrict__ ssa, TF* __restrict__ g)
    {
        // Fetch the three coordinates.
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
        const int ilay = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
        {
            const int idx_in  = igpt + ilay*ngpt + icol*(ngpt*nlay);
            const int idx_out = icol + ilay*ncol + igpt*(ncol*nlay);
	   
            const TF tau_tot = tau_abs[idx_in] + tau_rayleigh[idx_in];
            tau[idx_out] = tau_tot;
            g  [idx_out] = TF(0.);
            if (tau_tot>(TF(2.)*tmin))
                ssa[idx_out] = tau_rayleigh[idx_in]/tau_tot;
            else
                ssa[idx_out] = 0.;
        }
    }
    
    
    
}

namespace rrtmgp_kernel_launcher_cuda
{
    template<typename TF>
    void fill_gases(
            const int ncol, const int nlay, const int ngas, 
            Array<TF,3>& vmr_out, 
            Array<TF,3>& col_gas, const Array<TF,2>& col_dry,
            const Gas_concs<TF>& gas_desc, const Array<std::string,1>& gas_names)
    {
        const int arr_in_size = col_dry.size() * sizeof(TF);
        const int vmr_out_size = vmr_out.size() * sizeof(TF);
        const int gas_out_size = col_gas.size() * sizeof(TF);
        int dim1;
        int dim2;
        TF* vmr_out_gpu;
        TF* vmr_in_gpu;
        TF* col_gas_gpu;
        TF* col_dry_gpu;

        cuda_safe_call(hipMalloc((void **) &vmr_out_gpu, vmr_out_size));
        cuda_safe_call(hipMalloc((void **) &vmr_in_gpu, arr_in_size));
        cuda_safe_call(hipMalloc((void **) &col_gas_gpu, gas_out_size));
        cuda_safe_call(hipMalloc((void **) &col_dry_gpu, arr_in_size));

        cuda_safe_call(hipMemcpy(col_dry_gpu, col_dry.ptr(), arr_in_size, hipMemcpyHostToDevice));
        
        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        const int block_col = 16;
        const int block_lay = 16;

        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);

        dim3 grid_gpu(grid_col, grid_lay);
        dim3 block_gpu(block_col, block_lay);
        std::cout<<ngas<<std::endl;
        for (int igas = 0; igas<=ngas; ++igas)
        {
            if (igas > 0)
            { 
                std::cout<<"#gas" <<igas<<std::endl;
                cuda_safe_call(hipMemcpy(vmr_in_gpu, gas_desc.get_vmr(gas_names({igas})).ptr(), arr_in_size, hipMemcpyHostToDevice));
                dim1 = gas_desc.get_vmr(gas_names({igas})).dim(1);
                dim2 = gas_desc.get_vmr(gas_names({igas})).dim(2);
            }
            else
            {
                dim1 = 1;
                dim2 = 1;
            }
            std::cout<<"#gas" <<igas<<std::endl;
            fill_gases_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngas, igas, dim1, dim2, 
                vmr_out_gpu, vmr_in_gpu,
                col_gas_gpu, col_dry_gpu);
            std::cout<<"#gas" <<igas<<std::endl;
            if (igas > 0)
                cuda_safe_call(hipMemcpy(&vmr_out.v()[(igas-1)*ncol*nlay],vmr_out_gpu, vmr_out_size, hipMemcpyDeviceToHost));
            cuda_safe_call(hipMemcpy(&col_gas.v()[igas*ncol*nlay],col_gas_gpu, gas_out_size, hipMemcpyDeviceToHost));
            std::cout<<"#gas" <<igas<<std::endl;
            cuda_check_error();
            cuda_safe_call(hipDeviceSynchronize());
        }
        std::cout<<"#gas" <<std::endl;

        //cuda_check_error();
        //cuda_safe_call(hipDeviceSynchronize());
        std::cout<<"#gas" <<std::endl;
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU fill gases: "<<elapsedtime<<" (ms)"<<std::endl;


        cuda_safe_call(hipFree(vmr_out_gpu));
        cuda_safe_call(hipFree(vmr_in_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(col_dry_gpu));
    }

    template<typename TF>
    void reorder123x321(const int ni, const int nj, const int nk,
                        const Array<TF,3>& arr_in, Array<TF,3>& arr_out)
    {
        const int arr_size = arr_in.size() * sizeof(TF);
        TF* arr_in_gpu;
        TF* arr_out_gpu;
        cuda_safe_call(hipMalloc((void **) &arr_in_gpu, arr_size));
        cuda_safe_call(hipMalloc((void **) &arr_out_gpu, arr_size));

        cuda_safe_call(hipMemcpy(arr_in_gpu, arr_in.ptr(), arr_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        const int block_i = 32;
        const int block_j = 16;
        const int block_k = 1;

        const int grid_i  = ni/block_i + (ni%block_i > 0);
        const int grid_j  = nj/block_j + (nj%block_j > 0);
        const int grid_k  = nk/block_k + (nk%block_k > 0);

        dim3 grid_gpu(grid_i, grid_j, grid_k);
        dim3 block_gpu(block_i, block_j, block_k);

        reorder123x321_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, nk, arr_in_gpu, arr_out_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU reorder123x321: "<<elapsedtime<<" (ms)"<<std::endl;

        cuda_safe_call(hipMemcpy(arr_out.ptr(), arr_out_gpu, arr_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipFree(arr_in_gpu));
        cuda_safe_call(hipFree(arr_out_gpu));
    }

    template<typename TF>
    void reorder12x21(const int ni, const int nj,
                        const Array<TF,2>& arr_in, Array<TF,2>& arr_out)
    {
        const int arr_size = arr_in.size() * sizeof(TF);
        TF* arr_in_gpu;
        TF* arr_out_gpu;
        cuda_safe_call(hipMalloc((void **) &arr_in_gpu, arr_size));
        cuda_safe_call(hipMalloc((void **) &arr_out_gpu, arr_size));

        cuda_safe_call(hipMemcpy(arr_in_gpu, arr_in.ptr(), arr_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        const int block_i = 32;
        const int block_j = 16;

        const int grid_i  = ni/block_i + (ni%block_i > 0);
        const int grid_j  = nj/block_j + (nj%block_j > 0);

        dim3 grid_gpu(grid_i, grid_j);
        dim3 block_gpu(block_i, block_j);

        reorder12x21_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, arr_in_gpu, arr_out_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU reorder12x21: "<<elapsedtime<<" (ms)"<<std::endl;

        cuda_safe_call(hipMemcpy(arr_out.ptr(), arr_out_gpu, arr_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipFree(arr_in_gpu));
        cuda_safe_call(hipFree(arr_out_gpu));
    }

    template<typename TF>
    void zero_array(const int ni, const int nj, const int nk, Array<TF,3>& arr)
    {
        const int arr_size = arr.size() * sizeof(TF);
        TF* arr_gpu;
        cuda_safe_call(hipMalloc((void **) &arr_gpu, arr_size));

        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        const int block_i = 32;
        const int block_j = 16;
        const int block_k = 1;

        const int grid_i  = ni/block_i + (ni%block_i > 0);
        const int grid_j  = nj/block_j + (nj%block_j > 0);
        const int grid_k  = nk/block_k + (nk%block_k > 0);

        dim3 grid_gpu(grid_i, grid_j, grid_k);
        dim3 block_gpu(block_i, block_j, block_k);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, nk, arr_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU zero_array: "<<elapsedtime<<" (ms)"<<std::endl;

        cuda_safe_call(hipMemcpy(arr.ptr(), arr_gpu, arr_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipFree(arr_gpu));
    }

    template<typename TF>
    void interpolation(
            const int ncol, const int nlay,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const Array<int,2>& flavor,
            const Array<TF,1>& press_ref_log,
            const Array<TF,1>& temp_ref,
            TF press_ref_log_delta,
            TF temp_ref_min,
            TF temp_ref_delta,
            TF press_ref_trop_log,
            const Array<TF,3>& vmr_ref,
            const Array<TF,2>& play,
            const Array<TF,2>& tlay,
            Array<TF,3>& col_gas,
            Array<int,2>& jtemp,
            Array<TF,6>& fmajor, Array<TF,5>& fminor,
            Array<TF,4>& col_mix,
            Array<BOOL_TYPE,2>& tropo,
            Array<int,4>& jeta,
            Array<int,2>& jpress)
    {
        const int flavor_size = flavor.size() * sizeof(int);
        const int press_ref_log_size = press_ref_log.size() * sizeof(TF);
        const int temp_ref_size = temp_ref.size() * sizeof(TF);
        const int vmr_ref_size = vmr_ref.size() * sizeof(TF);
        const int collay_tf_size = ncol * nlay * sizeof(TF);
        const int collay_int_size = ncol * nlay * sizeof(int);
        const int col_gas_size = col_gas.size() * sizeof(TF);
        const int jtemp_size = jtemp.size() * sizeof(int);
        const int fmajor_size = fmajor.size() * sizeof(TF);
        const int fminor_size = fminor.size() * sizeof(TF);
        const int col_mix_size = col_mix.size() * sizeof(TF);
        const int tropo_size = tropo.size() * sizeof(BOOL_TYPE);
        const int jeta_size = jeta.size() * sizeof(int);
        const int jpress_size = jpress.size() * sizeof(int);

        int* flavor_gpu;
        TF* press_ref_log_gpu;
        TF* temp_ref_gpu;
        TF* vmr_ref_gpu;
        TF* play_gpu;
        TF* tlay_gpu;
        TF* col_gas_gpu;
        int* jtemp_gpu;
        TF* fmajor_gpu;
        TF* fminor_gpu;
        TF* col_mix_gpu;
        BOOL_TYPE* tropo_gpu;
        int* jeta_gpu;
        int* jpress_gpu;

        cuda_safe_call(hipMalloc((void **) &flavor_gpu, flavor_size));
        cuda_safe_call(hipMalloc((void **) &press_ref_log_gpu, press_ref_log_size));
        cuda_safe_call(hipMalloc((void **) &temp_ref_gpu, temp_ref_size));
        cuda_safe_call(hipMalloc((void **) &vmr_ref_gpu, vmr_ref_size));
        cuda_safe_call(hipMalloc((void **) &play_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void **) &tlay_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void **) &col_gas_gpu, col_gas_size));
        cuda_safe_call(hipMalloc((void **) &jtemp_gpu, collay_int_size));
        cuda_safe_call(hipMalloc((void **) &fmajor_gpu, fmajor_size));
        cuda_safe_call(hipMalloc((void **) &fminor_gpu, fminor_size));
        cuda_safe_call(hipMalloc((void **) &col_mix_gpu, col_mix_size));
        cuda_safe_call(hipMalloc((void **) &tropo_gpu, tropo_size));
        cuda_safe_call(hipMalloc((void **) &jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void **) &jpress_gpu, collay_int_size));

        cuda_safe_call(hipMemcpy(flavor_gpu, flavor.ptr(), flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(press_ref_log_gpu, press_ref_log.ptr(), press_ref_log_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(temp_ref_gpu, temp_ref.ptr(), temp_ref_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(vmr_ref_gpu, vmr_ref.ptr(), vmr_ref_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(play_gpu, play.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tlay_gpu, tlay.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_gas_gpu, col_gas.ptr(), col_gas_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        const int block_lay = 16;
        const int block_col = 32;

        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);

        dim3 grid_gpu(grid_lay, grid_col);
        dim3 block_gpu(block_lay, block_col);

        TF tmin = std::numeric_limits<TF>::min();
        interpolation_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngas, nflav, neta, npres, ntemp, tmin,
                flavor_gpu, press_ref_log_gpu, temp_ref_gpu,
                press_ref_log_delta, temp_ref_min,
                temp_ref_delta, press_ref_trop_log,
                vmr_ref_gpu, play_gpu, tlay_gpu,
                col_gas_gpu, jtemp_gpu, fmajor_gpu,
                fminor_gpu, col_mix_gpu, tropo_gpu,
                jeta_gpu, jpress_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU interpolation: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(jtemp.ptr(), jtemp_gpu, jtemp_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(fmajor.ptr(), fmajor_gpu, fmajor_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(fminor.ptr(), fminor_gpu, fminor_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(col_mix.ptr(), col_mix_gpu, col_mix_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(tropo.ptr(), tropo_gpu, tropo_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(jeta.ptr(), jeta_gpu, jeta_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(jpress.ptr(), jpress_gpu, jpress_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(flavor_gpu));
        cuda_safe_call(hipFree(press_ref_log_gpu));
        cuda_safe_call(hipFree(temp_ref_gpu));
        cuda_safe_call(hipFree(vmr_ref_gpu));
        cuda_safe_call(hipFree(play_gpu));
        cuda_safe_call(hipFree(tlay_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(fmajor_gpu));
        cuda_safe_call(hipFree(fminor_gpu));
        cuda_safe_call(hipFree(col_mix_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(jpress_gpu));
    }

    template<typename TF>
    void combine_and_reorder_2str(
            const int ncol, const int nlay, const int ngpt,
            const Array<TF,3>& tau_abs, const Array<TF,3>& tau_rayleigh,
            Array<TF,3>& tau, Array<TF,3>& ssa, Array<TF,3>& g)
    {
        // Store the sizes, all the same
        const int array_size = tau_abs.size()*sizeof(TF);

        TF* tau_abs_gpu;
        TF* tau_rayleigh_gpu;
        TF* tau_gpu;
        TF* ssa_gpu;
        TF* g_gpu;

        TF tmin = std::numeric_limits<TF>::min();
        // Allocate a CUDA array.
        cuda_safe_call(hipMalloc((void**)&tau_abs_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&tau_rayleigh_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&tau_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&ssa_gpu, array_size));
        cuda_safe_call(hipMalloc((void**)&g_gpu, array_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(tau_abs_gpu, tau_abs.ptr(), array_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tau_rayleigh_gpu, tau_rayleigh.ptr(), array_size, hipMemcpyHostToDevice));
        hipEvent_t startEvent, stopEvent;
        float elapsedtime;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);

        hipEventRecord(startEvent, 0);
        // Call the kernel.
        const int block_col = 32;
        const int block_gpt = 32;
        const int block_lay = 1;

        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt  = ngpt/block_gpt + (ngpt%block_gpt > 0);
        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);

        dim3 grid_gpu(grid_col, grid_gpt, grid_lay);
        dim3 block_gpu(block_col, block_gpt, block_lay);

        combine_and_reorder_2str_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngpt, tmin,
                tau_abs_gpu, tau_rayleigh_gpu,
                tau_gpu, ssa_gpu, g_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU combine_and_reorder_2str: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(tau.ptr(), tau_gpu, array_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(ssa.ptr(), ssa_gpu, array_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(g.ptr(), g_gpu, array_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(tau_abs_gpu));
        cuda_safe_call(hipFree(tau_rayleigh_gpu));
        cuda_safe_call(hipFree(tau_gpu));
        cuda_safe_call(hipFree(ssa_gpu));
        cuda_safe_call(hipFree(g_gpu));
    }
    
    template<typename TF>
    void compute_tau_rayleigh(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const Array<int,2>& gpoint_flavor,
            const Array<int,2>& band_lims_gpt,
            const Array<TF,4>& krayl,
            int idx_h2o, const Array<TF,2>& col_dry, const Array<TF,3>& col_gas,
            const Array<TF,5>& fminor, const Array<int,4>& jeta,
            const Array<BOOL_TYPE,2>& tropo, const Array<int,2>& jtemp,
            Array<TF,3>& tau_rayleigh)
    {
        float elapsedtime;
        const int gpoint_flavor_size = gpoint_flavor.size()*sizeof(int);
        const int band_lims_gpt_size = band_lims_gpt.size()*sizeof(int);
        const int krayl_size = krayl.size()*sizeof(TF);
        const int col_dry_size = col_dry.size()*sizeof(TF);
        const int col_gas_size = col_gas.size()*sizeof(TF);
        const int fminor_size = fminor.size()*sizeof(TF);
        const int jeta_size = jeta.size()*sizeof(int);
        const int tropo_size = tropo.size()*sizeof(BOOL_TYPE);
        const int jtemp_size = jtemp.size()*sizeof(int);
        const int tau_rayleigh_size = tau_rayleigh.size()*sizeof(TF);

        int* gpoint_flavor_gpu;
        int* band_lims_gpt_gpu;
        int* jeta_gpu;
        int* jtemp_gpu;
        BOOL_TYPE* tropo_gpu;
        TF* krayl_gpu;
        TF* col_dry_gpu;
        TF* col_gas_gpu;
        TF* fminor_gpu;
        TF* tau_rayleigh_gpu;
        TF* k_gpu;

        // Allocate a CUDA array.
        cuda_safe_call(hipMalloc((void**)&gpoint_flavor_gpu, gpoint_flavor_size));
        cuda_safe_call(hipMalloc((void**)&band_lims_gpt_gpu, band_lims_gpt_size));
        cuda_safe_call(hipMalloc((void**)&krayl_gpu, krayl_size));
        cuda_safe_call(hipMalloc((void**)&col_dry_gpu, col_dry_size));
        cuda_safe_call(hipMalloc((void**)&col_gas_gpu, col_gas_size));
        cuda_safe_call(hipMalloc((void**)&fminor_gpu, fminor_size));
        cuda_safe_call(hipMalloc((void**)&jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void**)&tropo_gpu, tropo_size));
        cuda_safe_call(hipMalloc((void**)&jtemp_gpu, jtemp_size));
        cuda_safe_call(hipMalloc((void**)&tau_rayleigh_gpu, tau_rayleigh_size));
        cuda_safe_call(hipMalloc((void**)&k_gpu, tau_rayleigh_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(gpoint_flavor_gpu, gpoint_flavor.ptr(), gpoint_flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(band_lims_gpt_gpu, band_lims_gpt.ptr(), band_lims_gpt_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(krayl_gpu, krayl.ptr(), krayl_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_dry_gpu, col_dry.ptr(), col_dry_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_gas_gpu, col_gas.ptr(), col_gas_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fminor_gpu, fminor.ptr(), fminor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jeta_gpu, jeta.ptr(), jeta_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tropo_gpu, tropo.ptr(), tropo_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jtemp_gpu, jtemp.ptr(), jtemp_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        // Call the kernel.
        const int block_bnd = 14;
        const int block_lay = 1;
        const int block_col = 32;

        const int grid_bnd  = nbnd/block_bnd + (nbnd%block_bnd > 0);
        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);

        dim3 grid_gpu(grid_bnd, grid_lay, grid_col);
        dim3 block_gpu(block_bnd, block_lay, block_col);

        compute_tau_rayleigh_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                gpoint_flavor_gpu,
                band_lims_gpt_gpu,
                krayl_gpu,
                idx_h2o, col_dry_gpu, col_gas_gpu,
                fminor_gpu, jeta_gpu,
                tropo_gpu, jtemp_gpu,
                tau_rayleigh_gpu, k_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU compute_tau_rayleigh: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(tau_rayleigh.ptr(), tau_rayleigh_gpu, tau_rayleigh_size, hipMemcpyDeviceToHost));
        
        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(gpoint_flavor_gpu));
        cuda_safe_call(hipFree(band_lims_gpt_gpu));
        cuda_safe_call(hipFree(krayl_gpu));
        cuda_safe_call(hipFree(col_dry_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(fminor_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(tau_rayleigh_gpu));
        cuda_safe_call(hipFree(k_gpu));
    }

    template<typename TF>
    void compute_tau_absorption(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int nminorlower, const int nminorklower,
            const int nminorupper, const int nminorkupper,
            const int idx_h2o,
            const Array<int,2>& gpoint_flavor,
            const Array<int,2>& band_lims_gpt,
            const Array<TF,4>& kmajor,
            const Array<TF,3>& kminor_lower,
            const Array<TF,3>& kminor_upper,
            const Array<int,2>& minor_limits_gpt_lower,
            const Array<int,2>& minor_limits_gpt_upper,
            const Array<BOOL_TYPE,1>& minor_scales_with_density_lower,
            const Array<BOOL_TYPE,1>& minor_scales_with_density_upper,
            const Array<BOOL_TYPE,1>& scale_by_complement_lower,
            const Array<BOOL_TYPE,1>& scale_by_complement_upper,
            const Array<int,1>& idx_minor_lower,
            const Array<int,1>& idx_minor_upper,
            const Array<int,1>& idx_minor_scaling_lower,
            const Array<int,1>& idx_minor_scaling_upper,
            const Array<int,1>& kminor_start_lower,
            const Array<int,1>& kminor_start_upper,
            const Array<BOOL_TYPE,2>& tropo,
            const Array<TF,4>& col_mix, const Array<TF,6>& fmajor,
            const Array<TF,5>& fminor, const Array<TF,2>& play,
            const Array<TF,2>& tlay, const Array<TF,3>& col_gas,
            const Array<int,4>& jeta, const Array<int,2>& jtemp,
            const Array<int,2>& jpress, Array<TF,3>& tau)
    {
        float elapsedtime;
        const int gpoint_flavor_size = gpoint_flavor.size()*sizeof(int);
        const int band_lims_gpt_size = band_lims_gpt.size()*sizeof(int);
        const int kmajor_size = kmajor.size()*sizeof(TF);
        const int kminor_lower_size = kminor_lower.size()*sizeof(TF);
        const int kminor_upper_size = kminor_upper.size()*sizeof(TF);
        const int minor_limits_gpt_lower_size = minor_limits_gpt_lower.size()*sizeof(int);
        const int minor_limits_gpt_upper_size = minor_limits_gpt_upper.size()*sizeof(int);
        const int nminorlower_bool_size = nminorlower*sizeof(BOOL_TYPE); //minor scales with/scale by complement
        const int nminorupper_bool_size = nminorlower*sizeof(BOOL_TYPE); //minor scales with/scale by complement
        const int nminorlower_int_size = nminorlower*sizeof(int); //idx_minor(scaling) kminor
        const int nminorupper_int_size = nminorlower*sizeof(int);
        const int tropo_size = tropo.size()*sizeof(BOOL_TYPE);
        const int col_mix_size = col_mix.size()*sizeof(TF);
        const int fmajor_size = fmajor.size()*sizeof(TF);
        const int fminor_size = fminor.size()*sizeof(TF);
        const int collay_tf_size = ncol*nlay*sizeof(TF); //tlay,play
        const int col_gas_size = col_gas.size()*sizeof(TF);
        const int jeta_size =  jeta.size()*sizeof(int);
        const int collay_int_size = ncol*nlay*sizeof(int);
        const int itropo_size = 2*ncol*sizeof(int);
        const int tau_size = tau.size()*sizeof(TF);
        
        int* gpoint_flavor_gpu;
        int* band_lims_gpt_gpu;
        TF* kmajor_gpu;
        TF* kminor_lower_gpu;
        TF* kminor_upper_gpu;
        int* minor_limits_gpt_lower_gpu;
        int* minor_limits_gpt_upper_gpu;
        BOOL_TYPE* minor_scales_with_density_lower_gpu;
        BOOL_TYPE* minor_scales_with_density_upper_gpu;
        BOOL_TYPE* scale_by_complement_lower_gpu;
        BOOL_TYPE* scale_by_complement_upper_gpu;
        int* idx_minor_lower_gpu;
        int* idx_minor_upper_gpu;
        int* idx_minor_scaling_lower_gpu;
        int* idx_minor_scaling_upper_gpu;
        int* kminor_start_lower_gpu;
        int* kminor_start_upper_gpu;
        BOOL_TYPE* tropo_gpu;
        TF* col_mix_gpu;
        TF* fmajor_gpu;
        TF* fminor_gpu;
        TF* play_gpu;
        TF* tlay_gpu;
        TF* col_gas_gpu;
        int* jeta_gpu;
        int* jtemp_gpu;
        int* jpress_gpu;
        int* itropo_lower_gpu;
        int* itropo_upper_gpu;
        TF* tau_gpu;
        TF* tau_major_gpu;
        TF* tau_minor_gpu;

        // Allocate a CUDA array.
        cuda_safe_call(hipMalloc((void**)& gpoint_flavor_gpu, gpoint_flavor_size));
        cuda_safe_call(hipMalloc((void**)& band_lims_gpt_gpu, band_lims_gpt_size));
        cuda_safe_call(hipMalloc((void**)& kmajor_gpu, kmajor_size));
        cuda_safe_call(hipMalloc((void**)& kminor_lower_gpu, kminor_lower_size));
        cuda_safe_call(hipMalloc((void**)& kminor_upper_gpu, kminor_upper_size));
        cuda_safe_call(hipMalloc((void**)& minor_limits_gpt_lower_gpu, minor_limits_gpt_lower_size));
        cuda_safe_call(hipMalloc((void**)& minor_limits_gpt_upper_gpu, minor_limits_gpt_upper_size));
        cuda_safe_call(hipMalloc((void**)& minor_scales_with_density_lower_gpu, nminorlower_bool_size));
        cuda_safe_call(hipMalloc((void**)& minor_scales_with_density_upper_gpu, nminorupper_bool_size));
        cuda_safe_call(hipMalloc((void**)& scale_by_complement_lower_gpu, nminorlower_bool_size));
        cuda_safe_call(hipMalloc((void**)& scale_by_complement_upper_gpu, nminorupper_bool_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_lower_gpu, nminorlower_int_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_upper_gpu, nminorupper_int_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_scaling_lower_gpu, nminorlower_int_size));
        cuda_safe_call(hipMalloc((void**)& idx_minor_scaling_upper_gpu, nminorupper_int_size));
        cuda_safe_call(hipMalloc((void**)& kminor_start_lower_gpu, nminorlower_int_size));
        cuda_safe_call(hipMalloc((void**)& kminor_start_upper_gpu, nminorupper_int_size));
        cuda_safe_call(hipMalloc((void**)& tropo_gpu, tropo_size));
        cuda_safe_call(hipMalloc((void**)& col_mix_gpu, col_mix_size));
        cuda_safe_call(hipMalloc((void**)& fmajor_gpu, fmajor_size));
        cuda_safe_call(hipMalloc((void**)& fminor_gpu, fminor_size));
        cuda_safe_call(hipMalloc((void**)& play_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void**)& tlay_gpu, collay_tf_size));
        cuda_safe_call(hipMalloc((void**)& col_gas_gpu, col_gas_size));
        cuda_safe_call(hipMalloc((void**)& jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void**)& jtemp_gpu, collay_int_size));
        cuda_safe_call(hipMalloc((void**)& jpress_gpu, collay_int_size));
        cuda_safe_call(hipMalloc((void**)& itropo_lower_gpu, itropo_size));
        cuda_safe_call(hipMalloc((void**)& itropo_upper_gpu, itropo_size));
        cuda_safe_call(hipMalloc((void**)& tau_gpu, tau_size));
        cuda_safe_call(hipMalloc((void**)& tau_major_gpu, tau_size));
        cuda_safe_call(hipMalloc((void**)& tau_minor_gpu, tau_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(gpoint_flavor_gpu, gpoint_flavor.ptr(), gpoint_flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(band_lims_gpt_gpu, band_lims_gpt.ptr(), band_lims_gpt_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kmajor_gpu, kmajor.ptr(), kmajor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_lower_gpu, kminor_lower.ptr(), kminor_lower_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_upper_gpu, kminor_upper.ptr(), kminor_upper_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_limits_gpt_lower_gpu, minor_limits_gpt_lower.ptr(), minor_limits_gpt_lower_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_limits_gpt_upper_gpu, minor_limits_gpt_upper.ptr(), minor_limits_gpt_upper_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_scales_with_density_lower_gpu, minor_scales_with_density_lower.ptr(), nminorlower_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(minor_scales_with_density_upper_gpu, minor_scales_with_density_upper.ptr(), nminorupper_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(scale_by_complement_lower_gpu, scale_by_complement_lower.ptr(), nminorlower_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(scale_by_complement_upper_gpu, scale_by_complement_upper.ptr(), nminorupper_bool_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_lower_gpu, idx_minor_lower.ptr(), nminorlower_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_upper_gpu, idx_minor_upper.ptr(), nminorupper_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_scaling_lower_gpu, idx_minor_scaling_lower.ptr(), nminorlower_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(idx_minor_scaling_upper_gpu, idx_minor_scaling_upper.ptr(), nminorupper_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_start_lower_gpu, kminor_start_lower.ptr(), nminorlower_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(kminor_start_upper_gpu, kminor_start_upper.ptr(), nminorupper_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tropo_gpu, tropo.ptr(), tropo_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_mix_gpu, col_mix.ptr(), col_mix_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fmajor_gpu, fmajor.ptr(), fmajor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fminor_gpu, fminor.ptr(), fminor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(play_gpu, play.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tlay_gpu, tlay.ptr(), collay_tf_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(col_gas_gpu, col_gas.ptr(), col_gas_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jeta_gpu, jeta.ptr(), jeta_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jtemp_gpu, jtemp.ptr(), collay_int_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jpress_gpu, jpress.ptr(), collay_int_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        // Call the kernel.
        const int block_bnd_maj = 14;
        const int block_lay_maj = 1;
        const int block_col_maj = 32;

        const int grid_bnd_maj  = nband/block_bnd_maj + (nband%block_bnd_maj > 0);
        const int grid_lay_maj  = nlay/block_lay_maj + (nlay%block_lay_maj > 0);
        const int grid_col_maj  = ncol/block_col_maj + (ncol%block_col_maj > 0);

        dim3 grid_gpu_maj(grid_bnd_maj, grid_lay_maj, grid_col_maj);
        dim3 block_gpu_maj(block_bnd_maj, block_lay_maj, block_col_maj);

        compute_tau_major_absorption_kernel<<<grid_gpu_maj, block_gpu_maj>>>(
                ncol, nlay, nband, ngpt,
                nflav, neta, npres, ntemp,
                gpoint_flavor_gpu, band_lims_gpt_gpu,
                kmajor_gpu, col_mix_gpu, fmajor_gpu, jeta_gpu,
                tropo_gpu, jtemp_gpu, jpress_gpu,
                tau_gpu, tau_major_gpu);

        const int nscale_lower = scale_by_complement_lower.dim(1);
        const int nscale_upper = scale_by_complement_upper.dim(1);
        const int block_lay_min = 32;
        const int block_col_min = 32;

        const int grid_lay_min  = nlay/block_lay_min + (nlay%block_lay_min > 0);
        const int grid_col_min  = ncol/block_col_min + (ncol%block_col_min > 0);

        dim3 grid_gpu_min(grid_lay_min, grid_col_min);
        dim3 block_gpu_min(block_lay_min, block_col_min);

        compute_tau_minor_absorption_kernel<<<grid_gpu_min, block_gpu_min>>>(
                ncol, nlay, ngpt,
                ngas, nflav, ntemp, neta,
                nscale_lower, nscale_upper,
                nminorlower, nminorupper,
                nminorklower,nminorkupper,
                idx_h2o,
                gpoint_flavor_gpu,
                kminor_lower_gpu, kminor_upper_gpu,
                minor_limits_gpt_lower_gpu, minor_limits_gpt_upper_gpu,
                minor_scales_with_density_lower_gpu, minor_scales_with_density_upper_gpu,
                scale_by_complement_lower_gpu, scale_by_complement_upper_gpu,
                idx_minor_lower_gpu, idx_minor_upper_gpu,
                idx_minor_scaling_lower_gpu, idx_minor_scaling_upper_gpu,
                kminor_start_lower_gpu, kminor_start_upper_gpu,
                play_gpu, tlay_gpu, col_gas_gpu,
                fminor_gpu, jeta_gpu, jtemp_gpu,
                tropo_gpu, tau_gpu, tau_minor_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU compute_tau_abs: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(tau.ptr(), tau_gpu, tau_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(gpoint_flavor_gpu));
        cuda_safe_call(hipFree(band_lims_gpt_gpu));
        cuda_safe_call(hipFree(kmajor_gpu));
        cuda_safe_call(hipFree(kminor_lower_gpu));
        cuda_safe_call(hipFree(kminor_upper_gpu));
        cuda_safe_call(hipFree(minor_limits_gpt_lower_gpu));
        cuda_safe_call(hipFree(minor_limits_gpt_upper_gpu));
        cuda_safe_call(hipFree(minor_scales_with_density_lower_gpu));
        cuda_safe_call(hipFree(minor_scales_with_density_upper_gpu));
        cuda_safe_call(hipFree(scale_by_complement_lower_gpu));
        cuda_safe_call(hipFree(scale_by_complement_upper_gpu));
        cuda_safe_call(hipFree(idx_minor_lower_gpu));
        cuda_safe_call(hipFree(idx_minor_upper_gpu));
        cuda_safe_call(hipFree(idx_minor_scaling_lower_gpu));
        cuda_safe_call(hipFree(idx_minor_scaling_upper_gpu));
        cuda_safe_call(hipFree(kminor_start_lower_gpu));
        cuda_safe_call(hipFree(kminor_start_upper_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
        cuda_safe_call(hipFree(col_mix_gpu));
        cuda_safe_call(hipFree(fmajor_gpu));
        cuda_safe_call(hipFree(fminor_gpu));
        cuda_safe_call(hipFree(play_gpu));
        cuda_safe_call(hipFree(tlay_gpu));
        cuda_safe_call(hipFree(col_gas_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(jpress_gpu));
        cuda_safe_call(hipFree(itropo_lower_gpu));
        cuda_safe_call(hipFree(itropo_upper_gpu));
        cuda_safe_call(hipFree(tau_major_gpu));
        cuda_safe_call(hipFree(tau_minor_gpu));
        cuda_safe_call(hipFree(tau_gpu));
    }

    template<typename TF>
    void Planck_source(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int nPlanckTemp,
            const Array<TF,2>& tlay, 
            const Array<TF,2>& tlev,
            const Array<TF,1>& tsfc,
            const int sfc_lay,
            const Array<TF,6>& fmajor, 
            const Array<int,4>& jeta,
            const Array<BOOL_TYPE,2>& tropo, 
            const Array<int,2>& jtemp,
            const Array<int,2>& jpress, 
            const Array<int,1>& gpoint_bands,
            const Array<int,2>& band_lims_gpt, 
            const Array<TF,4>& pfracin,
            const TF temp_ref_min, const TF totplnk_delta,
            const Array<TF,2>& totplnk, 
            const Array<int,2>& gpoint_flavor,
            const TF delta_Tsurf,
            Array<TF,2>& sfc_src, 
            Array<TF,3>& lay_src,
            Array<TF,3>& lev_src_inc, 
            Array<TF,3>& lev_src_dec,
            Array<TF,2>& sfc_src_jac, 
            Array<TF,3>& pfrac)
    {
        TF ones[2] = {TF(1.), TF(1.)}; 
        
        float elapsedtime; 
        const int ones_size = 2 * sizeof(TF);
        const int tlay_size = tlay.size() * sizeof(TF);
        const int tlev_size = tlev.size() * sizeof(TF);
        const int tsfc_size = tsfc.size() * sizeof(TF);
        const int fmajor_size = fmajor.size() * sizeof(TF);
        const int pfracin_size = pfracin.size() * sizeof(TF);
        const int totplnk_size = totplnk.size() * sizeof(TF);
        const int sfc_src_size = sfc_src.size() * sizeof(TF);
        const int lay_src_size = lay_src.size() * sizeof(TF);
        const int lev_src_inc_size = lev_src_inc.size() * sizeof(TF);
        const int lev_src_dec_size = lev_src_dec.size() * sizeof(TF);
        const int sfc_src_jac_size = sfc_src_jac.size() * sizeof(TF);
        const int pfrac_size = pfrac.size() * sizeof(TF);
        const int jeta_size = jeta.size() * sizeof(int);
        const int jtemp_size = jtemp.size() * sizeof(int);
        const int jpress_size = jpress.size() * sizeof(int);
        const int gpoint_bands_size = gpoint_bands.size() * sizeof(int);
        const int band_lims_gpt_size = band_lims_gpt.size() * sizeof(int);
        const int gpoint_flavor_size = gpoint_flavor.size() * sizeof(int);
        const int tropo_size = tropo.size() * sizeof(BOOL_TYPE);

        TF* tlay_gpu;
        TF* tlev_gpu;
        TF* tsfc_gpu;
        TF* fmajor_gpu;
        TF* pfracin_gpu;
        TF* totplnk_gpu;
        TF* sfc_src_gpu;
        TF* lay_src_gpu;
        TF* lev_src_inc_gpu;
        TF* lev_src_dec_gpu;
        TF* sfc_src_jac_gpu;
        TF* pfrac_gpu;
        TF* ones_gpu;
        int* jeta_gpu;
        int* jtemp_gpu;
        int* jpress_gpu;
        int* gpoint_bands_gpu;
        int* band_lims_gpt_gpu;
        int* gpoint_flavor_gpu;
        BOOL_TYPE* tropo_gpu;

        cuda_safe_call(hipMalloc((void**)& tlay_gpu, tlay_size));
        cuda_safe_call(hipMalloc((void**)& tlev_gpu, tlev_size));
        cuda_safe_call(hipMalloc((void**)& tsfc_gpu, tsfc_size));
        cuda_safe_call(hipMalloc((void**)& fmajor_gpu, fmajor_size));
        cuda_safe_call(hipMalloc((void**)& pfracin_gpu, pfracin_size));
        cuda_safe_call(hipMalloc((void**)& totplnk_gpu, totplnk_size));
        cuda_safe_call(hipMalloc((void**)& sfc_src_gpu, sfc_src_size));
        cuda_safe_call(hipMalloc((void**)& lay_src_gpu, lay_src_size));
        cuda_safe_call(hipMalloc((void**)& lev_src_inc_gpu, lev_src_inc_size));
        cuda_safe_call(hipMalloc((void**)& lev_src_dec_gpu, lev_src_dec_size));
        cuda_safe_call(hipMalloc((void**)& sfc_src_jac_gpu, sfc_src_jac_size));
        cuda_safe_call(hipMalloc((void**)& pfrac_gpu, pfrac_size));
        cuda_safe_call(hipMalloc((void**)& ones_gpu, ones_size));
        cuda_safe_call(hipMalloc((void**)& jeta_gpu, jeta_size));
        cuda_safe_call(hipMalloc((void**)& jtemp_gpu, jtemp_size));
        cuda_safe_call(hipMalloc((void**)& jpress_gpu, jpress_size));
        cuda_safe_call(hipMalloc((void**)& gpoint_bands_gpu, gpoint_bands_size));
        cuda_safe_call(hipMalloc((void**)& band_lims_gpt_gpu, band_lims_gpt_size));
        cuda_safe_call(hipMalloc((void**)& gpoint_flavor_gpu, gpoint_flavor_size));
        cuda_safe_call(hipMalloc((void**)& tropo_gpu, tropo_size));

        // Copy the data to the GPU.
        cuda_safe_call(hipMemcpy(ones_gpu, ones, ones_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tlay_gpu, tlay.ptr(), tlay_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tlev_gpu, tlev.ptr(), tlev_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tsfc_gpu, tsfc.ptr(), tsfc_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fmajor_gpu, fmajor.ptr(), fmajor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(pfracin_gpu, pfracin.ptr(), pfracin_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(totplnk_gpu, totplnk.ptr(), totplnk_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jeta_gpu, jeta.ptr(), jeta_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jtemp_gpu, jtemp.ptr(), jtemp_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(jpress_gpu, jpress.ptr(), jpress_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(gpoint_bands_gpu, gpoint_bands.ptr(), gpoint_bands_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(band_lims_gpt_gpu, band_lims_gpt.ptr(), band_lims_gpt_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(gpoint_flavor_gpu, gpoint_flavor.ptr(), gpoint_flavor_size, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(tropo_gpu, tropo.ptr(), tropo_size, hipMemcpyHostToDevice));

        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        // Call the kernel.
        const int block_bnd = 14;
        const int block_lay = 1;
        const int block_col = 32;

        const int grid_bnd  = nbnd/block_bnd + (nbnd%block_bnd > 0);
        const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);

        dim3 grid_gpu(grid_bnd, grid_lay, grid_col);
        dim3 block_gpu(block_bnd, block_lay, block_col);

        Planck_source_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, nbnd, ngpt,
                nflav, neta, npres, ntemp, nPlanckTemp,
                tlay_gpu, tlev_gpu, tsfc_gpu, sfc_lay,
                fmajor_gpu, jeta_gpu, tropo_gpu, jtemp_gpu,
                jpress_gpu, gpoint_bands_gpu, band_lims_gpt_gpu,
                pfracin_gpu, temp_ref_min, totplnk_delta,
                totplnk_gpu, gpoint_flavor_gpu, ones_gpu, 
                delta_Tsurf, sfc_src_gpu, lay_src_gpu,
                lev_src_inc_gpu, lev_src_dec_gpu,
                sfc_src_jac_gpu, pfrac_gpu);

        cuda_check_error();
        cuda_safe_call(hipDeviceSynchronize());
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedtime,startEvent,stopEvent);
        std::cout<<"GPU compute_Planck: "<<elapsedtime<<" (ms)"<<std::endl;

        // Copy back the results.
        cuda_safe_call(hipMemcpy(sfc_src.ptr(), sfc_src_gpu, sfc_src_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(lay_src.ptr(), lay_src_gpu, lay_src_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(lev_src_inc.ptr(), lev_src_inc_gpu, lev_src_inc_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(lev_src_dec.ptr(), lev_src_dec_gpu, lev_src_dec_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(sfc_src_jac.ptr(), sfc_src_jac_gpu, sfc_src_jac_size, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(pfrac.ptr(), pfrac_gpu, pfrac_size, hipMemcpyDeviceToHost));

        // Deallocate a CUDA array.
        cuda_safe_call(hipFree(tlay_gpu));
        cuda_safe_call(hipFree(tlev_gpu));
        cuda_safe_call(hipFree(tsfc_gpu));
        cuda_safe_call(hipFree(fmajor_gpu));
        cuda_safe_call(hipFree(pfracin_gpu));
        cuda_safe_call(hipFree(totplnk_gpu));
        cuda_safe_call(hipFree(sfc_src_gpu));
        cuda_safe_call(hipFree(lay_src_gpu));
        cuda_safe_call(hipFree(lev_src_inc_gpu));
        cuda_safe_call(hipFree(lev_src_dec_gpu));
        cuda_safe_call(hipFree(sfc_src_jac_gpu));
        cuda_safe_call(hipFree(pfrac_gpu));
        cuda_safe_call(hipFree(ones_gpu));
        cuda_safe_call(hipFree(jeta_gpu));
        cuda_safe_call(hipFree(jtemp_gpu));
        cuda_safe_call(hipFree(jpress_gpu));
        cuda_safe_call(hipFree(gpoint_bands_gpu));
        cuda_safe_call(hipFree(band_lims_gpt_gpu));
        cuda_safe_call(hipFree(gpoint_flavor_gpu));
        cuda_safe_call(hipFree(tropo_gpu));
    }

}


#ifdef FLOAT_SINGLE_RRTMGP
template void rrtmgp_kernel_launcher_cuda::fill_gases<float>(
            const int ncol, const int nlay, const int ngas, 
            Array<float,3>&, const Array<float,2>&,
            Array<float,3>&, const Array<float,2>&);

template void rrtmgp_kernel_launcher_cuda::reorder123x321<float>(const int, const int, const int, const Array<float,3>&, Array<float,3>&);
template void rrtmgp_kernel_launcher_cuda::reorder12x21<float>(const int, const int, const Array<float,2>&, Array<float,2>&);

template void rrtmgp_kernel_launcher_cuda::zero_array<float>(const int, const int, const int, Array<float,3>&);

template void rrtmgp_kernel_launcher_cuda::interpolation<float>(
        const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<float,1>&, const Array<float,1>&,
        float, float, float, float, const Array<float,3>&, const Array<float,2>&,
        const Array<float,2>&, Array<float,3>&, Array<int,2>&, Array<float,6>&, Array<float,5>&,
        Array<float,4>&, Array<BOOL_TYPE,2>&, Array<int,4>&, Array<int,2>&);

template void rrtmgp_kernel_launcher_cuda::combine_and_reorder_2str<float>(
        const int, const int, const int, const Array<float,3>&, const Array<float,3>&, Array<float,3>&, Array<float,3>&, Array<float,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_rayleigh<float>(
        const int, const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<float,4>&, int, const Array<float,2>&, 
        const Array<float,3>&, const Array<float,5>&, const Array<int,4>&, const Array<BOOL_TYPE,2>&, 
        const Array<int,2>&, Array<float,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_absorption<float>(const int, const int, const int, const int, const int, const int, 
	const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<float,4>&, const Array<float,3>&, const Array<float,3>&,
        const Array<int,2>&, const Array<int,2>&, const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&,
        const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&, const Array<int,1>&, const Array<int,1>&,
        const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<BOOL_TYPE,2>& tropo,
        const Array<float,4>&, const Array<float,6>&, const Array<float,5>&, const Array<float,2>&, const Array<float,2>&, const Array<float,3>&,
        const Array<int,4>&, const Array<int,2>&, const Array<int,2>&, Array<float,3>&);

template void rrtmgp_kernel_launcher_cuda::Planck_source<float>(const int ncol, const int nlay, const int nbnd, const int ngpt,
        const int nflav, const int neta, const int npres, const int ntemp,
        const int nPlanckTemp, const Array<float,2>& tlay, const Array<float,2>& tlev,
        const Array<float,1>& tsfc, const int sfc_lay, const Array<float,6>& fmajor, 
        const Array<int,4>& jeta, const Array<BOOL_TYPE,2>& tropo, const Array<int,2>& jtemp,
        const Array<int,2>& jpress, const Array<int,1>& gpoint_bands, const Array<int,2>& band_lims_gpt, 
        const Array<float,4>& pfracin, const float temp_ref_min, const float totplnk_delta,
        const Array<float,2>& totplnk, const Array<int,2>& gpoint_flavor, const float delta_Tsurf,
        Array<float,2>& sfc_src,  Array<float,3>& lay_src, Array<float,3>& lev_src_inc, 
        Array<float,3>& lev_src_dec, Array<float,2>& sfc_src_jac, Array<float,3>& pfrac)
	    
#else
template void rrtmgp_kernel_launcher_cuda::fill_gases<double>(
            const int ncol, const int nlay, const int ngas, 
            Array<double,3>&,
            Array<double,3>&, const Array<double,2>&,
            const Gas_concs<double>&, const Array<std::string,1>&);

template void rrtmgp_kernel_launcher_cuda::reorder123x321<double>(const int, const int, const int, const Array<double,3>&, Array<double,3>&);
template void rrtmgp_kernel_launcher_cuda::reorder12x21<double>(const int, const int, const Array<double,2>&, Array<double,2>&);

template void rrtmgp_kernel_launcher_cuda::zero_array<double>(const int, const int, const int, Array<double,3>&);

template void rrtmgp_kernel_launcher_cuda::interpolation<double>(
        const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<double,1>&, const Array<double,1>&,
        double, double, double, double, const Array<double,3>&, const Array<double,2>&,
        const Array<double,2>&, Array<double,3>&, Array<int,2>&, Array<double,6>&, Array<double,5>&,
        Array<double,4>&, Array<BOOL_TYPE,2>&, Array<int,4>&, Array<int,2>&);

template void rrtmgp_kernel_launcher_cuda::combine_and_reorder_2str<double>(
        const int, const int, const int, const Array<double,3>&, const Array<double,3>&, Array<double,3>&, Array<double,3>&, Array<double,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_rayleigh<double>(
        const int, const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<double,4>&, int, const Array<double,2>&, 
        const Array<double,3>&, const Array<double,5>&, const Array<int,4>&, const Array<BOOL_TYPE,2>&, 
        const Array<int,2>&, Array<double,3>&);

template void rrtmgp_kernel_launcher_cuda::compute_tau_absorption<double>(const int, const int, const int, const int, const int, const int, 
	const int, const int, const int, const int, const int, const int, const int, const int,
        const Array<int,2>&, const Array<int,2>&, const Array<double,4>&, const Array<double,3>&, const Array<double,3>&,
        const Array<int,2>&, const Array<int,2>&, const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&,
        const Array<BOOL_TYPE,1>&, const Array<BOOL_TYPE,1>&, const Array<int,1>&, const Array<int,1>&,
        const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<int,1>&, const Array<BOOL_TYPE,2>& tropo,
        const Array<double,4>&, const Array<double,6>&, const Array<double,5>&, const Array<double,2>&, const Array<double,2>&, const Array<double,3>&,
        const Array<int,4>&, const Array<int,2>&, const Array<int,2>&, Array<double,3>&);

template void rrtmgp_kernel_launcher_cuda::Planck_source<double>(const int ncol, const int nlay, const int nbnd, const int ngpt,
        const int nflav, const int neta, const int npres, const int ntemp,
        const int nPlanckTemp, const Array<double,2>& tlay, const Array<double,2>& tlev,
        const Array<double,1>& tsfc, const int sfc_lay, const Array<double,6>& fmajor, 
        const Array<int,4>& jeta, const Array<BOOL_TYPE,2>& tropo, const Array<int,2>& jtemp,
        const Array<int,2>& jpress, const Array<int,1>& gpoint_bands, const Array<int,2>& band_lims_gpt, 
        const Array<double,4>& pfracin, const double temp_ref_min, const double totplnk_delta,
        const Array<double,2>& totplnk, const Array<int,2>& gpoint_flavor, const double delta_Tsurf,
        Array<double,2>& sfc_src,  Array<double,3>& lay_src, Array<double,3>& lev_src_inc, 
        Array<double,3>& lev_src_dec, Array<double,2>& sfc_src_jac, Array<double,3>& pfrac);

#endif


